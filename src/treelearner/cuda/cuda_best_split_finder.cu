
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include <LightGBM/cuda/cuda_algorithms.hpp>
#include "cuda_best_split_finder.hpp"

namespace LightGBM {

__device__ void ReduceBestGainWarp(double gain, bool found, uint32_t thread_index, double* out_gain, bool* out_found, uint32_t* out_thread_index) {
  const uint32_t mask = 0xffffffff;
  const uint32_t warpLane = threadIdx.x % warpSize;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const bool other_found = __shfl_down_sync(mask, found, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    const uint32_t other_thread_index = __shfl_down_sync(mask, thread_index, offset);
    if ((other_found && found && other_gain > gain) || (!found && other_found)) {
      found = other_found;
      gain = other_gain;
      thread_index = other_thread_index;
    }
  }
  if (warpLane == 0) {
    *out_gain = gain;
    *out_found = found;
    *out_thread_index = thread_index;
  }
}

__device__ uint32_t ReduceBestGainBlock(double gain, bool found, uint32_t thread_index) {
  const uint32_t mask = 0xffffffff;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const bool other_found = __shfl_down_sync(mask, found, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    const uint32_t other_thread_index = __shfl_down_sync(mask, thread_index, offset);
    if ((other_found && found && other_gain > gain) || (!found && other_found)) {
      found = other_found;
      gain = other_gain;
      thread_index = other_thread_index;
    }
  }
  return thread_index;
}

__device__ uint32_t ReduceBestGain(double gain, bool found, uint32_t thread_index,
    double* shared_gain_buffer, bool* shared_found_buffer, uint32_t* shared_thread_index_buffer) {
  const uint32_t warpID = threadIdx.x / warpSize;
  const uint32_t warpLane = threadIdx.x % warpSize;
  const uint32_t num_warp = blockDim.x / warpSize;
  ReduceBestGainWarp(gain, found, thread_index, shared_gain_buffer + warpID, shared_found_buffer + warpID, shared_thread_index_buffer + warpID);
  __syncthreads();
  if (warpID == 0) {
    gain = warpLane < num_warp ? shared_gain_buffer[warpLane] : kMinScore;
    found = warpLane < num_warp ? shared_found_buffer[warpLane] : false;
    thread_index = warpLane < num_warp ? shared_thread_index_buffer[warpLane] : 0;
    thread_index = ReduceBestGainBlock(gain, found, thread_index);
  }
  return thread_index;
}

__device__ void ReduceBestGainForLeaves(double* gain, int* leaves, int cuda_cur_num_leaves) {
  const unsigned int tid = threadIdx.x;
  for (unsigned int s = 1; s < cuda_cur_num_leaves; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < cuda_cur_num_leaves) {
      const uint32_t tid_s = tid + s;
      if ((leaves[tid] == -1 && leaves[tid_s] != -1) || (leaves[tid] != -1 && leaves[tid_s] != -1 && gain[tid_s] > gain[tid])) {
        gain[tid] = gain[tid_s];
        leaves[tid] = leaves[tid_s];
      }
    }
    __syncthreads();
  }
}

__device__ void ReduceBestGainForLeavesWarp(double gain, int leaf_index, double* out_gain, int* out_leaf_index) {
  const uint32_t mask = 0xffffffff;
  const uint32_t warpLane = threadIdx.x % warpSize;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const int other_leaf_index = __shfl_down_sync(mask, leaf_index, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    if ((leaf_index != -1 && other_leaf_index != -1 && other_gain > gain) || (leaf_index == -1 && other_leaf_index != -1)) {
      gain = other_gain;
      leaf_index = other_leaf_index;
    }
  }
  if (warpLane == 0) {
    *out_gain = gain;
    *out_leaf_index = leaf_index;
  }
}

__device__ int ReduceBestGainForLeavesBlock(double gain, int leaf_index) {
  const uint32_t mask = 0xffffffff;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const int other_leaf_index = __shfl_down_sync(mask, leaf_index, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    if ((leaf_index != -1 && other_leaf_index != -1 && other_gain > gain) || (leaf_index == -1 && other_leaf_index != -1)) {
      gain = other_gain;
      leaf_index = other_leaf_index;
    }
  }
  return leaf_index;
}

__device__ int ReduceBestGainForLeaves(double gain, int leaf_index, double* shared_gain_buffer, int* shared_leaf_index_buffer) {
  const uint32_t warpID = threadIdx.x / warpSize;
  const uint32_t warpLane = threadIdx.x % warpSize;
  const uint32_t num_warp = blockDim.x / warpSize;
  ReduceBestGainForLeavesWarp(gain, leaf_index, shared_gain_buffer + warpID, shared_leaf_index_buffer + warpID);
  __syncthreads();
  if (warpID == 0) {
    gain = warpLane < num_warp ? shared_gain_buffer[warpLane] : 0.0f;
    leaf_index = warpLane < num_warp ? shared_leaf_index_buffer[warpLane] : -1;
    leaf_index = ReduceBestGainForLeavesBlock(gain, leaf_index);
  }
  return leaf_index;
}

__device__ double ThresholdL1(double s, double l1) {
  const double reg_s = fmax(0.0, fabs(s) - l1);
  if (s >= 0.0f) {
    return reg_s;
  } else {
    return -reg_s;
  }
}

__device__ double CalculateSplittedLeafOutput(double sum_gradients,
                                          double sum_hessians, double l1, const bool use_l1,
                                          double l2) {
  double ret;
  if (use_l1) {
    ret = -ThresholdL1(sum_gradients, l1) / (sum_hessians + l2);
  } else {
    ret = -sum_gradients / (sum_hessians + l2);
  }
  return ret;
}

__device__ double GetLeafGainGivenOutput(double sum_gradients,
                                      double sum_hessians, double l1, const bool use_l1,
                                      double l2, double output) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return -(2.0 * sg_l1 * output + (sum_hessians + l2) * output * output);
  } else {
    return -(2.0 * sum_gradients * output +
              (sum_hessians + l2) * output * output);
  }
}

__device__ double GetLeafGain(double sum_gradients, double sum_hessians,
                          double l1, const bool use_l1, double l2) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return (sg_l1 * sg_l1) / (sum_hessians + l2);
  } else {
    return (sum_gradients * sum_gradients) / (sum_hessians + l2);
  }
}

__device__ double GetSplitGains(double sum_left_gradients,
                            double sum_left_hessians,
                            double sum_right_gradients,
                            double sum_right_hessians,
                            double l1, const bool use_l1, double l2) {
  return GetLeafGain(sum_left_gradients,
                     sum_left_hessians,
                     l1, use_l1, l2) +
         GetLeafGain(sum_right_gradients,
                     sum_right_hessians,
                     l1, use_l1, l2);
}

__device__ void FindBestSplitsForLeafKernelInner(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // input task information
  const bool reverse,
  const bool skip_default_bin,
  const bool na_as_missing,
  const uint8_t assume_out_default_left,
  // output parameters
  CUDASplitInfo* cuda_best_split_info) {

  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;

  cuda_best_split_info->is_valid = false;

  __shared__ hist_t shared_mem_buffer[32];
  hist_t local_grad_hist = 0.0f;
  hist_t local_hess_hist = 0.0f;
  double local_gain = 0.0f;
  bool threshold_found = false;
  uint32_t threshold_value = 0;
  __shared__ uint32_t best_thread_index;
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  const unsigned int threadIdx_x = threadIdx.x;
  const bool skip_sum = reverse ?
    (skip_default_bin && (feature_num_bin - 1 - threadIdx_x) == static_cast<int>(feature_default_bin)) :
    (skip_default_bin && (threadIdx_x + feature_mfb_offset) == static_cast<int>(feature_default_bin));
  const uint32_t feature_num_bin_minus_offset = feature_num_bin - feature_mfb_offset;
  if (!reverse) {
    if (threadIdx_x < feature_num_bin_minus_offset && !skip_sum) {
      const unsigned int bin_offset = threadIdx_x << 1;
      local_grad_hist = feature_hist_ptr[bin_offset];
      local_hess_hist = feature_hist_ptr[bin_offset + 1];
    }
  } else {
    if (threadIdx_x < feature_num_bin_minus_offset && !skip_sum) {
      const unsigned int read_index = feature_num_bin_minus_offset - 1 - threadIdx_x;
      const unsigned int bin_offset = read_index << 1;
      local_grad_hist = feature_hist_ptr[bin_offset];
      local_hess_hist = feature_hist_ptr[bin_offset + 1];
    }
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    local_hess_hist += kEpsilon;
  }
  local_gain = kMinScore;
  local_grad_hist = ShufflePrefixSum(local_grad_hist, shared_mem_buffer);
  __syncthreads();
  local_hess_hist = ShufflePrefixSum(local_hess_hist, shared_mem_buffer);
  if (reverse) {
    if (threadIdx_x >= static_cast<unsigned int>(na_as_missing) && threadIdx_x <= feature_num_bin - 2 && !skip_sum) {
      const double sum_right_gradient = local_grad_hist;
      const double sum_right_hessian = local_hess_hist;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian;
      const data_size_t left_count = num_data - right_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain > min_gain_shift) {
          local_gain = current_gain - min_gain_shift;
          threshold_value = static_cast<uint32_t>(feature_num_bin - 2 - threadIdx_x);
          threshold_found = true;
        }
      }
    }
  } else {
    if (threadIdx_x <= feature_num_bin_minus_offset - 2/* && !skip_sum*/) {
      const double sum_left_gradient = local_grad_hist;
      const double sum_left_hessian = local_hess_hist;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain > min_gain_shift) {
          local_gain = current_gain - min_gain_shift;
          threshold_value = static_cast<uint32_t>(threadIdx_x + feature_mfb_offset);
          threshold_found = true;
        }
      }
    }
  }
  __syncthreads();
  const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
  if (threadIdx_x == 0) {
    best_thread_index = result;
  }
  __syncthreads();
  if (threshold_found && threadIdx_x == best_thread_index) {
    cuda_best_split_info->is_valid = true;
    cuda_best_split_info->threshold = threshold_value;
    cuda_best_split_info->gain = local_gain;
    cuda_best_split_info->default_left = assume_out_default_left;
    if (reverse) {
      const double sum_right_gradient = local_grad_hist;
      const double sum_right_hessian = local_hess_hist - kEpsilon;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian - kEpsilon;
      const data_size_t left_count = num_data - right_count;
      const double left_output = CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    } else {
      const double sum_left_gradient = local_grad_hist;
      const double sum_left_hessian = local_hess_hist - kEpsilon;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian - kEpsilon;
      const data_size_t right_count = num_data - left_count;
      const double left_output = CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    }
  }
}

__global__ void FindBestSplitsForLeafKernel(
  // input feature information
  const uint32_t* feature_hist_offsets,
  const uint8_t* feature_mfb_offsets,
  const uint32_t* feature_default_bins,
  const uint32_t* feature_num_bins,
  // input task information
  const bool larger_only,
  const int num_tasks,
  const int* task_feature_index,
  const uint8_t* task_reverse,
  const uint8_t* task_skip_default_bin,
  const uint8_t* task_na_as_missing,
  const uint8_t* task_out_default_left,
  // input leaf information
  const int smaller_leaf_index,
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const int larger_leaf_index,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  // input config parameter values
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double lambda_l1,
  const double lambda_l2,
  // output
  CUDASplitInfo* cuda_best_split_info) {

  const unsigned int task_index = blockIdx.x % num_tasks;
  const bool is_larger = static_cast<bool>(blockIdx.x >= num_tasks || larger_only);
  const int inner_feature_index = task_feature_index[task_index];
  const bool reverse = static_cast<bool>(task_reverse[task_index]);
  const bool skip_default_bin = static_cast<bool>(task_skip_default_bin[task_index]);
  const bool na_as_missing = static_cast<bool>(task_na_as_missing[task_index]);
  const bool assume_out_default_left = task_out_default_left[task_index];
  const double parent_gain = is_larger ? larger_leaf_splits->gain : smaller_leaf_splits->gain;
  const double sum_gradients = is_larger ? larger_leaf_splits->sum_of_gradients : smaller_leaf_splits->sum_of_gradients;
  const double sum_hessians = (is_larger ? larger_leaf_splits->sum_of_hessians : smaller_leaf_splits->sum_of_hessians) + 2 * kEpsilon;
  const double num_data = is_larger ? larger_leaf_splits->num_data_in_leaf : smaller_leaf_splits->num_data_in_leaf;
  const unsigned int output_offset = is_larger ? (task_index + num_tasks) : task_index;
  CUDASplitInfo* out = cuda_best_split_info + output_offset;
  const hist_t* hist_ptr = (is_larger ? larger_leaf_splits->hist_in_leaf : smaller_leaf_splits->hist_in_leaf) + feature_hist_offsets[inner_feature_index] * 2;
  FindBestSplitsForLeafKernelInner(
    // input feature information
    hist_ptr,
    feature_num_bins[inner_feature_index],
    feature_mfb_offsets[inner_feature_index],
    feature_default_bins[inner_feature_index],
    inner_feature_index,
    // input config parameter values
    lambda_l1,
    lambda_l2,
    min_data_in_leaf,
    min_sum_hessian_in_leaf,
    min_gain_to_split,
    // input parent node information
    parent_gain,
    sum_gradients,
    sum_hessians,
    num_data,
    // input task information
    reverse,
    skip_default_bin,
    na_as_missing,
    assume_out_default_left,
    // output parameters
    out);
}

void CUDABestSplitFinder::LaunchFindBestSplitsForLeafKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {
  if (!is_smaller_leaf_valid && !is_larger_leaf_valid) {
    return;
  }
  bool larger_only = false;
  if (!is_smaller_leaf_valid) {
    larger_only = true;
  }
  if (!larger_only) {
    FindBestSplitsForLeafKernel<<<num_tasks_, MAX_NUM_BIN_IN_FEATURE, 0, cuda_streams_[0]>>>(
      // input feature information
      cuda_feature_hist_offsets_,
      cuda_feature_mfb_offsets_,
      cuda_feature_default_bins_,
      cuda_feature_num_bins_,
      // input task information
      larger_only,
      num_tasks_,
      cuda_task_feature_index_,
      cuda_task_reverse_,
      cuda_task_skip_default_bin_,
      cuda_task_na_as_missing_,
      cuda_task_out_default_left_,
      // input leaf information
      smaller_leaf_index,
      smaller_leaf_splits,
      larger_leaf_index,
      larger_leaf_splits,
      // configuration parameter values
      min_data_in_leaf_,
      min_sum_hessian_in_leaf_,
      min_gain_to_split_,
      lambda_l1_,
      lambda_l2_,
      // output parameters
      cuda_best_split_info_);
  }
  SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
  if (larger_leaf_index >= 0) {
    FindBestSplitsForLeafKernel<<<num_tasks_, MAX_NUM_BIN_IN_FEATURE, 0, cuda_streams_[1]>>>(
      // input feature information
      cuda_feature_hist_offsets_,
      cuda_feature_mfb_offsets_,
      cuda_feature_default_bins_,
      cuda_feature_num_bins_,
      // input task information
      true,
      num_tasks_,
      cuda_task_feature_index_,
      cuda_task_reverse_,
      cuda_task_skip_default_bin_,
      cuda_task_na_as_missing_,
      cuda_task_out_default_left_,
      // input leaf information
      smaller_leaf_index,
      smaller_leaf_splits,
      larger_leaf_index,
      larger_leaf_splits,
      // configuration parameter values
      min_data_in_leaf_,
      min_sum_hessian_in_leaf_,
      min_gain_to_split_,
      lambda_l1_,
      lambda_l2_,
      // output parameters
      cuda_best_split_info_);
  }
}

__device__ void ReduceBestSplit(bool* found, double* gain, uint32_t* shared_read_index,
  uint32_t num_features_aligned) {
  const uint32_t threadIdx_x = threadIdx.x;
  for (unsigned int s = 1; s < num_features_aligned; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < num_features_aligned) {
      const uint32_t pos_to_compare = threadIdx_x + s;
      if ((!found[threadIdx_x] && found[pos_to_compare]) ||
        (found[threadIdx_x] && found[pos_to_compare] && gain[threadIdx_x] < gain[pos_to_compare])) {
        found[threadIdx_x] = found[pos_to_compare];
        gain[threadIdx_x] = gain[pos_to_compare];
        shared_read_index[threadIdx_x] = shared_read_index[pos_to_compare];
      }
    }
    __syncthreads();
  } 
}

__global__ void SyncBestSplitForLeafKernel(const int smaller_leaf_index, const int larger_leaf_index,
  CUDASplitInfo* cuda_leaf_best_split_info,
  // input parameters
  const int* cuda_task_feature_index,
  const CUDASplitInfo* cuda_best_split_info,
  const uint32_t* cuda_feature_default_bins,
  const int num_tasks,
  const int num_tasks_aligned,
  const int num_blocks_per_leaf,
  const bool larger_only,
  const int num_leaves) {
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  const uint32_t threadIdx_x = threadIdx.x;
  const uint32_t blockIdx_x = blockIdx.x;

  bool best_found = false;
  double best_gain = kMinScore;
  uint32_t shared_read_index = 0;

  const bool is_smaller = (blockIdx_x < static_cast<unsigned int>(num_blocks_per_leaf) && !larger_only);
  const uint32_t leaf_block_index = (is_smaller || larger_only) ? blockIdx_x : (blockIdx_x - static_cast<unsigned int>(num_blocks_per_leaf));
  const int task_index = static_cast<int>(leaf_block_index * blockDim.x + threadIdx_x);
  const uint32_t read_index = is_smaller ? static_cast<uint32_t>(task_index) : static_cast<uint32_t>(task_index + num_tasks);
  if (task_index < num_tasks) {
    best_found = cuda_best_split_info[read_index].is_valid;
    best_gain = cuda_best_split_info[read_index].gain;
    shared_read_index = read_index;
  } else {
    best_found = false;
  }

  __syncthreads();
  const uint32_t best_read_index = ReduceBestGain(best_gain, best_found, shared_read_index,
      shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
  if (threadIdx.x == 0) {
    const int leaf_index_ref = is_smaller ? smaller_leaf_index : larger_leaf_index;
    const unsigned buffer_write_pos = static_cast<unsigned int>(leaf_index_ref) + leaf_block_index * num_leaves;
    CUDASplitInfo* cuda_split_info = cuda_leaf_best_split_info + buffer_write_pos;
    const CUDASplitInfo* best_split_info = cuda_best_split_info + best_read_index;
    if (best_split_info->is_valid) {
      /*cuda_split_info->gain = best_split_info->gain;
      cuda_split_info->inner_feature_index = is_smaller ? cuda_task_feature_index[best_read_index] :
        cuda_task_feature_index[static_cast<int>(best_read_index) - num_tasks];
      cuda_split_info->default_left = best_split_info->default_left;
      cuda_split_info->threshold = best_split_info->threshold;
      cuda_split_info->left_sum_gradients = best_split_info->left_sum_gradients;
      cuda_split_info->left_sum_hessians = best_split_info->left_sum_hessians;
      cuda_split_info->left_count = best_split_info->left_count;
      cuda_split_info->left_gain = best_split_info->left_gain; 
      cuda_split_info->left_value = best_split_info->left_value;
      cuda_split_info->right_sum_gradients = best_split_info->right_sum_gradients;
      cuda_split_info->right_sum_hessians = best_split_info->right_sum_hessians;
      cuda_split_info->right_count = best_split_info->right_count;
      cuda_split_info->right_gain = best_split_info->right_gain; 
      cuda_split_info->right_value = best_split_info->right_value;
      cuda_split_info->is_valid = true;*/
      *cuda_split_info = *best_split_info;
      cuda_split_info->inner_feature_index = is_smaller ? cuda_task_feature_index[best_read_index] :
        cuda_task_feature_index[static_cast<int>(best_read_index) - num_tasks];
      cuda_split_info->is_valid = true;
    } else {
      cuda_split_info->gain = kMinScore;
      cuda_split_info->is_valid = false;
    }
  }
}

__global__ void SyncBestSplitForLeafKernelAllBlocks(
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const unsigned int num_blocks_per_leaf,
  const int num_leaves,
  CUDASplitInfo* cuda_leaf_best_split_info,
  const bool larger_only) {
  if (!larger_only) {
    if (blockIdx.x == 0) {
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(smaller_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        CUDASplitInfo* smaller_leaf_split_info = cuda_leaf_best_split_info + smaller_leaf_index;
        const CUDASplitInfo* other_split_info = cuda_leaf_best_split_info + leaf_read_pos;
        if ((other_split_info->is_valid && smaller_leaf_split_info->is_valid &&
          other_split_info->gain > smaller_leaf_split_info->gain) ||
            (!smaller_leaf_split_info->is_valid && other_split_info->is_valid)) {
            smaller_leaf_split_info->is_valid = other_split_info->is_valid;
            smaller_leaf_split_info->inner_feature_index = other_split_info->inner_feature_index;
            smaller_leaf_split_info->default_left = other_split_info->default_left;
            smaller_leaf_split_info->threshold = other_split_info->threshold;
            smaller_leaf_split_info->gain = other_split_info->gain;
            smaller_leaf_split_info->left_sum_gradients = other_split_info->left_sum_gradients;
            smaller_leaf_split_info->left_sum_hessians = other_split_info->left_sum_hessians;
            smaller_leaf_split_info->left_count = other_split_info->left_count;
            smaller_leaf_split_info->left_gain = other_split_info->left_gain;
            smaller_leaf_split_info->left_value = other_split_info->left_value;
            smaller_leaf_split_info->right_sum_gradients = other_split_info->right_sum_gradients;
            smaller_leaf_split_info->right_sum_hessians = other_split_info->right_sum_hessians;
            smaller_leaf_split_info->right_count = other_split_info->right_count;
            smaller_leaf_split_info->right_gain = other_split_info->right_gain;
            smaller_leaf_split_info->right_value = other_split_info->right_value;
        }
      }
    }
  }
  if (larger_leaf_index >= 0) {
    if (blockIdx.x == 1 || larger_only) {
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(larger_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        CUDASplitInfo* larger_leaf_split_info = cuda_leaf_best_split_info + larger_leaf_index;
        const CUDASplitInfo* other_split_info = cuda_leaf_best_split_info + leaf_read_pos;
        if ((other_split_info->is_valid && larger_leaf_split_info->is_valid &&
          other_split_info->gain > larger_leaf_split_info->gain) ||
            (!larger_leaf_split_info->is_valid && other_split_info->is_valid)) {
            larger_leaf_split_info->is_valid = other_split_info->is_valid;
            larger_leaf_split_info->inner_feature_index = other_split_info->inner_feature_index;
            larger_leaf_split_info->default_left = other_split_info->default_left;
            larger_leaf_split_info->threshold = other_split_info->threshold;
            larger_leaf_split_info->gain = other_split_info->gain;
            larger_leaf_split_info->left_sum_gradients = other_split_info->left_sum_gradients;
            larger_leaf_split_info->left_sum_hessians = other_split_info->left_sum_hessians;
            larger_leaf_split_info->left_count = other_split_info->left_count;
            larger_leaf_split_info->left_gain = other_split_info->left_gain;
            larger_leaf_split_info->left_value = other_split_info->left_value;
            larger_leaf_split_info->right_sum_gradients = other_split_info->right_sum_gradients;
            larger_leaf_split_info->right_sum_hessians = other_split_info->right_sum_hessians;
            larger_leaf_split_info->right_count = other_split_info->right_count;
            larger_leaf_split_info->right_gain = other_split_info->right_gain;
            larger_leaf_split_info->right_value = other_split_info->right_value;
        }
      }
    }
  }
}

void CUDABestSplitFinder::LaunchSyncBestSplitForLeafKernel(
  const int host_smaller_leaf_index,
  const int host_larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {

  int num_tasks = num_tasks_;
  int num_tasks_aligned = 1;
  num_tasks -= 1;
  while (num_tasks > 0) {
    num_tasks_aligned <<= 1;
    num_tasks >>= 1;
  }
  const int num_blocks_per_leaf = (num_tasks_ + NUM_TASKS_PER_SYNC_BLOCK - 1) / NUM_TASKS_PER_SYNC_BLOCK;
  if (host_larger_leaf_index >= 0 && is_smaller_leaf_valid && is_larger_leaf_valid) {
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[0]>>>(
      host_smaller_leaf_index,
      host_larger_leaf_index,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      false,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[0]>>>(
        host_smaller_leaf_index,
        host_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        false);
    }
    SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[1]>>>(
      host_smaller_leaf_index,
      host_larger_leaf_index,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      true,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[1]>>>(
        host_smaller_leaf_index,
        host_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        true);
    }
  } else {
    const bool larger_only = (!is_smaller_leaf_valid && is_larger_leaf_valid);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK>>>(
      host_smaller_leaf_index,
      host_larger_leaf_index,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      larger_only,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1>>>(
        host_smaller_leaf_index,
        host_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        larger_only);
    }
  }
}

__global__ void FindBestFromAllSplitsKernel(const int cur_num_leaves,
  CUDASplitInfo* cuda_leaf_best_split_info,
  int* cuda_best_split_info_buffer) {
  __shared__ double gain_shared_buffer[32];
  __shared__ int leaf_index_shared_buffer[32];
  double thread_best_gain = kMinScore;
  int thread_best_leaf_index = -1;
  const int threadIdx_x = static_cast<int>(threadIdx.x);
  for (int leaf_index = threadIdx_x; leaf_index < cur_num_leaves; leaf_index += static_cast<int>(blockDim.x)) {
    const double leaf_best_gain = cuda_leaf_best_split_info[leaf_index].gain;
    if (cuda_leaf_best_split_info[leaf_index].is_valid && leaf_best_gain > thread_best_gain) {
      thread_best_gain = leaf_best_gain;
      thread_best_leaf_index = leaf_index;
    }
  }
  const int best_leaf_index = ReduceBestGainForLeaves(thread_best_gain, thread_best_leaf_index, gain_shared_buffer, leaf_index_shared_buffer);
  if (threadIdx_x == 0) {
    cuda_best_split_info_buffer[6] = best_leaf_index;
    if (best_leaf_index != -1) {
      cuda_leaf_best_split_info[best_leaf_index].is_valid = false;
      cuda_leaf_best_split_info[cur_num_leaves].is_valid = false;
    }
  }
}

__global__ void PrepareLeafBestSplitInfo(const int smaller_leaf_index, const int larger_leaf_index,
  int* cuda_best_split_info_buffer,
  const CUDASplitInfo* cuda_leaf_best_split_info) {
  const unsigned int threadIdx_x = blockIdx.x;
  if (threadIdx_x == 0) {
    cuda_best_split_info_buffer[0] = cuda_leaf_best_split_info[smaller_leaf_index].inner_feature_index;
  } else if (threadIdx_x == 1) {
    cuda_best_split_info_buffer[1] = cuda_leaf_best_split_info[smaller_leaf_index].threshold;
  } else if (threadIdx_x == 2) {
    cuda_best_split_info_buffer[2] = cuda_leaf_best_split_info[smaller_leaf_index].default_left;
  }
  if (larger_leaf_index >= 0) { 
    if (threadIdx_x == 3) {
      cuda_best_split_info_buffer[3] = cuda_leaf_best_split_info[larger_leaf_index].inner_feature_index;
    } else if (threadIdx_x == 4) {
      cuda_best_split_info_buffer[4] = cuda_leaf_best_split_info[larger_leaf_index].threshold;
    } else if (threadIdx_x == 5) {
      cuda_best_split_info_buffer[5] = cuda_leaf_best_split_info[larger_leaf_index].default_left;
    }
  }
}

void CUDABestSplitFinder::LaunchFindBestFromAllSplitsKernel(const int cur_num_leaves,
  const int smaller_leaf_index, const int larger_leaf_index, 
  int* smaller_leaf_best_split_feature,
  uint32_t* smaller_leaf_best_split_threshold,
  uint8_t* smaller_leaf_best_split_default_left,
  int* larger_leaf_best_split_feature,
  uint32_t* larger_leaf_best_split_threshold,
  uint8_t* larger_leaf_best_split_default_left,
  int* best_leaf_index) {
  FindBestFromAllSplitsKernel<<<1, NUM_THREADS_FIND_BEST_LEAF, 0, cuda_streams_[1]>>>(cur_num_leaves,
    cuda_leaf_best_split_info_,
    cuda_best_split_info_buffer_);
  PrepareLeafBestSplitInfo<<<6, 1, 0, cuda_streams_[0]>>>(smaller_leaf_index, larger_leaf_index,
    cuda_best_split_info_buffer_,
    cuda_leaf_best_split_info_);
  std::vector<int> host_leaf_best_split_info_buffer(7);
  SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
  CopyFromCUDADeviceToHostOuter<int>(host_leaf_best_split_info_buffer.data(), cuda_best_split_info_buffer_, 7, __FILE__, __LINE__);
  *smaller_leaf_best_split_feature = host_leaf_best_split_info_buffer[0];
  *smaller_leaf_best_split_threshold = static_cast<uint32_t>(host_leaf_best_split_info_buffer[1]);
  *smaller_leaf_best_split_default_left = static_cast<uint8_t>(host_leaf_best_split_info_buffer[2]);
  if (larger_leaf_index >= 0) {
    *larger_leaf_best_split_feature = host_leaf_best_split_info_buffer[3];
    *larger_leaf_best_split_threshold = static_cast<uint32_t>(host_leaf_best_split_info_buffer[4]);
    *larger_leaf_best_split_default_left = static_cast<uint8_t>(host_leaf_best_split_info_buffer[5]);
  }
  *best_leaf_index = host_leaf_best_split_info_buffer[6];
}

}  // namespace LightGBM

#endif  // USE_CUDA
