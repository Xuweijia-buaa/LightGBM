
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_best_split_finder.hpp"

namespace LightGBM {

#define K_MIN_SCORE (-1000000.0)

#define K_EPSILON (1e-15f)

#define CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n) \
  ((n) + ((n) >> LOG_NUM_BANKS_DATA_PARTITION_BEST_SPLIT_FINDER)) \

__device__ void PrefixSumHist(hist_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n - 1);
  const hist_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(dst_pos)] += elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 2) - 1);
      const unsigned int src_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 1) - 1);
      const hist_t src_val = elements[src_pos];
      elements[src_pos] = elements[dst_pos];
      elements[dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
  __syncthreads();
}

__device__ void PrefixSumHistCnt(data_size_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n - 1);
  const data_size_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(dst_pos)] += elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 2) - 1);
      const unsigned int src_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 1) - 1);
      const data_size_t src_val = elements[src_pos];
      elements[src_pos] = elements[dst_pos];
      elements[dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__device__ void ReduceBestGain(double* gain, hist_t* sum_gradients,
  hist_t* sum_hessians, /*data_size_t* num_data,*/ uint8_t* found,
  uint32_t* threshold_value) {
  const unsigned int tid = threadIdx.x;
  const unsigned int conflict_free_tid_plus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(tid + 1);
  for (unsigned int s = 1; s < MAX_NUM_BIN_IN_FEATURE; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < MAX_NUM_BIN_IN_FEATURE) {
      const uint32_t tid_s = tid + s;
      const uint32_t conflict_free_tid_s_plus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(tid_s + 1);
      if ((found[tid_s] && !found[tid]) || (found[tid_s] && found[tid] && gain[tid_s] > gain[tid])) {
        gain[tid] = gain[tid_s];
        sum_gradients[conflict_free_tid_plus_1] = sum_gradients[conflict_free_tid_s_plus_1];
        sum_hessians[conflict_free_tid_plus_1] = sum_hessians[conflict_free_tid_s_plus_1];
        found[tid] = found[tid_s];
        threshold_value[tid] = threshold_value[tid_s];
      }
    }
    __syncthreads();
  }
}

__device__ void ReduceBestGainForLeaves(double* gain, int* leaves, int cuda_cur_num_leaves) {
  const unsigned int tid = threadIdx.x;
  for (unsigned int s = 1; s < cuda_cur_num_leaves; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < cuda_cur_num_leaves) {
      const uint32_t tid_s = tid + s;
      if ((leaves[tid] == -1 && leaves[tid_s] != -1) || (leaves[tid] != -1 && leaves[tid_s] != -1 && gain[tid_s] > gain[tid])) {
        gain[tid] = gain[tid_s];
        leaves[tid] = leaves[tid_s];
      }
    }
    __syncthreads();
  }
}

__device__ double ThresholdL1(double s, double l1) {
  const double reg_s = fmax(0.0, fabs(s) - l1);
  if (s >= 0.0f) {
    return reg_s;
  } else {
    return -reg_s;
  }
}

__device__ double CalculateSplittedLeafOutput(double sum_gradients,
                                          double sum_hessians, double l1, const bool use_l1,
                                          double l2) {
  double ret;
  if (use_l1) {
    ret = -ThresholdL1(sum_gradients, l1) / (sum_hessians + l2);
  } else {
    ret = -sum_gradients / (sum_hessians + l2);
  }
  return ret;
}

__device__ double GetLeafGainGivenOutput(double sum_gradients,
                                      double sum_hessians, double l1, const bool use_l1,
                                      double l2, double output) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return -(2.0 * sg_l1 * output + (sum_hessians + l2) * output * output);
  } else {
    return -(2.0 * sum_gradients * output +
              (sum_hessians + l2) * output * output);
  }
}

__device__ double GetLeafGain(double sum_gradients, double sum_hessians,
                          double l1, const bool use_l1, double l2) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return (sg_l1 * sg_l1) / (sum_hessians + l2);
  } else {
    return (sum_gradients * sum_gradients) / (sum_hessians + l2);
  }
}

__device__ double GetSplitGains(double sum_left_gradients,
                            double sum_left_hessians,
                            double sum_right_gradients,
                            double sum_right_hessians,
                            double l1, const bool use_l1, double l2) {
  return GetLeafGain(sum_left_gradients,
                     sum_left_hessians,
                     l1, use_l1, l2) +
         GetLeafGain(sum_right_gradients,
                     sum_right_hessians,
                     l1, use_l1, l2);
}

__device__ void FindBestSplitsForLeafKernelInner(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const uint8_t feature_missing_type,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // input task information
  const bool reverse,
  const bool skip_default_bin,
  const bool na_as_missing,
  const uint8_t assume_out_default_left,
  // output parameters
  uint32_t* output_threshold,
  double* output_gain,
  uint8_t* output_default_left,
  double* output_left_sum_gradients,
  double* output_left_sum_hessians,
  data_size_t* output_left_num_data,
  double* output_left_gain,
  double* output_left_output,
  double* output_right_sum_gradients,
  double* output_right_sum_hessians,
  data_size_t* output_right_num_data,
  double* output_right_gain,
  double* output_right_output,
  uint8_t* output_found) {

  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;

  *output_found = 0;

  __shared__ hist_t local_grad_hist[MAX_NUM_BIN_IN_FEATURE + 1 + (MAX_NUM_BIN_IN_FEATURE + 1) / LOG_NUM_BANKS_DATA_PARTITION_BEST_SPLIT_FINDER];
  __shared__ hist_t local_hess_hist[MAX_NUM_BIN_IN_FEATURE + 1 + (MAX_NUM_BIN_IN_FEATURE + 1) / LOG_NUM_BANKS_DATA_PARTITION_BEST_SPLIT_FINDER];
  __shared__ double local_gain[MAX_NUM_BIN_IN_FEATURE];
  __shared__ uint8_t threshold_found[MAX_NUM_BIN_IN_FEATURE];
  __shared__ uint32_t threshold_value[MAX_NUM_BIN_IN_FEATURE];

  const unsigned int threadIdx_x = threadIdx.x;
  const bool skip_sum = (skip_default_bin && (threadIdx_x + feature_mfb_offset) == static_cast<int>(feature_default_bin));
  const uint32_t feature_num_bin_minus_offset = feature_num_bin - feature_mfb_offset;
  const bool skip_split = (skip_default_bin && (feature_num_bin_minus_offset - 1 - threadIdx_x + feature_mfb_offset == static_cast<int>(feature_default_bin)));
  const unsigned int bin_offset = threadIdx_x << 1;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(threadIdx_x);
  if (!reverse) {
    if (threadIdx_x < feature_num_bin_minus_offset && !skip_sum) {
      local_grad_hist[conflict_free_threadIdx_x] = feature_hist_ptr[bin_offset];
      const hist_t hess = feature_hist_ptr[bin_offset + 1];
      local_hess_hist[conflict_free_threadIdx_x] = hess;
    } else {
      local_grad_hist[conflict_free_threadIdx_x] = 0.0f;
      local_hess_hist[conflict_free_threadIdx_x] = 0.0f;
    }
  } else {
    if (threadIdx_x < feature_num_bin_minus_offset) {
      const unsigned int write_index = feature_num_bin_minus_offset - 1 - threadIdx_x;
      const unsigned int conflict_free_write_index = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(write_index);
      if (!skip_sum) {
        local_grad_hist[conflict_free_write_index] = feature_hist_ptr[bin_offset];
        const hist_t hess = feature_hist_ptr[bin_offset + 1];
        local_hess_hist[conflict_free_write_index] = hess;
      } else {
        local_grad_hist[conflict_free_write_index] = 0.0f;
        local_hess_hist[conflict_free_write_index] = 0.0f;
      }
    } else {
      local_grad_hist[conflict_free_threadIdx_x] = 0.0f;
      local_hess_hist[conflict_free_threadIdx_x] = 0.0f;
    }
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    local_hess_hist[conflict_free_threadIdx_x] += K_EPSILON;
  }
  local_gain[threadIdx_x] = K_MIN_SCORE;
  __syncthreads();
  PrefixSumHist(local_grad_hist, MAX_NUM_BIN_IN_FEATURE);
  PrefixSumHist(local_hess_hist, MAX_NUM_BIN_IN_FEATURE);
  __syncthreads();
  const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(threadIdx_x + 1);
  if (reverse) {
    if (threadIdx_x >= static_cast<unsigned int>(na_as_missing) && threadIdx_x <= feature_num_bin - 2 && !skip_split) {
      const double sum_right_gradient = local_grad_hist[conflict_free_threadIdx_x_plus_1];
      const double sum_right_hessian = local_hess_hist[conflict_free_threadIdx_x_plus_1];
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian;
      const data_size_t left_count = num_data - right_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain <= min_gain_shift) {
          threshold_found[threadIdx_x] = 0;
        } else {
          local_gain[threadIdx_x] = current_gain - min_gain_shift;
          threshold_value[threadIdx_x] = static_cast<uint32_t>(feature_num_bin - 2 - threadIdx_x);
          threshold_found[threadIdx_x] = 1;
        }
      } else {
        threshold_found[threadIdx_x] = 0;
      }
    } else {
      threshold_found[threadIdx_x] = 0;
    }
  } else {
    if (threadIdx_x <= feature_num_bin_minus_offset - 2 /* TODO(shiyu1994): skip default */) {
      const double sum_left_gradient = local_grad_hist[conflict_free_threadIdx_x_plus_1];
      const double sum_left_hessian = local_hess_hist[conflict_free_threadIdx_x_plus_1];
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain <= min_gain_shift) {
          threshold_found[threadIdx_x] = 0;
        } else {
          local_gain[threadIdx_x] = current_gain - min_gain_shift;
          threshold_value[threadIdx_x] = static_cast<uint32_t>(threadIdx_x + feature_mfb_offset);
          threshold_found[threadIdx_x] = 1;
        }
      } else {
        threshold_found[threadIdx_x] = 0;
      }
    } else {
      threshold_found[threadIdx_x] = 0;
    }
  }
  __syncthreads();
  ReduceBestGain(local_gain, local_grad_hist, local_hess_hist, threshold_found, threshold_value);
  const uint8_t found = threshold_found[0];
  if (found && threadIdx_x == 0) {
    *output_found = 1;
    *output_threshold = threshold_value[0];
    *output_gain = local_gain[0];
    *output_default_left = assume_out_default_left;
    if (reverse) {
      const double sum_right_gradient = local_grad_hist[1];
      const double sum_right_hessian = local_hess_hist[1] - K_EPSILON;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian - K_EPSILON;
      const data_size_t left_count = num_data - right_count;
      const double left_output = CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      *output_left_sum_gradients = sum_left_gradient;
      *output_left_sum_hessians = sum_left_hessian;
      *output_left_num_data = left_count;
      *output_right_sum_gradients = sum_right_gradient;
      *output_right_sum_hessians = sum_right_hessian;
      *output_right_num_data = right_count;
      *output_left_output = left_output;
      *output_left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      *output_right_output = right_output;
      *output_right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    } else {
      const double sum_left_gradient = local_grad_hist[1];
      const double sum_left_hessian = local_hess_hist[1] - K_EPSILON;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian - K_EPSILON;
      const data_size_t right_count = num_data - left_count;
      const double left_output = CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      *output_left_sum_gradients = sum_left_gradient;
      *output_left_sum_hessians = sum_left_hessian;
      *output_left_num_data = left_count;
      *output_right_sum_gradients = sum_right_gradient;
      *output_right_sum_hessians = sum_right_hessian;
      *output_right_num_data = right_count;
      *output_left_output = left_output;
      *output_left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      *output_right_output = right_output;
      *output_right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    }
  }
}

__global__ void FindBestSplitsForLeafKernel(
  // input feature information
  const uint32_t* feature_hist_offsets,
  const uint8_t* feature_mfb_offsets,
  const uint32_t* feature_default_bins, 
  const uint8_t* feature_missing_types,
  const uint32_t* feature_num_bins,
  // input task information
  const bool larger_only,
  const int num_tasks,
  const int* task_feature_index,
  const uint8_t* task_reverse,
  const uint8_t* task_skip_default_bin,
  const uint8_t* task_na_as_missing,
  const uint8_t* task_out_default_left,
  // input leaf information
  const int smaller_leaf_index,
  const double* smaller_leaf_gain,
  const double* smaller_sum_gradients_in_leaf,
  const double* smaller_sum_hessians_in_leaf,
  const data_size_t* smaller_num_data_in_leaf,
  hist_t** smaller_leaf_hist,
  const int larger_leaf_index,
  const double* larger_leaf_gain,
  const double* larger_sum_gradients_in_leaf,
  const double* larger_sum_hessians_in_leaf,
  const data_size_t* larger_num_data_in_leaf,
  hist_t** larger_leaf_hist,
  // input config parameter values
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double lambda_l1,
  const double lambda_l2,
  // output
  uint32_t* cuda_best_split_threshold,
  uint8_t* cuda_best_split_default_left,
  double* cuda_best_split_gain,
  double* cuda_best_split_left_sum_gradient,
  double* cuda_best_split_left_sum_hessian,
  data_size_t* cuda_best_split_left_count, 
  double* cuda_best_split_left_gain,
  double* cuda_best_split_left_output,
  double* cuda_best_split_right_sum_gradient,
  double* cuda_best_split_right_sum_hessian,
  data_size_t* cuda_best_split_right_count,
  double* cuda_best_split_right_gain,
  double* cuda_best_split_right_output,
  uint8_t* cuda_best_split_found) {

  const unsigned int task_index = blockIdx.x % num_tasks;
  const bool is_larger = static_cast<bool>(blockIdx.x >= num_tasks || larger_only);
  const int inner_feature_index = task_feature_index[task_index];
  const bool reverse = static_cast<bool>(task_reverse[task_index]);
  const bool skip_default_bin = static_cast<bool>(task_skip_default_bin[task_index]);
  const bool na_as_missing = static_cast<bool>(task_na_as_missing[task_index]);
  const bool assume_out_default_left = task_out_default_left[task_index];
  const double parent_gain = is_larger ? *larger_leaf_gain : *smaller_leaf_gain;
  const double sum_gradients = is_larger ? *larger_sum_gradients_in_leaf : *smaller_sum_gradients_in_leaf;
  const double sum_hessians = (is_larger ? *larger_sum_hessians_in_leaf : *smaller_sum_hessians_in_leaf) + 2 * K_EPSILON;
  const double num_data = is_larger ? *larger_num_data_in_leaf : *smaller_num_data_in_leaf;
  const unsigned int output_offset = is_larger ? (task_index + num_tasks) : task_index;
  uint8_t* out_default_left = cuda_best_split_default_left + output_offset;
  uint32_t* out_threshold = cuda_best_split_threshold + output_offset;
  double* out_left_sum_gradients = cuda_best_split_left_sum_gradient + output_offset;
  double* out_left_sum_hessians = cuda_best_split_left_sum_hessian + output_offset;
  double* out_right_sum_gradients = cuda_best_split_right_sum_gradient + output_offset;
  double* out_right_sum_hessians = cuda_best_split_right_sum_hessian + output_offset;
  data_size_t* out_left_num_data = cuda_best_split_left_count + output_offset;
  data_size_t* out_right_num_data = cuda_best_split_right_count + output_offset;
  double* out_left_output = cuda_best_split_left_output + output_offset;
  double* out_right_output = cuda_best_split_right_output + output_offset;
  double* out_left_gain = cuda_best_split_left_gain + output_offset;
  double* out_right_gain = cuda_best_split_right_gain + output_offset;
  uint8_t* out_found = cuda_best_split_found + output_offset;
  double* out_gain = cuda_best_split_gain + output_offset;
  const hist_t* hist_ptr = (is_larger ? *larger_leaf_hist : *smaller_leaf_hist) + feature_hist_offsets[inner_feature_index] * 2;
  FindBestSplitsForLeafKernelInner(
    // input feature information
    hist_ptr,
    feature_num_bins[inner_feature_index],
    feature_mfb_offsets[inner_feature_index],
    feature_default_bins[inner_feature_index],
    feature_missing_types[inner_feature_index],
    inner_feature_index,
    // input config parameter values
    lambda_l1,
    lambda_l2,
    min_data_in_leaf,
    min_sum_hessian_in_leaf,
    min_gain_to_split,
    // input parent node information
    parent_gain,
    sum_gradients,
    sum_hessians,
    num_data,
    // input task information
    reverse,
    skip_default_bin,
    na_as_missing,
    assume_out_default_left,
    // output parameters
    out_threshold,
    out_gain,
    out_default_left,
    out_left_sum_gradients,
    out_left_sum_hessians,
    out_left_num_data,
    out_left_gain,
    out_left_output,
    out_right_sum_gradients,
    out_right_sum_hessians,
    out_right_num_data,
    out_right_gain,
    out_right_output,
    out_found);
}

void CUDABestSplitFinder::LaunchFindBestSplitsForLeafKernel(
  const CUDALeafSplits* smaller_leaf_splits,
  const CUDALeafSplits* larger_leaf_splits,
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {
  if (!is_smaller_leaf_valid && !is_larger_leaf_valid) {
    return;
  }
  bool larger_only = false;
  if (!is_smaller_leaf_valid) {
    larger_only = true;
  }
  if (!larger_only) {
    FindBestSplitsForLeafKernel<<<num_tasks_, MAX_NUM_BIN_IN_FEATURE, 0, cuda_streams_[0]>>>(
      // input feature information
      cuda_feature_hist_offsets_,
      cuda_feature_mfb_offsets_,
      cuda_feature_default_bins_,
      cuda_feature_missing_type_,
      cuda_feature_num_bins_,
      // input task information
      larger_only,
      num_tasks_,
      cuda_task_feature_index_,
      cuda_task_reverse_,
      cuda_task_skip_default_bin_,
      cuda_task_na_as_missing_,
      cuda_task_out_default_left_,
      // input leaf information
      smaller_leaf_index,
      smaller_leaf_splits->cuda_gain(),
      smaller_leaf_splits->cuda_sum_of_gradients(),
      smaller_leaf_splits->cuda_sum_of_hessians(),
      smaller_leaf_splits->cuda_num_data_in_leaf(),
      smaller_leaf_splits->cuda_hist_in_leaf_pointer_pointer(),
      larger_leaf_index,
      larger_leaf_splits->cuda_gain(),
      larger_leaf_splits->cuda_sum_of_gradients(),
      larger_leaf_splits->cuda_sum_of_hessians(),
      larger_leaf_splits->cuda_num_data_in_leaf(),
      larger_leaf_splits->cuda_hist_in_leaf_pointer_pointer(),
      // configuration parameter values
      min_data_in_leaf_,
      min_sum_hessian_in_leaf_,
      min_gain_to_split_,
      lambda_l1_,
      lambda_l2_,
      // output parameters
      cuda_best_split_threshold_,
      cuda_best_split_default_left_,
      cuda_best_split_gain_,
      cuda_best_split_left_sum_gradient_,
      cuda_best_split_left_sum_hessian_,
      cuda_best_split_left_count_,
      cuda_best_split_left_gain_,
      cuda_best_split_left_output_,
      cuda_best_split_right_sum_gradient_,
      cuda_best_split_right_sum_hessian_,
      cuda_best_split_right_count_,
      cuda_best_split_right_gain_,
      cuda_best_split_right_output_,
      cuda_best_split_found_);
  }
  SynchronizeCUDADevice();
  if (larger_leaf_index >= 0) {
    FindBestSplitsForLeafKernel<<<num_tasks_, MAX_NUM_BIN_IN_FEATURE, 0, cuda_streams_[1]>>>(
      // input feature information
      cuda_feature_hist_offsets_,
      cuda_feature_mfb_offsets_,
      cuda_feature_default_bins_,
      cuda_feature_missing_type_,
      cuda_feature_num_bins_,
      // input task information
      true,
      num_tasks_,
      cuda_task_feature_index_,
      cuda_task_reverse_,
      cuda_task_skip_default_bin_,
      cuda_task_na_as_missing_,
      cuda_task_out_default_left_,
      // input leaf information
      smaller_leaf_index,
      smaller_leaf_splits->cuda_gain(),
      smaller_leaf_splits->cuda_sum_of_gradients(),
      smaller_leaf_splits->cuda_sum_of_hessians(),
      smaller_leaf_splits->cuda_num_data_in_leaf(),
      smaller_leaf_splits->cuda_hist_in_leaf_pointer_pointer(),
      larger_leaf_index,
      larger_leaf_splits->cuda_gain(),
      larger_leaf_splits->cuda_sum_of_gradients(),
      larger_leaf_splits->cuda_sum_of_hessians(),
      larger_leaf_splits->cuda_num_data_in_leaf(),
      larger_leaf_splits->cuda_hist_in_leaf_pointer_pointer(),
      // configuration parameter values
      min_data_in_leaf_,
      min_sum_hessian_in_leaf_,
      min_gain_to_split_,
      lambda_l1_,
      lambda_l2_,
      // output parameters
      cuda_best_split_threshold_,
      cuda_best_split_default_left_,
      cuda_best_split_gain_,
      cuda_best_split_left_sum_gradient_,
      cuda_best_split_left_sum_hessian_,
      cuda_best_split_left_count_,
      cuda_best_split_left_gain_,
      cuda_best_split_left_output_,
      cuda_best_split_right_sum_gradient_,
      cuda_best_split_right_sum_hessian_,
      cuda_best_split_right_count_,
      cuda_best_split_right_gain_,
      cuda_best_split_right_output_,
      cuda_best_split_found_);
  }
}

__device__ void ReduceBestSplit(uint8_t* found, double* gain, uint32_t* shared_read_index,
  uint32_t num_features_aligned) {
  const uint32_t threadIdx_x = threadIdx.x;
  for (unsigned int s = 1; s < num_features_aligned; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < num_features_aligned) {
      const uint32_t pos_to_compare = threadIdx_x + s;
      if ((!found[threadIdx_x] && found[pos_to_compare]) ||
        (found[threadIdx_x] && found[pos_to_compare] && gain[threadIdx_x] < gain[pos_to_compare])) {
        found[threadIdx_x] = found[pos_to_compare];
        gain[threadIdx_x] = gain[pos_to_compare];
        shared_read_index[threadIdx_x] = shared_read_index[pos_to_compare];
      }
    }
    __syncthreads();
  } 
}

__global__ void SyncBestSplitForLeafKernel(const int smaller_leaf_index, const int larger_leaf_index,
  const int* cuda_num_features, int* cuda_leaf_best_split_feature, uint8_t* cuda_leaf_best_split_default_left,
  uint32_t* cuda_leaf_best_split_threshold, double* cuda_leaf_best_split_gain,
  double* cuda_leaf_best_split_left_sum_gradient, double* cuda_leaf_best_split_left_sum_hessian,
  data_size_t* cuda_leaf_best_split_left_count, double* cuda_leaf_best_split_left_gain,
  double* cuda_leaf_best_split_left_output,
  double* cuda_leaf_best_split_right_sum_gradient, double* cuda_leaf_best_split_right_sum_hessian,
  data_size_t* cuda_leaf_best_split_right_count, double* cuda_leaf_best_split_right_gain,
  double* cuda_leaf_best_split_right_output,
  uint8_t* cuda_leaf_best_split_found,
  // input parameters
  const int* cuda_task_feature_index,
  const uint8_t* cuda_best_split_default_left,
  const uint32_t* cuda_best_split_threshold,
  const double* cuda_best_split_gain,
  const double* cuda_best_split_left_sum_gradient,
  const double* cuda_best_split_left_sum_hessian,
  const data_size_t* cuda_best_split_left_count,
  const double* cuda_best_split_left_gain,
  const double* cuda_best_split_left_output,
  const double* cuda_best_split_right_sum_gradient,
  const double* cuda_best_split_right_sum_hessian,
  const data_size_t* cuda_best_split_right_count,
  const double* cuda_best_split_right_gain,
  const double* cuda_best_split_right_output,
  const uint8_t* cuda_best_split_found,
  const uint32_t* cuda_feature_default_bins,
  const int num_tasks,
  const int num_tasks_aligned,
  const int num_blocks_per_leaf,
  const bool larger_only,
  const int num_leaves) {

  const uint32_t threadIdx_x = threadIdx.x;
  const uint32_t blockIdx_x = blockIdx.x;

  __shared__ uint8_t best_found[NUM_TASKS_PER_SYNC_BLOCK];
  __shared__ double best_gain[NUM_TASKS_PER_SYNC_BLOCK];
  __shared__ uint32_t shared_read_index[NUM_TASKS_PER_SYNC_BLOCK];

  const bool is_smaller = (blockIdx_x < static_cast<unsigned int>(num_blocks_per_leaf) && !larger_only);
  const uint32_t leaf_block_index = (is_smaller || larger_only) ? blockIdx_x : (blockIdx_x - static_cast<unsigned int>(num_blocks_per_leaf));
  const int task_index = static_cast<int>(leaf_block_index * blockDim.x + threadIdx_x);
  const uint32_t read_index = is_smaller ? static_cast<uint32_t>(task_index) : static_cast<uint32_t>(task_index + num_tasks);
  if (task_index < num_tasks) {
    best_found[threadIdx_x] = cuda_best_split_found[read_index];
    best_gain[threadIdx_x] = cuda_best_split_gain[read_index];
    shared_read_index[threadIdx_x] = read_index;
  } else {
    best_found[threadIdx_x] = 0;
  }

  __syncthreads();
  ReduceBestSplit(best_found, best_gain, shared_read_index, NUM_TASKS_PER_SYNC_BLOCK);
  if (threadIdx.x == 0) {
    const int leaf_index_ref = is_smaller ? smaller_leaf_index : larger_leaf_index;
    const unsigned buffer_write_pos = static_cast<unsigned int>(leaf_index_ref) + leaf_block_index * num_leaves;
    const uint32_t best_read_index = shared_read_index[0];
    if (best_found[0]) {
      cuda_leaf_best_split_gain[buffer_write_pos] = best_gain[0];
      cuda_leaf_best_split_feature[buffer_write_pos] = is_smaller ? cuda_task_feature_index[best_read_index] :
        cuda_task_feature_index[static_cast<int>(best_read_index) - num_tasks];
      cuda_leaf_best_split_default_left[buffer_write_pos] = cuda_best_split_default_left[best_read_index];
      cuda_leaf_best_split_threshold[buffer_write_pos] = cuda_best_split_threshold[best_read_index];
      cuda_leaf_best_split_left_sum_gradient[buffer_write_pos] = cuda_best_split_left_sum_gradient[best_read_index];
      cuda_leaf_best_split_left_sum_hessian[buffer_write_pos] = cuda_best_split_left_sum_hessian[best_read_index];
      cuda_leaf_best_split_left_count[buffer_write_pos] = cuda_best_split_left_count[best_read_index];
      cuda_leaf_best_split_left_gain[buffer_write_pos] = cuda_best_split_left_gain[best_read_index]; 
      cuda_leaf_best_split_left_output[buffer_write_pos] = cuda_best_split_left_output[best_read_index];
      cuda_leaf_best_split_right_sum_gradient[buffer_write_pos] = cuda_best_split_right_sum_gradient[best_read_index];
      cuda_leaf_best_split_right_sum_hessian[buffer_write_pos] = cuda_best_split_right_sum_hessian[best_read_index];
      cuda_leaf_best_split_right_count[buffer_write_pos] = cuda_best_split_right_count[best_read_index];
      cuda_leaf_best_split_right_gain[buffer_write_pos] = cuda_best_split_right_gain[best_read_index]; 
      cuda_leaf_best_split_right_output[buffer_write_pos] = cuda_best_split_right_output[best_read_index];
      cuda_leaf_best_split_found[buffer_write_pos] = 1;
    } else {
      cuda_leaf_best_split_gain[buffer_write_pos] = K_MIN_SCORE;
      cuda_leaf_best_split_found[buffer_write_pos] = 0;
    }
  }
}

__global__ void SyncBestSplitForLeafKernelAllBlocks(
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const unsigned int num_blocks_per_leaf,
  const int num_leaves,
  int* cuda_leaf_best_split_feature, uint8_t* cuda_leaf_best_split_default_left,
  uint32_t* cuda_leaf_best_split_threshold, double* cuda_leaf_best_split_gain,
  double* cuda_leaf_best_split_left_sum_gradient, double* cuda_leaf_best_split_left_sum_hessian,
  data_size_t* cuda_leaf_best_split_left_count, double* cuda_leaf_best_split_left_gain,
  double* cuda_leaf_best_split_left_output,
  double* cuda_leaf_best_split_right_sum_gradient, double* cuda_leaf_best_split_right_sum_hessian,
  data_size_t* cuda_leaf_best_split_right_count, double* cuda_leaf_best_split_right_gain,
  double* cuda_leaf_best_split_right_output,
  uint8_t* cuda_leaf_best_split_found,
  const bool larger_only) {
  if (!larger_only) {
    if (blockIdx.x == 0) {
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(smaller_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        if ((cuda_leaf_best_split_found[leaf_read_pos] == 1 && cuda_leaf_best_split_found[smaller_leaf_index] == 1 &&
          cuda_leaf_best_split_gain[leaf_read_pos] > cuda_leaf_best_split_gain[smaller_leaf_index]) ||
            (cuda_leaf_best_split_found[smaller_leaf_index] == 0 && cuda_leaf_best_split_found[leaf_read_pos] == 1)) {
            cuda_leaf_best_split_found[smaller_leaf_index] = cuda_leaf_best_split_found[leaf_read_pos];
            cuda_leaf_best_split_feature[smaller_leaf_index] = cuda_leaf_best_split_feature[leaf_read_pos];
            cuda_leaf_best_split_default_left[smaller_leaf_index] = cuda_leaf_best_split_default_left[leaf_read_pos];
            cuda_leaf_best_split_threshold[smaller_leaf_index] = cuda_leaf_best_split_threshold[leaf_read_pos];
            cuda_leaf_best_split_gain[smaller_leaf_index] = cuda_leaf_best_split_gain[leaf_read_pos];
            cuda_leaf_best_split_left_sum_gradient[smaller_leaf_index] = cuda_leaf_best_split_left_sum_gradient[leaf_read_pos];
            cuda_leaf_best_split_left_sum_hessian[smaller_leaf_index] = cuda_leaf_best_split_left_sum_hessian[leaf_read_pos];
            cuda_leaf_best_split_left_count[smaller_leaf_index] = cuda_leaf_best_split_left_count[leaf_read_pos];
            cuda_leaf_best_split_left_gain[smaller_leaf_index] = cuda_leaf_best_split_left_gain[leaf_read_pos];
            cuda_leaf_best_split_left_output[smaller_leaf_index] = cuda_leaf_best_split_left_output[leaf_read_pos];
            cuda_leaf_best_split_right_sum_gradient[smaller_leaf_index] = cuda_leaf_best_split_right_sum_gradient[leaf_read_pos];
            cuda_leaf_best_split_right_sum_hessian[smaller_leaf_index] = cuda_leaf_best_split_right_sum_hessian[leaf_read_pos];
            cuda_leaf_best_split_right_count[smaller_leaf_index] = cuda_leaf_best_split_right_count[leaf_read_pos];
            cuda_leaf_best_split_right_gain[smaller_leaf_index] = cuda_leaf_best_split_right_gain[leaf_read_pos];
            cuda_leaf_best_split_right_output[smaller_leaf_index] = cuda_leaf_best_split_right_output[leaf_read_pos];
        }
      }
    }
  }
  if (larger_leaf_index >= 0) {
    if (blockIdx.x == 1 || larger_only) {
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(larger_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        if ((cuda_leaf_best_split_found[leaf_read_pos] == 1 && cuda_leaf_best_split_found[larger_leaf_index] == 1 &&
          cuda_leaf_best_split_gain[leaf_read_pos] > cuda_leaf_best_split_gain[larger_leaf_index]) ||
            (cuda_leaf_best_split_found[larger_leaf_index] == 0 && cuda_leaf_best_split_found[leaf_read_pos] == 1)) {
            cuda_leaf_best_split_found[larger_leaf_index] = cuda_leaf_best_split_found[leaf_read_pos];
            cuda_leaf_best_split_feature[larger_leaf_index] = cuda_leaf_best_split_feature[leaf_read_pos];
            cuda_leaf_best_split_default_left[larger_leaf_index] = cuda_leaf_best_split_default_left[leaf_read_pos];
            cuda_leaf_best_split_threshold[larger_leaf_index] = cuda_leaf_best_split_threshold[leaf_read_pos];
            cuda_leaf_best_split_gain[larger_leaf_index] = cuda_leaf_best_split_gain[leaf_read_pos];
            cuda_leaf_best_split_left_sum_gradient[larger_leaf_index] = cuda_leaf_best_split_left_sum_gradient[leaf_read_pos];
            cuda_leaf_best_split_left_sum_hessian[larger_leaf_index] = cuda_leaf_best_split_left_sum_hessian[leaf_read_pos];
            cuda_leaf_best_split_left_count[larger_leaf_index] = cuda_leaf_best_split_left_count[leaf_read_pos];
            cuda_leaf_best_split_left_gain[larger_leaf_index] = cuda_leaf_best_split_left_gain[leaf_read_pos];
            cuda_leaf_best_split_left_output[larger_leaf_index] = cuda_leaf_best_split_left_output[leaf_read_pos];
            cuda_leaf_best_split_right_sum_gradient[larger_leaf_index] = cuda_leaf_best_split_right_sum_gradient[leaf_read_pos];
            cuda_leaf_best_split_right_sum_hessian[larger_leaf_index] = cuda_leaf_best_split_right_sum_hessian[leaf_read_pos];
            cuda_leaf_best_split_right_count[larger_leaf_index] = cuda_leaf_best_split_right_count[leaf_read_pos];
            cuda_leaf_best_split_right_gain[larger_leaf_index] = cuda_leaf_best_split_right_gain[leaf_read_pos];
            cuda_leaf_best_split_right_output[larger_leaf_index] = cuda_leaf_best_split_right_output[leaf_read_pos];
        }
      }
    }
  }
}

void CUDABestSplitFinder::LaunchSyncBestSplitForLeafKernel(
  const int cpu_smaller_leaf_index,
  const int cpu_larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {

  int num_tasks = num_tasks_;
  int num_tasks_aligned = 1;
  num_tasks -= 1;
  while (num_tasks > 0) {
    num_tasks_aligned <<= 1;
    num_tasks >>= 1;
  }
  const int num_blocks_per_leaf = (num_tasks_ + NUM_TASKS_PER_SYNC_BLOCK - 1) / NUM_TASKS_PER_SYNC_BLOCK;
  if (cpu_larger_leaf_index >= 0 && is_smaller_leaf_valid && is_larger_leaf_valid) {
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[0]>>>(
      cpu_smaller_leaf_index,
      cpu_larger_leaf_index,
      cuda_num_features_,
      cuda_leaf_best_split_feature_,
      cuda_leaf_best_split_default_left_,
      cuda_leaf_best_split_threshold_,
      cuda_leaf_best_split_gain_,
      cuda_leaf_best_split_left_sum_gradient_,
      cuda_leaf_best_split_left_sum_hessian_,
      cuda_leaf_best_split_left_count_,
      cuda_leaf_best_split_left_gain_,
      cuda_leaf_best_split_left_output_,
      cuda_leaf_best_split_right_sum_gradient_,
      cuda_leaf_best_split_right_sum_hessian_,
      cuda_leaf_best_split_right_count_,
      cuda_leaf_best_split_right_gain_,
      cuda_leaf_best_split_right_output_,
      cuda_leaf_best_split_found_,
      cuda_task_feature_index_,
      cuda_best_split_default_left_,
      cuda_best_split_threshold_,
      cuda_best_split_gain_,
      cuda_best_split_left_sum_gradient_,
      cuda_best_split_left_sum_hessian_,
      cuda_best_split_left_count_,
      cuda_best_split_left_gain_,
      cuda_best_split_left_output_,
      cuda_best_split_right_sum_gradient_,
      cuda_best_split_right_sum_hessian_,
      cuda_best_split_right_count_,
      cuda_best_split_right_gain_,
      cuda_best_split_right_output_,
      cuda_best_split_found_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      false,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[0]>>>(
        cpu_smaller_leaf_index,
        cpu_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_feature_,
        cuda_leaf_best_split_default_left_,
        cuda_leaf_best_split_threshold_,
        cuda_leaf_best_split_gain_,
        cuda_leaf_best_split_left_sum_gradient_,
        cuda_leaf_best_split_left_sum_hessian_,
        cuda_leaf_best_split_left_count_,
        cuda_leaf_best_split_left_gain_,
        cuda_leaf_best_split_left_output_,
        cuda_leaf_best_split_right_sum_gradient_,
        cuda_leaf_best_split_right_sum_hessian_,
        cuda_leaf_best_split_right_count_,
        cuda_leaf_best_split_right_gain_,
        cuda_leaf_best_split_right_output_,
        cuda_leaf_best_split_found_,
        false);
    }
    SynchronizeCUDADevice();
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[1]>>>(
      cpu_smaller_leaf_index,
      cpu_larger_leaf_index,
      cuda_num_features_,
      cuda_leaf_best_split_feature_,
      cuda_leaf_best_split_default_left_,
      cuda_leaf_best_split_threshold_,
      cuda_leaf_best_split_gain_,
      cuda_leaf_best_split_left_sum_gradient_,
      cuda_leaf_best_split_left_sum_hessian_,
      cuda_leaf_best_split_left_count_,
      cuda_leaf_best_split_left_gain_,
      cuda_leaf_best_split_left_output_,
      cuda_leaf_best_split_right_sum_gradient_,
      cuda_leaf_best_split_right_sum_hessian_,
      cuda_leaf_best_split_right_count_,
      cuda_leaf_best_split_right_gain_,
      cuda_leaf_best_split_right_output_,
      cuda_leaf_best_split_found_,
      cuda_task_feature_index_,
      cuda_best_split_default_left_,
      cuda_best_split_threshold_,
      cuda_best_split_gain_,
      cuda_best_split_left_sum_gradient_,
      cuda_best_split_left_sum_hessian_,
      cuda_best_split_left_count_,
      cuda_best_split_left_gain_,
      cuda_best_split_left_output_,
      cuda_best_split_right_sum_gradient_,
      cuda_best_split_right_sum_hessian_,
      cuda_best_split_right_count_,
      cuda_best_split_right_gain_,
      cuda_best_split_right_output_,
      cuda_best_split_found_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      true,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[1]>>>(
        cpu_smaller_leaf_index,
        cpu_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_feature_,
        cuda_leaf_best_split_default_left_,
        cuda_leaf_best_split_threshold_,
        cuda_leaf_best_split_gain_,
        cuda_leaf_best_split_left_sum_gradient_,
        cuda_leaf_best_split_left_sum_hessian_,
        cuda_leaf_best_split_left_count_,
        cuda_leaf_best_split_left_gain_,
        cuda_leaf_best_split_left_output_,
        cuda_leaf_best_split_right_sum_gradient_,
        cuda_leaf_best_split_right_sum_hessian_,
        cuda_leaf_best_split_right_count_,
        cuda_leaf_best_split_right_gain_,
        cuda_leaf_best_split_right_output_,
        cuda_leaf_best_split_found_,
        true);
    }
  } else {
    const bool larger_only = (!is_smaller_leaf_valid && is_larger_leaf_valid);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK>>>(
      cpu_smaller_leaf_index,
      cpu_larger_leaf_index,
      cuda_num_features_,
      cuda_leaf_best_split_feature_,
      cuda_leaf_best_split_default_left_,
      cuda_leaf_best_split_threshold_,
      cuda_leaf_best_split_gain_,
      cuda_leaf_best_split_left_sum_gradient_,
      cuda_leaf_best_split_left_sum_hessian_,
      cuda_leaf_best_split_left_count_,
      cuda_leaf_best_split_left_gain_,
      cuda_leaf_best_split_left_output_,
      cuda_leaf_best_split_right_sum_gradient_,
      cuda_leaf_best_split_right_sum_hessian_,
      cuda_leaf_best_split_right_count_,
      cuda_leaf_best_split_right_gain_,
      cuda_leaf_best_split_right_output_,
      cuda_leaf_best_split_found_,
      cuda_task_feature_index_,
      cuda_best_split_default_left_,
      cuda_best_split_threshold_,
      cuda_best_split_gain_,
      cuda_best_split_left_sum_gradient_,
      cuda_best_split_left_sum_hessian_,
      cuda_best_split_left_count_,
      cuda_best_split_left_gain_,
      cuda_best_split_left_output_,
      cuda_best_split_right_sum_gradient_,
      cuda_best_split_right_sum_hessian_,
      cuda_best_split_right_count_,
      cuda_best_split_right_gain_,
      cuda_best_split_right_output_,
      cuda_best_split_found_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      larger_only,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SynchronizeCUDADevice();
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1>>>(
        cpu_smaller_leaf_index,
        cpu_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_feature_,
        cuda_leaf_best_split_default_left_,
        cuda_leaf_best_split_threshold_,
        cuda_leaf_best_split_gain_,
        cuda_leaf_best_split_left_sum_gradient_,
        cuda_leaf_best_split_left_sum_hessian_,
        cuda_leaf_best_split_left_count_,
        cuda_leaf_best_split_left_gain_,
        cuda_leaf_best_split_left_output_,
        cuda_leaf_best_split_right_sum_gradient_,
        cuda_leaf_best_split_right_sum_hessian_,
        cuda_leaf_best_split_right_count_,
        cuda_leaf_best_split_right_gain_,
        cuda_leaf_best_split_right_output_,
        cuda_leaf_best_split_found_,
        larger_only);
    }
  }
}

__global__ void FindBestFromAllSplitsKernel(const int* cuda_cur_num_leaves,
  const double* cuda_leaf_best_split_gain, int* out_best_leaf,
  const int* cuda_leaf_best_split_feature, const uint32_t* cuda_leaf_best_split_threshold,
  const uint32_t* cuda_feature_default_bins,
  const double* cuda_leaf_best_split_left_sum_gradient,
  const double* cuda_leaf_best_split_left_sum_hessian,
  const double* cuda_leaf_best_split_right_sum_gradient,
  const double* cuda_leaf_best_split_right_sum_hessian,
  const data_size_t* cuda_leaf_best_split_left_count,
  const data_size_t* cuda_leaf_best_split_right_count,
  const uint8_t* cuda_leaf_best_split_found,
  int* cuda_best_split_info_buffer) {
  const int cuda_cur_num_leaves_ref = *cuda_cur_num_leaves;
  __shared__ double thread_best_gain[NUM_THREADS_FIND_BEST_LEAF];
  __shared__ int thread_best_leaf[NUM_THREADS_FIND_BEST_LEAF];
  const unsigned int threadIdx_x = threadIdx.x;
  thread_best_gain[threadIdx_x] = K_MIN_SCORE;
  thread_best_leaf[threadIdx_x] = -1;
  const int num_leaves_per_thread = (cuda_cur_num_leaves_ref + NUM_THREADS_FIND_BEST_LEAF - 1) / NUM_THREADS_FIND_BEST_LEAF;
  const int cur_num_valid_threads = (cuda_cur_num_leaves_ref + num_leaves_per_thread - 1) / num_leaves_per_thread;
  if (threadIdx_x < static_cast<unsigned int>(cur_num_valid_threads)) {
    const int start = num_leaves_per_thread * threadIdx_x;
    const int end = min(start + num_leaves_per_thread, cuda_cur_num_leaves_ref);
    for (int leaf_index = threadIdx_x; leaf_index < cuda_cur_num_leaves_ref; leaf_index += cur_num_valid_threads) {
      const double leaf_best_gain = cuda_leaf_best_split_gain[leaf_index];
      if (cuda_leaf_best_split_found[leaf_index] && leaf_best_gain > thread_best_gain[threadIdx_x]) {
        thread_best_gain[threadIdx_x] = leaf_best_gain;
        thread_best_leaf[threadIdx_x] = leaf_index;
      }
    }
  }
  __syncthreads();
  ReduceBestGainForLeaves(thread_best_gain, thread_best_leaf, cur_num_valid_threads);
  if (threadIdx_x == 0) {
    *out_best_leaf = thread_best_leaf[0];
    cuda_best_split_info_buffer[6] = thread_best_leaf[0];
  }
}

__global__ void PrepareLeafBestSplitInfo(const int smaller_leaf_index, const int larger_leaf_index,
  int* cuda_best_split_info_buffer, const int* cuda_leaf_best_split_feature,
  const uint32_t* cuda_leaf_best_split_threshold, const uint8_t* cuda_leaf_best_split_default_left) {
  const unsigned int threadIdx_x = blockIdx.x;
  if (threadIdx_x == 0) {
    cuda_best_split_info_buffer[0] = cuda_leaf_best_split_feature[smaller_leaf_index];
  } else if (threadIdx_x == 1) {
    cuda_best_split_info_buffer[1] = cuda_leaf_best_split_threshold[smaller_leaf_index];
  } else if (threadIdx_x == 2) {
    cuda_best_split_info_buffer[2] = cuda_leaf_best_split_default_left[smaller_leaf_index];
  }
  if (larger_leaf_index >= 0) { 
    if (threadIdx_x == 3) {
      cuda_best_split_info_buffer[3] = cuda_leaf_best_split_feature[larger_leaf_index];
    } else if (threadIdx_x == 4) {
      cuda_best_split_info_buffer[4] = cuda_leaf_best_split_threshold[larger_leaf_index];
    } else if (threadIdx_x == 5) {
      cuda_best_split_info_buffer[5] = cuda_leaf_best_split_default_left[larger_leaf_index];
    }
  }
}

void CUDABestSplitFinder::LaunchFindBestFromAllSplitsKernel(const int* cuda_cur_num_leaves,
  const int smaller_leaf_index, const int larger_leaf_index, std::vector<int>* leaf_best_split_feature,
  std::vector<uint32_t>* leaf_best_split_threshold, std::vector<uint8_t>* leaf_best_split_default_left, int* best_leaf_index) {
  FindBestFromAllSplitsKernel<<<1, NUM_THREADS_FIND_BEST_LEAF, 0, cuda_streams_[1]>>>(cuda_cur_num_leaves, cuda_leaf_best_split_gain_, cuda_best_leaf_,
    cuda_leaf_best_split_feature_, cuda_leaf_best_split_threshold_, cuda_feature_default_bins_,
    cuda_leaf_best_split_left_sum_gradient_,
    cuda_leaf_best_split_left_sum_hessian_,
    cuda_leaf_best_split_right_sum_gradient_,
    cuda_leaf_best_split_right_sum_hessian_,
    cuda_leaf_best_split_left_count_,
    cuda_leaf_best_split_right_count_,
    cuda_leaf_best_split_found_,
    cuda_best_split_info_buffer_);
  PrepareLeafBestSplitInfo<<<6, 1, 0, cuda_streams_[0]>>>(smaller_leaf_index, larger_leaf_index,
    cuda_best_split_info_buffer_, cuda_leaf_best_split_feature_,
    cuda_leaf_best_split_threshold_, cuda_leaf_best_split_default_left_);
  std::vector<int> cpu_leaf_best_split_info_buffer(7);
  SynchronizeCUDADevice();
  CopyFromCUDADeviceToHost<int>(cpu_leaf_best_split_info_buffer.data(), cuda_best_split_info_buffer_, 7);
  (*leaf_best_split_feature)[smaller_leaf_index] = cpu_leaf_best_split_info_buffer[0];
  (*leaf_best_split_threshold)[smaller_leaf_index] = static_cast<uint32_t>(cpu_leaf_best_split_info_buffer[1]);
  (*leaf_best_split_default_left)[smaller_leaf_index] = static_cast<uint8_t>(cpu_leaf_best_split_info_buffer[2]);
  if (larger_leaf_index >= 0) {
    (*leaf_best_split_feature)[larger_leaf_index] = cpu_leaf_best_split_info_buffer[3];
    (*leaf_best_split_threshold)[larger_leaf_index] = static_cast<uint32_t>(cpu_leaf_best_split_info_buffer[4]);
    (*leaf_best_split_default_left)[larger_leaf_index] = static_cast<uint8_t>(cpu_leaf_best_split_info_buffer[5]);
  }
  *best_leaf_index = cpu_leaf_best_split_info_buffer[6];
  /*if (smaller_leaf_index == 0) {
    Log::Warning("smaller_leaf_index = %d, best_split_feature = %d, best_split_threshold = %d",
      smaller_leaf_index, cpu_leaf_best_split_info_buffer[0], cpu_leaf_best_split_info_buffer[1]);
  }*/
}

}  // namespace LightGBM

#endif  // USE_CUDA
