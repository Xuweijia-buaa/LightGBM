#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#include "cuda_exp_tree_learner.hpp"

namespace LightGBM {

__global__ void ReduceRootNodeInformationKernel(
  const CUDALeafSplitsStruct* leaf_splits_buffer,
  const int num_gpu,
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t num_data,
  CUDALeafSplitsStruct* out,
  double* out_sum_hessians) {
  double sum_of_gradients = 0.0;
  double sum_of_hessians = 0.0f;
  int64_t sum_of_gradients_hessians = 0;
  data_size_t num_data_in_leaf = 0;
  for (int gpu_index = 0; gpu_index < num_gpu; ++gpu_index) {
    const CUDALeafSplitsStruct* leaf_splits = leaf_splits_buffer + gpu_index;
    const double gpu_sum_of_gradients = leaf_splits->sum_of_gradients;
    const double gpu_sum_of_hessians = leaf_splits->sum_of_hessians;
    const int64_t gpu_sum_of_gradients_hessians = leaf_splits->sum_of_gradients_hessians;
    const data_size_t gpu_num_data_in_leaf = leaf_splits->num_data_in_leaf;
    sum_of_gradients += gpu_sum_of_gradients;
    sum_of_hessians += gpu_sum_of_hessians;
    sum_of_gradients_hessians += gpu_sum_of_gradients_hessians;
    num_data_in_leaf += gpu_num_data_in_leaf;
  }
  out->sum_of_gradients = sum_of_gradients;
  out->sum_of_hessians = sum_of_hessians;
  *out_sum_hessians = sum_of_hessians;
  out->sum_of_gradients_hessians = sum_of_gradients_hessians;
  out->num_data_in_leaf = num_data_in_leaf;
  assert(num_data_in_leaf == num_data);
  out->leaf_index = 0;
  const bool use_l1 = lambda_l1 > 0.0f;
  if (!use_l1) {
    // no smoothing on root node
    out->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
  } else {
    // no smoothing on root node
    out->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
  }
  if (!use_l1) {
    // no smoothing on root node
    out->leaf_value =
      CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
  } else {
    // no smoothing on root node
    out->leaf_value =
      CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
  }
}

void CUDAExpTreeLearner::LaunchReduceRootNodeInformationKernel(CUDALeafSplitsStruct* out) {
  ReduceRootNodeInformationKernel<<<1, 1>>>(
    leaf_splits_buffer_[0].RawData(),
    config_->num_gpu,
    config_->lambda_l1,
    config_->lambda_l2,
    // TODO(shiyu1994): bagging is not supported by now
    train_data_->num_data(),
    out,
    cuda_root_sum_hessians_.RawData());
}

__global__ void ReduceBestSplitsForLeafKernel(
  int* leaf_best_split_info_buffer,
  const int num_gpu) {
  int best_gpu_for_smaller_leaf = -1;
  double best_gain = kMinScore;
  for (int gpu_index = 0; gpu_index < num_gpu; ++gpu_index) {
    const double gain = (reinterpret_cast<const double*>(leaf_best_split_info_buffer + 6))[0];
    if (gain > best_gain) {
      best_gain = gain;
      best_gpu_for_smaller_leaf = gpu_index;
    }
  }
  if (best_gpu_for_smaller_leaf >= 0) {
    const int* buffer = leaf_best_split_info_buffer + best_gpu_for_smaller_leaf * 10;
    leaf_best_split_info_buffer[0] = buffer[0];
    leaf_best_split_info_buffer[1] = buffer[1];
    leaf_best_split_info_buffer[2] = buffer[2];
    double* gain_buffer = reinterpret_cast<double*>(leaf_best_split_info_buffer + 6);
    gain_buffer[0] = (reinterpret_cast<const double*>(buffer + 6))[0];
  }
  
  int best_gpu_for_larger_leaf = -1;
  best_gain = kMinScore;
  for (int gpu_index = 0; gpu_index < num_gpu; ++gpu_index) {
    const double gain = (reinterpret_cast<const double*>(leaf_best_split_info_buffer + 6))[1];
    if (gain > best_gain) {
      best_gain = gain;
      best_gpu_for_larger_leaf = gpu_index;
    }
  }
  if (best_gpu_for_larger_leaf >= 0) {
    const int* buffer = leaf_best_split_info_buffer + best_gpu_for_larger_leaf * 10;
    leaf_best_split_info_buffer[3] = buffer[3];
    leaf_best_split_info_buffer[4] = buffer[4];
    leaf_best_split_info_buffer[5] = buffer[5];
    double* gain_buffer = reinterpret_cast<double*>(leaf_best_split_info_buffer + 6);
    gain_buffer[1] = (reinterpret_cast<const double*>(buffer + 6))[1];
  }
  leaf_best_split_info_buffer[10] = best_gpu_for_smaller_leaf;
  leaf_best_split_info_buffer[11] = best_gpu_for_larger_leaf;
}

void CUDAExpTreeLearner::LaunchReduceBestSplitsForLeafKernel() {
  ReduceBestSplitsForLeafKernel<<<1, 1>>>(
    best_split_info_buffer_.RawData(),
    config_->num_gpu);
}

}  // namespace LightGBM
