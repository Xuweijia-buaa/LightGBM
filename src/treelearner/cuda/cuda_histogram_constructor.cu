
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_histogram_constructor.hpp"

#include <LightGBM/cuda/cuda_algorithms.hpp>

#include <algorithm>

namespace LightGBM {

void CUDAHistogramConstructor::CalcConstructHistogramKernelDim(
  int* grid_dim_x,
  int* grid_dim_y,
  int* block_dim_x,
  int* block_dim_y,
  const data_size_t num_data_in_leaf) {
  *block_dim_x = cuda_row_data_->max_num_column_per_partition();
  *block_dim_y = NUM_THRADS_PER_BLOCK / cuda_row_data_->max_num_column_per_partition();
  *grid_dim_x = cuda_row_data_->num_feature_partitions();
  *grid_dim_y = std::max(min_grid_dim_y_,
    ((num_data_in_leaf + NUM_DATA_PER_THREAD - 1) / NUM_DATA_PER_THREAD + (*block_dim_y) - 1) / (*block_dim_y));
}

__device__ void CUDAHistogramConstructor::CalcConstructHistogramKernelDim(
  int* grid_dim_x,
  int* grid_dim_y,
  int* block_dim_x,
  int* block_dim_y,
  const data_size_t num_data_in_smaller_leaf,
  const int max_num_column_per_partition,
  const int num_feature_partitions,
  const int min_grid_dim_y) {
  *block_dim_x = max_num_column_per_partition;
  *block_dim_y = NUM_THRADS_PER_BLOCK / max_num_column_per_partition;
  *grid_dim_x = num_feature_partitions;
  *grid_dim_y = max(min_grid_dim_y,
    ((num_data_in_smaller_leaf + NUM_DATA_PER_THREAD - 1) / NUM_DATA_PER_THREAD + (*block_dim_y) - 1) / (*block_dim_y));
}

template <typename BIN_TYPE, typename HIST_TYPE, int SHARED_HIST_SIZE>
__global__ void CUDAConstructHistogramDenseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ HIST_TYPE shared_hist[SHARED_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + partition_column_start * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    HIST_TYPE* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[data_index * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      HIST_TYPE* pos_ptr = shared_hist_ptr + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
      inner_data_index += blockDim.y;
    }
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE, typename HIST_TYPE, int SHARED_HIST_SIZE>
__global__ void CUDAConstructHistogramDenseKernelLaunch(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessians_in_leaf,
  const int max_num_column_per_partition,
  const int num_feature_partitions,
  const int min_grid_dim_y) {
  if ((smaller_leaf_splits->num_data_in_leaf <= min_data_in_leaf || smaller_leaf_splits->sum_of_hessians <= min_sum_hessians_in_leaf) &&
      (larger_leaf_splits->num_data_in_leaf <= min_data_in_leaf || larger_leaf_splits->sum_of_hessians <= min_sum_hessians_in_leaf)) {
    return;
  }
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CUDAHistogramConstructor::CalcConstructHistogramKernelDim(
    &grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y,
    smaller_leaf_splits->num_data_in_leaf,
    max_num_column_per_partition, num_feature_partitions, min_grid_dim_y);
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  CUDAConstructHistogramDenseKernel<BIN_TYPE, HIST_TYPE, SHARED_HIST_SIZE>
    <<<dim3(grid_dim_x, grid_dim_y), dim3(block_dim_x, block_dim_y), 0, cuda_stream>>>(
      smaller_leaf_splits,
      cuda_gradients,
      cuda_hessians,
      data,
      column_hist_offsets,
      column_hist_offsets_full,
      feature_partition_column_index_offsets,
      num_data);
  cudaStreamDestroy(cuda_stream);
}

template <typename BIN_TYPE, int SHARED_HIST_SIZE>
__global__ void CUDAConstructDiscretizedHistogramDenseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const int32_t* cuda_gradients_and_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ int16_t shared_hist[SHARED_HIST_SIZE];
  int32_t* shared_hist_packed = reinterpret_cast<int32_t*>(shared_hist);
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + partition_column_start * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    int32_t* shared_hist_ptr = shared_hist_packed + (column_hist_offsets[column_index]);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const int32_t grad_and_hess = cuda_gradients_and_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[data_index * num_columns_in_partition + threadIdx.x]);
      int32_t* pos_ptr = shared_hist_ptr + bin;
      atomicAdd_block(pos_ptr, grad_and_hess);
      inner_data_index += blockDim.y;
    }
  }
  __syncthreads();
  int32_t* feature_histogram_ptr = reinterpret_cast<int32_t*>(smaller_leaf_splits->hist_in_leaf) + (partition_hist_start << 2);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + (i * 2), static_cast<int32_t>(shared_hist[i]));
  }
}

template <typename BIN_TYPE, int SHARED_HIST_SIZE>
__global__ void CUDAConstructDiscretizedHistogramDenseKernelLaunch(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  const int32_t* cuda_gradients_and_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessians_in_leaf,
  const int max_num_column_per_partition,
  const int num_feature_partitions,
  const int min_grid_dim_y) {
  if ((smaller_leaf_splits->num_data_in_leaf <= min_data_in_leaf || smaller_leaf_splits->sum_of_hessians <= min_sum_hessians_in_leaf) &&
      (larger_leaf_splits->num_data_in_leaf <= min_data_in_leaf || larger_leaf_splits->sum_of_hessians <= min_sum_hessians_in_leaf)) {
    return;
  }
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CUDAHistogramConstructor::CalcConstructHistogramKernelDim(
    &grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y,
    smaller_leaf_splits->num_data_in_leaf,
    max_num_column_per_partition, num_feature_partitions, min_grid_dim_y);
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  CUDAConstructDiscretizedHistogramDenseKernel<BIN_TYPE, SHARED_HIST_SIZE>
    <<<dim3(grid_dim_x, grid_dim_y), dim3(block_dim_x, block_dim_y), 0, cuda_stream>>>(
      smaller_leaf_splits,
      cuda_gradients_and_hessians,
      data,
      column_hist_offsets,
      column_hist_offsets_full,
      feature_partition_column_index_offsets,
      num_data);
  cudaStreamDestroy(cuda_stream);
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE, typename HIST_TYPE, int SHARED_HIST_SIZE>
__global__ void CUDAConstructHistogramSparseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ HIST_TYPE shared_hist[SHARED_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + blockIdx.x * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      HIST_TYPE* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE, typename HIST_TYPE, int SHARED_HIST_SIZE>
__global__ void CUDAConstructHistogramSparseKernelLaunch(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessians_in_leaf,
  const int max_num_column_per_partition,
  const int num_feature_partitions,
  const int min_grid_dim_y) {
  if ((smaller_leaf_splits->num_data_in_leaf <= min_data_in_leaf || smaller_leaf_splits->sum_of_hessians <= min_sum_hessians_in_leaf) &&
      (larger_leaf_splits->num_data_in_leaf <= min_data_in_leaf || larger_leaf_splits->sum_of_hessians <= min_sum_hessians_in_leaf)) {
    return;
  }
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CUDAHistogramConstructor::CalcConstructHistogramKernelDim(
    &grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y,
    smaller_leaf_splits->num_data_in_leaf,
    max_num_column_per_partition, num_feature_partitions, min_grid_dim_y);
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  CUDAConstructHistogramSparseKernel<BIN_TYPE, DATA_PTR_TYPE, HIST_TYPE, SHARED_HIST_SIZE>
    <<<dim3(grid_dim_x, grid_dim_y), dim3(block_dim_x, block_dim_y), 0, cuda_stream>>>(
      smaller_leaf_splits,
      cuda_gradients,
      cuda_hessians,
      data,
      row_ptr,
      partition_ptr,
      column_hist_offsets_full,
      num_data);
  cudaStreamDestroy(cuda_stream);
}

// TODO(shiyu1994): global memory buffer should also has double precision option
/*template <typename BIN_TYPE>
__global__ void CUDAConstructHistogramDenseKernel_GlobalMemory(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data,
  float* global_hist_buffer) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + partition_column_start * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  const int num_total_bin = column_hist_offsets_full[gridDim.x];
  float* shared_hist = global_hist_buffer + (blockIdx.y * num_total_bin + partition_hist_start) * 2;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    float* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[data_index * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist_ptr + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
      inner_data_index += blockDim.y;
    }
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

// TODO(shiyu1994): global memory buffer should also has double precision option
template <typename BIN_TYPE, typename DATA_PTR_TYPE>
__global__ void CUDAConstructHistogramSparseKernel_GlobalMemory(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data,
  float* global_hist_buffer) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + blockIdx.x * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  const int num_total_bin = column_hist_offsets_full[gridDim.x];
  float* shared_hist = global_hist_buffer + (blockIdx.y * num_total_bin + partition_hist_start) * 2;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}*/

void CUDAHistogramConstructor::LaunchConstructHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  if (gpu_use_discretized_grad_) {
    CHECK_EQ(cuda_row_data_->shared_hist_size(), 6144 * 4);
    LaunchConstructDiscretizedHistogramKernel(cuda_smaller_leaf_splits, cuda_larger_leaf_splits);
  } else if (cuda_row_data_->use_dp()) {
    CHECK_EQ(cuda_row_data_->shared_hist_size(), 6144);
    LaunchConstructHistogramKernelInner<double, 6144>(cuda_smaller_leaf_splits, cuda_larger_leaf_splits);
  } else {
    CHECK_EQ(cuda_row_data_->shared_hist_size(), 6144 * 2);
    LaunchConstructHistogramKernelInner<float, 6144 * 2>(cuda_smaller_leaf_splits, cuda_larger_leaf_splits);
  }
}

void CUDAHistogramConstructor::LaunchConstructDiscretizedHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  if (!cuda_row_data_->is_sparse()) {
    if (cuda_row_data_->bit_type() == 8) {
      CHECK_EQ(cuda_row_data_->shared_hist_size(), 6144 * 4);
      CUDAConstructDiscretizedHistogramDenseKernelLaunch<uint8_t, 6144 * 4><<<1, 1, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_larger_leaf_splits,
        reinterpret_cast<const int32_t*>(cuda_gradients_),
        cuda_row_data_->cuda_data_uint8(),
        cuda_row_data_->cuda_column_hist_offsets(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        cuda_row_data_->cuda_feature_partition_column_index_offsets(),
        num_data_,
        min_data_in_leaf_,
        min_sum_hessian_in_leaf_,
        cuda_row_data_->max_num_column_per_partition(),
        cuda_row_data_->num_feature_partitions(),
        min_grid_dim_y_);
    }
  }
}

template <typename HIST_TYPE, int SHARED_HIST_SIZE>
void CUDAHistogramConstructor::LaunchConstructHistogramKernelInner(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  if (cuda_row_data_->NumLargeBinPartition() == 0) {
    if (cuda_row_data_->is_sparse()) {
      if (cuda_row_data_->bit_type() == 8) {
        if (cuda_row_data_->row_ptr_bit_type() == 16) {
          CUDAConstructHistogramSparseKernelLaunch<uint8_t, uint16_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint8(),
            cuda_row_data_->cuda_row_ptr_uint16(),
            cuda_row_data_->cuda_partition_ptr_uint16(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
          CUDAConstructHistogramSparseKernelLaunch<uint8_t, uint32_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint8(),
            cuda_row_data_->cuda_row_ptr_uint32(),
            cuda_row_data_->cuda_partition_ptr_uint32(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
          CUDAConstructHistogramSparseKernelLaunch<uint8_t, uint64_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint8(),
            cuda_row_data_->cuda_row_ptr_uint64(),
            cuda_row_data_->cuda_partition_ptr_uint64(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        }
      } else if (cuda_row_data_->bit_type() == 16) {
        if (cuda_row_data_->row_ptr_bit_type() == 16) {
          CUDAConstructHistogramSparseKernelLaunch<uint16_t, uint16_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint16(),
            cuda_row_data_->cuda_row_ptr_uint16(),
            cuda_row_data_->cuda_partition_ptr_uint16(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
          CUDAConstructHistogramSparseKernelLaunch<uint16_t, uint32_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint16(),
            cuda_row_data_->cuda_row_ptr_uint32(),
            cuda_row_data_->cuda_partition_ptr_uint32(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
          CUDAConstructHistogramSparseKernelLaunch<uint16_t, uint64_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint16(),
            cuda_row_data_->cuda_row_ptr_uint64(),
            cuda_row_data_->cuda_partition_ptr_uint64(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        }
      } else if (cuda_row_data_->bit_type() == 32) {
        if (cuda_row_data_->row_ptr_bit_type() == 16) {
          CUDAConstructHistogramSparseKernelLaunch<uint32_t, uint16_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint32(),
            cuda_row_data_->cuda_row_ptr_uint16(),
            cuda_row_data_->cuda_partition_ptr_uint16(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
          CUDAConstructHistogramSparseKernelLaunch<uint32_t, uint32_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint32(),
            cuda_row_data_->cuda_row_ptr_uint32(),
            cuda_row_data_->cuda_partition_ptr_uint32(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
          CUDAConstructHistogramSparseKernelLaunch<uint32_t, uint64_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_larger_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint32(),
            cuda_row_data_->cuda_row_ptr_uint64(),
            cuda_row_data_->cuda_partition_ptr_uint64(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            min_data_in_leaf_,
            min_sum_hessian_in_leaf_,
            cuda_row_data_->max_num_column_per_partition(),
            cuda_row_data_->num_feature_partitions(),
            min_grid_dim_y_);
        }
      }
    } else {
      if (cuda_row_data_->bit_type() == 8) {
        CUDAConstructHistogramDenseKernelLaunch<uint8_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_larger_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint8(),
          cuda_row_data_->cuda_column_hist_offsets(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          cuda_row_data_->cuda_feature_partition_column_index_offsets(),
          num_data_,
          min_data_in_leaf_,
          min_sum_hessian_in_leaf_,
          cuda_row_data_->max_num_column_per_partition(),
          cuda_row_data_->num_feature_partitions(),
          min_grid_dim_y_);
      } else if (cuda_row_data_->bit_type() == 16) {
        CUDAConstructHistogramDenseKernelLaunch<uint16_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_larger_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint16(),
          cuda_row_data_->cuda_column_hist_offsets(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          cuda_row_data_->cuda_feature_partition_column_index_offsets(),
          num_data_,
          min_data_in_leaf_,
          min_sum_hessian_in_leaf_,
          cuda_row_data_->max_num_column_per_partition(),
          cuda_row_data_->num_feature_partitions(),
          min_grid_dim_y_);
      } else if (cuda_row_data_->bit_type() == 32) {
        CUDAConstructHistogramDenseKernelLaunch<uint32_t, HIST_TYPE, SHARED_HIST_SIZE><<<1, 1, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_larger_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint32(),
          cuda_row_data_->cuda_column_hist_offsets(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          cuda_row_data_->cuda_feature_partition_column_index_offsets(),
          num_data_,
          min_data_in_leaf_,
          min_sum_hessian_in_leaf_,
          cuda_row_data_->max_num_column_per_partition(),
          cuda_row_data_->num_feature_partitions(),
          min_grid_dim_y_);
      }
    }
  }/* else {
    if (cuda_row_data_->is_sparse()) {
      if (cuda_row_data_->bit_type() == 8) {
        if (cuda_row_data_->row_ptr_bit_type() == 16) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint8_t, uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint8(),
            cuda_row_data_->cuda_row_ptr_uint16(),
            cuda_row_data_->cuda_partition_ptr_uint16(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint8_t, uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint8(),
            cuda_row_data_->cuda_row_ptr_uint32(),
            cuda_row_data_->cuda_partition_ptr_uint32(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint8_t, uint64_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint8(),
            cuda_row_data_->cuda_row_ptr_uint64(),
            cuda_row_data_->cuda_partition_ptr_uint64(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        }
      } else if (cuda_row_data_->bit_type() == 16) {
        if (cuda_row_data_->row_ptr_bit_type() == 16) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint16_t, uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint16(),
            cuda_row_data_->cuda_row_ptr_uint16(),
            cuda_row_data_->cuda_partition_ptr_uint16(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint16_t, uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint16(),
            cuda_row_data_->cuda_row_ptr_uint32(),
            cuda_row_data_->cuda_partition_ptr_uint32(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint16_t, uint64_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint16(),
            cuda_row_data_->cuda_row_ptr_uint64(),
            cuda_row_data_->cuda_partition_ptr_uint64(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        }
      } else if (cuda_row_data_->bit_type() == 32) {
        if (cuda_row_data_->row_ptr_bit_type() == 16) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint32_t, uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint32(),
            cuda_row_data_->cuda_row_ptr_uint16(),
            cuda_row_data_->cuda_partition_ptr_uint16(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint32_t, uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint32(),
            cuda_row_data_->cuda_row_ptr_uint32(),
            cuda_row_data_->cuda_partition_ptr_uint32(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
          CUDAConstructHistogramSparseKernel_GlobalMemory<uint32_t, uint64_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
            cuda_smaller_leaf_splits,
            cuda_gradients_, cuda_hessians_,
            cuda_row_data_->cuda_data_uint32(),
            cuda_row_data_->cuda_row_ptr_uint64(),
            cuda_row_data_->cuda_partition_ptr_uint64(),
            cuda_row_data_->cuda_partition_hist_offsets(),
            num_data_,
            cuda_hist_buffer_);
        }
      }
    } else {
      if (cuda_row_data_->bit_type() == 8) {
        CUDAConstructHistogramDenseKernel_GlobalMemory<uint8_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint8(),
          cuda_row_data_->cuda_column_hist_offsets(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          cuda_row_data_->cuda_feature_partition_column_index_offsets(),
          num_data_,
          cuda_hist_buffer_);
      } else if (cuda_row_data_->bit_type() == 16) {
        CUDAConstructHistogramDenseKernel_GlobalMemory<uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint16(),
          cuda_row_data_->cuda_column_hist_offsets(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          cuda_row_data_->cuda_feature_partition_column_index_offsets(),
          num_data_,
          cuda_hist_buffer_);
      } else if (cuda_row_data_->bit_type() == 32) {
        CUDAConstructHistogramDenseKernel_GlobalMemory<uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint32(),
          cuda_row_data_->cuda_column_hist_offsets(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          cuda_row_data_->cuda_feature_partition_column_index_offsets(),
          num_data_,
          cuda_hist_buffer_);
      }
    }
  }*/
}

__global__ void SubtractHistogramKernel(
  const int num_total_bin,
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  const unsigned int global_thread_index = threadIdx.x + blockIdx.x * blockDim.x;
  const int cuda_larger_leaf_index_ref = cuda_larger_leaf_splits->leaf_index;
  if (cuda_larger_leaf_index_ref >= 0) {
    const hist_t* smaller_leaf_hist = cuda_smaller_leaf_splits->hist_in_leaf;
    hist_t* larger_leaf_hist = cuda_larger_leaf_splits->hist_in_leaf;
    if (global_thread_index < 2 * num_total_bin) {
      larger_leaf_hist[global_thread_index] -= smaller_leaf_hist[global_thread_index];
    }
  }
}

__global__ void FixHistogramKernel(
  const uint32_t* cuda_feature_num_bins,
  const uint32_t* cuda_feature_hist_offsets,
  const uint32_t* cuda_feature_most_freq_bins,
  const int* cuda_need_fix_histogram_features,
  const uint32_t* cuda_need_fix_histogram_features_num_bin_aligned,
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits) {
  __shared__ hist_t shared_mem_buffer[32];
  const unsigned int blockIdx_x = blockIdx.x;
  const int feature_index = cuda_need_fix_histogram_features[blockIdx_x];
  const uint32_t num_bin_aligned = cuda_need_fix_histogram_features_num_bin_aligned[blockIdx_x];
  const uint32_t feature_hist_offset = cuda_feature_hist_offsets[feature_index];
  const uint32_t most_freq_bin = cuda_feature_most_freq_bins[feature_index];
  const double leaf_sum_gradients = cuda_smaller_leaf_splits->sum_of_gradients;
  const double leaf_sum_hessians = cuda_smaller_leaf_splits->sum_of_hessians;
  hist_t* feature_hist = cuda_smaller_leaf_splits->hist_in_leaf + feature_hist_offset * 2;
  const unsigned int threadIdx_x = threadIdx.x;
  const uint32_t num_bin = cuda_feature_num_bins[feature_index];
  const uint32_t hist_pos = threadIdx_x << 1;
  const hist_t bin_gradient = (threadIdx_x < num_bin && threadIdx_x != most_freq_bin) ? feature_hist[hist_pos] : 0.0f;
  const hist_t bin_hessian = (threadIdx_x < num_bin && threadIdx_x != most_freq_bin) ? feature_hist[hist_pos + 1] : 0.0f;
  const hist_t sum_gradient = ShuffleReduceSum<hist_t>(bin_gradient, shared_mem_buffer, num_bin_aligned);
  const hist_t sum_hessian = ShuffleReduceSum<hist_t>(bin_hessian, shared_mem_buffer, num_bin_aligned);
  if (threadIdx_x == 0) {
    feature_hist[most_freq_bin << 1] = leaf_sum_gradients - sum_gradient;
    feature_hist[(most_freq_bin << 1) + 1] = leaf_sum_hessians - sum_hessian;
  }
}

void CUDAHistogramConstructor::LaunchSubtractHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  const int num_subtract_threads = 2 * num_total_bin_;
  const int num_subtract_blocks = (num_subtract_threads + SUBTRACT_BLOCK_SIZE - 1) / SUBTRACT_BLOCK_SIZE;
  global_timer.Start("CUDAHistogramConstructor::FixHistogramKernel");
  if (need_fix_histogram_features_.size() > 0) {
    FixHistogramKernel<<<need_fix_histogram_features_.size(), FIX_HISTOGRAM_BLOCK_SIZE, 0, cuda_stream_>>>(
      cuda_feature_num_bins_,
      cuda_feature_hist_offsets_,
      cuda_feature_most_freq_bins_,
      cuda_need_fix_histogram_features_,
      cuda_need_fix_histogram_features_num_bin_aligned_,
      cuda_smaller_leaf_splits);
  }
  global_timer.Stop("CUDAHistogramConstructor::FixHistogramKernel");
  global_timer.Start("CUDAHistogramConstructor::SubtractHistogramKernel");
  SubtractHistogramKernel<<<num_subtract_blocks, SUBTRACT_BLOCK_SIZE, 0, cuda_stream_>>>(
    num_total_bin_,
    cuda_smaller_leaf_splits,
    cuda_larger_leaf_splits);
  global_timer.Stop("CUDAHistogramConstructor::SubtractHistogramKernel");
}

}  // namespace LightGBM

#endif  // USE_CUDA
