
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include <LightGBM/cuda/cuda_algorithms.hpp>
#include "cuda_histogram_constructor.hpp"
#include "cuda_fp16.h"

namespace LightGBM {

template <typename BIN_TYPE>
__global__ void CUDAConstructHistogramDenseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ float shared_hist[SHRAE_HIST_SIZE];
  __half local_hist[32] {0.0f};
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + partition_column_start * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    //float* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[data_index * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      //float* pos_ptr = shared_hist_ptr + pos;
      //atomicAdd_block(pos_ptr, grad);
      //atomicAdd_block(pos_ptr + 1, hess);
      /*if (pos > 512 || (pos + 1 > 512)) {
        printf("error pos = %d\n");
      }*/
      local_hist[pos] += __half(grad);
      local_hist[pos + 1] += __half(hess);
      inner_data_index += blockDim.y;
    }
    float* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    const unsigned int num_items_in_column = (column_index == (partition_column_end - 1) ? (num_items_in_partition - 2 * column_hist_offsets[column_index]) :
                                            2 * (column_hist_offsets[column_index + 1] - column_hist_offsets[column_index]));
    //printf("column_index = %d, num_items_in_column = %d\n", column_index, num_items_in_column);
    for (unsigned int i = 0; i < num_items_in_column; ++i) {
      const unsigned int offset_i = (i + threadIdx_y) % num_items_in_column;
      atomicAdd_block(shared_hist_ptr + offset_i, local_hist[offset_i]);//);
    }
    /*if (threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 && blockIdx.y == 0) {
      printf("num_iteration_this = %d\n", num_iteration_this);
      for (int i = 0; i < 100; ++i) {
        printf("local_hist[%d] = %f\n", i, local_hist[i]);
      }
    }*/
    __syncthreads();
    hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
    for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
      atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
    }
  }
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE>
__global__ void CUDAConstructHistogramSparseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ float shared_hist[SHRAE_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + blockIdx.x * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

void CUDAHistogramConstructor::LaunchConstructHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const data_size_t num_data_in_smaller_leaf) {
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CalcConstructHistogramKernelDim(&grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y, num_data_in_smaller_leaf);
  dim3 grid_dim(grid_dim_x, grid_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  if (cuda_row_data_->is_sparse()) {
    if (cuda_row_data_->bit_type() == 8) {
      if (cuda_row_data_->row_ptr_bit_type() == 16) {
        CUDAConstructHistogramSparseKernel<uint8_t, uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint8(),
          cuda_row_data_->cuda_row_ptr_uint16(),
          cuda_row_data_->cuda_partition_ptr_uint16(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
        CUDAConstructHistogramSparseKernel<uint8_t, uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint8(),
          cuda_row_data_->cuda_row_ptr_uint32(),
          cuda_row_data_->cuda_partition_ptr_uint32(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
        CUDAConstructHistogramSparseKernel<uint8_t, uint64_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint8(),
          cuda_row_data_->cuda_row_ptr_uint64(),
          cuda_row_data_->cuda_partition_ptr_uint64(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      }
    } else if (cuda_row_data_->bit_type() == 16) {
      if (cuda_row_data_->row_ptr_bit_type() == 16) {
        CUDAConstructHistogramSparseKernel<uint16_t, uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint16(),
          cuda_row_data_->cuda_row_ptr_uint16(),
          cuda_row_data_->cuda_partition_ptr_uint16(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
        CUDAConstructHistogramSparseKernel<uint16_t, uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint16(),
          cuda_row_data_->cuda_row_ptr_uint32(),
          cuda_row_data_->cuda_partition_ptr_uint32(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
        CUDAConstructHistogramSparseKernel<uint16_t, uint64_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint16(),
          cuda_row_data_->cuda_row_ptr_uint64(),
          cuda_row_data_->cuda_partition_ptr_uint64(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      }
    } else if (cuda_row_data_->bit_type() == 32) {
      if (cuda_row_data_->row_ptr_bit_type() == 16) {
        CUDAConstructHistogramSparseKernel<uint32_t, uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint32(),
          cuda_row_data_->cuda_row_ptr_uint16(),
          cuda_row_data_->cuda_partition_ptr_uint16(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
        CUDAConstructHistogramSparseKernel<uint32_t, uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint32(),
          cuda_row_data_->cuda_row_ptr_uint32(),
          cuda_row_data_->cuda_partition_ptr_uint32(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
        CUDAConstructHistogramSparseKernel<uint32_t, uint64_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
          cuda_smaller_leaf_splits,
          cuda_gradients_, cuda_hessians_,
          cuda_row_data_->cuda_data_uint32(),
          cuda_row_data_->cuda_row_ptr_uint64(),
          cuda_row_data_->cuda_partition_ptr_uint64(),
          cuda_row_data_->cuda_partition_hist_offsets(),
          num_data_);
      }
    }
  } else {
    if (cuda_row_data_->bit_type() == 8) {
      CUDAConstructHistogramDenseKernel<uint8_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->cuda_data_uint8(),
        cuda_row_data_->cuda_column_hist_offsets(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        cuda_row_data_->cuda_feature_partition_column_index_offsets(),
        num_data_);
    } else if (cuda_row_data_->bit_type() == 16) {
      CUDAConstructHistogramDenseKernel<uint16_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->cuda_data_uint16(),
        cuda_row_data_->cuda_column_hist_offsets(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        cuda_row_data_->cuda_feature_partition_column_index_offsets(),
        num_data_);
    } else if (cuda_row_data_->bit_type() == 32) {
      CUDAConstructHistogramDenseKernel<uint32_t><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->cuda_data_uint32(),
        cuda_row_data_->cuda_column_hist_offsets(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        cuda_row_data_->cuda_feature_partition_column_index_offsets(),
        num_data_);
    }
  }
}

__global__ void SubtractHistogramKernel(
  const int num_total_bin,
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  const unsigned int global_thread_index = threadIdx.x + blockIdx.x * blockDim.x;
  const int cuda_larger_leaf_index_ref = cuda_larger_leaf_splits->leaf_index;
  if (cuda_larger_leaf_index_ref >= 0) { 
    const hist_t* smaller_leaf_hist = cuda_smaller_leaf_splits->hist_in_leaf;
    hist_t* larger_leaf_hist = cuda_larger_leaf_splits->hist_in_leaf;
    if (global_thread_index < 2 * num_total_bin) {
      larger_leaf_hist[global_thread_index] -= smaller_leaf_hist[global_thread_index];
    }
  }
}

__global__ void FixHistogramKernel(
  const uint32_t* cuda_feature_num_bins,
  const uint32_t* cuda_feature_hist_offsets,
  const uint32_t* cuda_feature_most_freq_bins,
  const int* cuda_need_fix_histogram_features,
  const uint32_t* cuda_need_fix_histogram_features_num_bin_aligned,
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits) {
  const unsigned int blockIdx_x = blockIdx.x;
  const int feature_index = cuda_need_fix_histogram_features[blockIdx_x];
  __shared__ double hist_gradients[FIX_HISTOGRAM_SHARED_MEM_SIZE + 1];
  __shared__ double hist_hessians[FIX_HISTOGRAM_SHARED_MEM_SIZE + 1];
  const uint32_t num_bin_aligned = cuda_need_fix_histogram_features_num_bin_aligned[blockIdx_x];
  const uint32_t feature_hist_offset = cuda_feature_hist_offsets[feature_index];
  const uint32_t most_freq_bin = cuda_feature_most_freq_bins[feature_index];
  const double leaf_sum_gradients = cuda_smaller_leaf_splits->sum_of_gradients;
  const double leaf_sum_hessians = cuda_smaller_leaf_splits->sum_of_hessians;
  hist_t* feature_hist = cuda_smaller_leaf_splits->hist_in_leaf + feature_hist_offset * 2;
  const unsigned int threadIdx_x = threadIdx.x;
  const uint32_t num_bin = cuda_feature_num_bins[feature_index];
  const uint32_t hist_pos = threadIdx_x << 1;
  if (threadIdx_x < num_bin) {
    if (threadIdx_x == most_freq_bin) {
      hist_gradients[threadIdx_x] = 0.0f;
      hist_hessians[threadIdx_x] = 0.0f;
    } else {
      hist_gradients[threadIdx_x] = feature_hist[hist_pos];
      hist_hessians[threadIdx_x] = feature_hist[hist_pos + 1];
    }
  } else {
    hist_gradients[threadIdx_x] = 0.0f;
    hist_hessians[threadIdx_x] = 0.0f;
  }
  __syncthreads();
  ReduceSum<hist_t>(hist_gradients, num_bin_aligned);
  ReduceSum<hist_t>(hist_hessians, num_bin_aligned);
  __syncthreads();
  if (threadIdx_x == most_freq_bin) {
    feature_hist[hist_pos] = leaf_sum_gradients - hist_gradients[0];
    feature_hist[hist_pos + 1] = leaf_sum_hessians - hist_hessians[0];
  }
}

void CUDAHistogramConstructor::LaunchSubtractHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  const int num_subtract_threads = 2 * num_total_bin_;
  const int num_subtract_blocks = (num_subtract_threads + SUBTRACT_BLOCK_SIZE - 1) / SUBTRACT_BLOCK_SIZE;
  global_timer.Start("CUDAHistogramConstructor::FixHistogramKernel");
  FixHistogramKernel<<<need_fix_histogram_features_.size(), FIX_HISTOGRAM_BLOCK_SIZE, 0, cuda_stream_>>>(
    cuda_feature_num_bins_,
    cuda_feature_hist_offsets_,
    cuda_feature_most_freq_bins_,
    cuda_need_fix_histogram_features_,
    cuda_need_fix_histogram_features_num_bin_aligned_,
    cuda_smaller_leaf_splits);
  global_timer.Stop("CUDAHistogramConstructor::FixHistogramKernel");
  global_timer.Start("CUDAHistogramConstructor::SubtractHistogramKernel");
  SubtractHistogramKernel<<<num_subtract_blocks, SUBTRACT_BLOCK_SIZE, 0, cuda_stream_>>>(
    num_total_bin_,
    cuda_smaller_leaf_splits,
    cuda_larger_leaf_splits);
  global_timer.Stop("CUDAHistogramConstructor::SubtractHistogramKernel");
}

}  // namespace LightGBM

#endif  // USE_CUDA
