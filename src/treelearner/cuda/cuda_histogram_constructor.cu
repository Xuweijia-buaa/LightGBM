
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_histogram_constructor.hpp"

namespace LightGBM {

__device__ void PrefixSum(hist_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = (n - 1);
  const hist_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[(dst_pos)] += elements[(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int conflict_free_dst_pos = (dst_pos);
      const unsigned int conflict_free_src_pos = (src_pos);
      const hist_t src_val = elements[conflict_free_src_pos];
      elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
      elements[conflict_free_dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__device__ void ReduceSumHistogramConstructor(hist_t* array, const size_t size) {
  const unsigned int threadIdx_x = threadIdx.x;
  const size_t atomic_size = size / 4;
  for (int s = 1; s < atomic_size; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < size) {
      array[threadIdx_x] += array[threadIdx_x + s];
    }
    __syncthreads();
  }
  if (threadIdx_x > 0 && threadIdx_x % atomic_size == 0) {
    atomicAdd_block(array, array[threadIdx_x]);
  }
  __syncthreads();
}

__device__ void ReduceSumHistogramConstructorMerge(hist_t* array, const size_t size) {
  const unsigned int threadIdx_x = (threadIdx.x % USED_HISTOGRAM_BUFFER_NUM);
  const size_t atomic_size = size / 4;
  for (int s = 1; s < atomic_size; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < size) {
      array[threadIdx_x] += array[threadIdx_x + s];
    }
    __syncthreads();
  }
  if (threadIdx_x > 0 && threadIdx_x % atomic_size == 0) {
    atomicAdd_block(array, array[threadIdx_x]);
  }
  __syncthreads();
}

template <typename BIN_TYPE>
__global__ void CUDAConstructHistogramDenseKernel(
  const int* leaf_index,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const data_size_t** data_indices_ptr,
  hist_t** feature_histogram,
  const int* num_feature_groups,
  const data_size_t* leaf_num_data,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data) {

  const int leaf_index_ref = *leaf_index;
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf_ref = leaf_num_data[leaf_index_ref];
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf_ref + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = *data_indices_ptr;
  __shared__ float shared_hist[SHRAE_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + partition_column_start * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf_ref - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    float* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[data_index * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist_ptr + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
      inner_data_index += blockDim.y;
    }
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = (*feature_histogram) + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE>
__global__ void CUDAConstructHistogramSparseKernel(
  const int* leaf_index,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const data_size_t** data_indices_ptr,
  hist_t** feature_histogram,
  const int* num_feature_groups,
  const data_size_t* leaf_num_data,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data) {

  const int leaf_index_ref = *leaf_index;
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf_ref = leaf_num_data[leaf_index_ref];
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf_ref + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = *data_indices_ptr;
  __shared__ float shared_hist[SHRAE_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + blockIdx.x * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf_ref - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = (*feature_histogram) + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE>
__global__ void CUDAConstructHistogramDenseKernel2(
  const int* leaf_index,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const data_size_t** data_indices_ptr,
  const int* num_feature_groups,
  const data_size_t* leaf_num_data,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data,
  hist_t* histogram_buffer,
  const int total_num_bin) {

  const int leaf_index_ref = *leaf_index;
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf_ref = leaf_num_data[leaf_index_ref];
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf_ref + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = *data_indices_ptr;
  __shared__ float shared_hist[SHRAE_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + partition_column_start * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf_ref - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    float* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[data_index * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist_ptr + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
      inner_data_index += blockDim.y;
    }
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = histogram_buffer + total_num_bin * (blockIdx.y % USED_HISTOGRAM_BUFFER_NUM) * 2 + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE>
__global__ void CUDAConstructHistogramSparseKernel2(
  const int* leaf_index,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const data_size_t** data_indices_ptr,
  const int* num_feature_groups,
  const data_size_t* leaf_num_data,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data,
  hist_t* histogram_buffer,
  const int total_num_bin) {

  const int leaf_index_ref = *leaf_index;
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf_ref = leaf_num_data[leaf_index_ref];
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf_ref + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = *data_indices_ptr;
  __shared__ float shared_hist[SHRAE_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + blockIdx.x * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf_ref - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = histogram_buffer + total_num_bin * (blockIdx.y % USED_HISTOGRAM_BUFFER_NUM) * 2 + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

__global__ void MergeHistogramBufferKernel(
  hist_t* histogram_buffer,
  const int num_total_bin,
  const int num_bin_per_block,
  hist_t** output_histogram_ptr) {
  hist_t* output_histogram = *output_histogram_ptr;
  __shared__ hist_t gradient_buffer[1024];
  __shared__ hist_t hessian_buffer[1024];
  const uint32_t threadIdx_x = threadIdx.x;
  const uint32_t blockIdx_x = blockIdx.x;
  const uint32_t bin_index = threadIdx_x / USED_HISTOGRAM_BUFFER_NUM + num_bin_per_block * blockIdx_x;
  const uint32_t histogram_position = (num_total_bin * (threadIdx_x % USED_HISTOGRAM_BUFFER_NUM) + bin_index) << 1;
  if (bin_index < num_total_bin) {
    gradient_buffer[threadIdx_x] = histogram_buffer[histogram_position];
    hessian_buffer[threadIdx_x] = histogram_buffer[histogram_position + 1];
  }
  const uint32_t start = threadIdx_x / USED_HISTOGRAM_BUFFER_NUM * USED_HISTOGRAM_BUFFER_NUM;
  __syncthreads();
  ReduceSumHistogramConstructorMerge(gradient_buffer + start, USED_HISTOGRAM_BUFFER_NUM);
  ReduceSumHistogramConstructorMerge(hessian_buffer + start, USED_HISTOGRAM_BUFFER_NUM);
  __syncthreads();
  const unsigned int global_histogram_position = bin_index << 1;
  if (threadIdx_x % USED_HISTOGRAM_BUFFER_NUM == 0 && bin_index < num_total_bin) {
    output_histogram[global_histogram_position] = gradient_buffer[threadIdx_x];
    output_histogram[global_histogram_position + 1] = hessian_buffer[threadIdx_x];
  }
}

void CUDAHistogramConstructor::LaunchConstructHistogramKernel(
  const int* cuda_smaller_leaf_index,
  const data_size_t* cuda_smaller_leaf_num_data,
  const data_size_t** cuda_data_indices_in_smaller_leaf,
  const data_size_t* cuda_leaf_num_data,
  hist_t** cuda_leaf_hist,
  const data_size_t num_data_in_smaller_leaf) {
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CalcConstructHistogramKernelDim(&grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y, num_data_in_smaller_leaf);
  dim3 grid_dim(grid_dim_x, grid_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  if (is_sparse_) {
    if (bit_type_ == 8) {
      if (data_ptr_bit_type_ == 16) {
        CUDAConstructHistogramSparseKernel<uint8_t, uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint8_t_,
          cuda_row_ptr_uint16_t_,
          cuda_partition_ptr_uint16_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      } else if (data_ptr_bit_type_ == 32) {
        CUDAConstructHistogramSparseKernel<uint8_t, uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint8_t_,
          cuda_row_ptr_uint32_t_,
          cuda_partition_ptr_uint32_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      } else if (data_ptr_bit_type_ == 64) {
        CUDAConstructHistogramSparseKernel<uint8_t, uint64_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint8_t_,
          cuda_row_ptr_uint64_t_,
          cuda_partition_ptr_uint64_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      }
    } else if (bit_type_ == 16) {
      if (data_ptr_bit_type_ == 16) {
        CUDAConstructHistogramSparseKernel<uint16_t, uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint16_t_,
          cuda_row_ptr_uint16_t_,
          cuda_partition_ptr_uint16_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      } else if (data_ptr_bit_type_ == 32) {
        CUDAConstructHistogramSparseKernel<uint16_t, uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint16_t_,
          cuda_row_ptr_uint32_t_,
          cuda_partition_ptr_uint32_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      } else if (data_ptr_bit_type_ == 64) {
        CUDAConstructHistogramSparseKernel<uint16_t, uint64_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint16_t_,
          cuda_row_ptr_uint64_t_,
          cuda_partition_ptr_uint64_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      }
    } else if (bit_type_ == 32) {
      if (data_ptr_bit_type_ == 16) {
        CUDAConstructHistogramSparseKernel<uint32_t, uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint32_t_,
          cuda_row_ptr_uint16_t_,
          cuda_partition_ptr_uint16_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      } else if (data_ptr_bit_type_ == 32) {
        CUDAConstructHistogramSparseKernel<uint32_t, uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint32_t_,
          cuda_row_ptr_uint32_t_,
          cuda_partition_ptr_uint32_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      } else if (data_ptr_bit_type_ == 64) {
        CUDAConstructHistogramSparseKernel<uint32_t, uint64_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint32_t_,
          cuda_row_ptr_uint64_t_,
          cuda_partition_ptr_uint64_t_,
          cuda_column_hist_offsets_full_,
          num_data_);
      }
    }
  } else {
    if (bit_type_ == 8) {
      CUDAConstructHistogramDenseKernel<uint8_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
        cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data, cuda_data_uint8_t_,
        cuda_column_hist_offsets_,
        cuda_column_hist_offsets_full_,
        cuda_feature_partition_column_index_offsets_,
        num_data_);
    } else if (bit_type_ == 16) {
      CUDAConstructHistogramDenseKernel<uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
        cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data, cuda_data_uint16_t_,
        cuda_column_hist_offsets_,
        cuda_column_hist_offsets_full_,
        cuda_feature_partition_column_index_offsets_,
        num_data_);
    } else if (bit_type_ == 32) {
      CUDAConstructHistogramDenseKernel<uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
        cuda_data_indices_in_smaller_leaf, cuda_leaf_hist, cuda_num_feature_groups_, cuda_leaf_num_data, cuda_data_uint32_t_,
        cuda_column_hist_offsets_,
        cuda_column_hist_offsets_full_,
        cuda_feature_partition_column_index_offsets_,
        num_data_);
    }
  }
}

void CUDAHistogramConstructor::LaunchConstructHistogramKernel2(
  const int* cuda_smaller_leaf_index,
  const data_size_t* cuda_smaller_leaf_num_data,
  const data_size_t** cuda_data_indices_in_smaller_leaf,
  const data_size_t* cuda_leaf_num_data,
  hist_t** cuda_leaf_hist,
  const data_size_t num_data_in_smaller_leaf) {
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CalcConstructHistogramKernelDim(&grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y, num_data_in_smaller_leaf);
  dim3 grid_dim(grid_dim_x, grid_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  SetCUDAMemory<hist_t>(block_cuda_hist_buffer_, 0, 2 * num_total_bin_ * USED_HISTOGRAM_BUFFER_NUM);
  global_timer.Start("CUDAConstructHistogramKernel2");
  if (is_sparse_) {
    if (bit_type_ == 8) {
      if (data_ptr_bit_type_ == 16) {
        CUDAConstructHistogramSparseKernel2<uint8_t, uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint8_t_,
          cuda_row_ptr_uint16_t_,
          cuda_partition_ptr_uint16_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      } else if (data_ptr_bit_type_ == 32) {
        CUDAConstructHistogramSparseKernel2<uint8_t, uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint8_t_,
          cuda_row_ptr_uint32_t_,
          cuda_partition_ptr_uint32_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      } else if (data_ptr_bit_type_ == 64) {
        CUDAConstructHistogramSparseKernel2<uint8_t, uint64_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint8_t_,
          cuda_row_ptr_uint64_t_,
          cuda_partition_ptr_uint64_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      }
    } else if (bit_type_ == 16) {
      if (data_ptr_bit_type_ == 16) {
        CUDAConstructHistogramSparseKernel2<uint16_t, uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint16_t_,
          cuda_row_ptr_uint16_t_,
          cuda_partition_ptr_uint16_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      } else if (data_ptr_bit_type_ == 32) {
        CUDAConstructHistogramSparseKernel2<uint16_t, uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint16_t_,
          cuda_row_ptr_uint32_t_,
          cuda_partition_ptr_uint32_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      } else if (data_ptr_bit_type_ == 64) {
        CUDAConstructHistogramSparseKernel2<uint16_t, uint64_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint16_t_,
          cuda_row_ptr_uint64_t_,
          cuda_partition_ptr_uint64_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      }
    } else if (bit_type_ == 32) {
      if (data_ptr_bit_type_ == 16) {
        CUDAConstructHistogramSparseKernel2<uint32_t, uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint32_t_,
          cuda_row_ptr_uint16_t_,
          cuda_partition_ptr_uint16_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      } else if (data_ptr_bit_type_ == 32) {
        CUDAConstructHistogramSparseKernel2<uint32_t, uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint32_t_,
          cuda_row_ptr_uint32_t_,
          cuda_partition_ptr_uint32_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      } else if (data_ptr_bit_type_ == 64) {
        CUDAConstructHistogramSparseKernel2<uint32_t, uint64_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
          cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data,
          cuda_data_uint32_t_,
          cuda_row_ptr_uint64_t_,
          cuda_partition_ptr_uint64_t_,
          cuda_column_hist_offsets_full_,
          num_data_, block_cuda_hist_buffer_, num_total_bin_);
      }
    }
  } else {
    if (bit_type_ == 8) {
      CUDAConstructHistogramDenseKernel2<uint8_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
        cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data, cuda_data_uint8_t_,
        cuda_column_hist_offsets_,
        cuda_column_hist_offsets_full_,
        cuda_feature_partition_column_index_offsets_,
        num_data_, block_cuda_hist_buffer_, num_total_bin_);
    } else if (bit_type_ == 16) {
      CUDAConstructHistogramDenseKernel2<uint16_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
        cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data, cuda_data_uint16_t_,
        cuda_column_hist_offsets_,
        cuda_column_hist_offsets_full_,
        cuda_feature_partition_column_index_offsets_,
        num_data_, block_cuda_hist_buffer_, num_total_bin_);
    } else if (bit_type_ == 32) {
      CUDAConstructHistogramDenseKernel2<uint32_t><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(cuda_smaller_leaf_index, cuda_gradients_, cuda_hessians_,
        cuda_data_indices_in_smaller_leaf, cuda_num_feature_groups_, cuda_leaf_num_data, cuda_data_uint32_t_,
        cuda_column_hist_offsets_,
        cuda_column_hist_offsets_full_,
        cuda_feature_partition_column_index_offsets_,
        num_data_, block_cuda_hist_buffer_, num_total_bin_);
    }
  }
  global_timer.Stop("CUDAConstructHistogramKernel2");
  const int merge_block_dim = 1024;
  const int num_bin_per_block = merge_block_dim / USED_HISTOGRAM_BUFFER_NUM;
  const int num_blocks = (num_total_bin_ + num_bin_per_block - 1) / num_bin_per_block;
  global_timer.Start("MergeHistogramBufferKernel");
  MergeHistogramBufferKernel<<<num_blocks, merge_block_dim, 0, cuda_streams_[0]>>>(
    block_cuda_hist_buffer_, num_total_bin_, num_bin_per_block, cuda_leaf_hist);
  global_timer.Stop("MergeHistogramBufferKernel");
}

__global__ void SubtractHistogramKernel(const int* /*cuda_smaller_leaf_index*/,
  const int* cuda_larger_leaf_index, const uint8_t* cuda_feature_mfb_offsets,
  const uint32_t* cuda_feature_num_bins, const int* cuda_num_total_bin,
  hist_t** cuda_smaller_leaf_hist, hist_t** cuda_larger_leaf_hist) {
  const int cuda_num_total_bin_ref = *cuda_num_total_bin;
  const unsigned int global_thread_index = threadIdx.x + blockIdx.x * blockDim.x;
  const int cuda_larger_leaf_index_ref = *cuda_larger_leaf_index;
  if (cuda_larger_leaf_index_ref >= 0) { 
    const hist_t* smaller_leaf_hist = *cuda_smaller_leaf_hist;
    hist_t* larger_leaf_hist = *cuda_larger_leaf_hist;
    if (global_thread_index < 2 * cuda_num_total_bin_ref) {
      larger_leaf_hist[global_thread_index] -= smaller_leaf_hist[global_thread_index];
    }
  }
}

__global__ void FixHistogramKernel(
  const uint32_t* cuda_feature_num_bins,
  const uint32_t* cuda_feature_hist_offsets,
  const uint32_t* cuda_feature_most_freq_bins,
  const double* smaller_leaf_sum_gradients, const double* smaller_leaf_sum_hessians,
  hist_t** cuda_smaller_leaf_hist,
  const int* cuda_need_fix_histogram_features,
  const uint32_t* cuda_need_fix_histogram_features_num_bin_aligned) {
  const unsigned int blockIdx_x = blockIdx.x;
  const int feature_index = cuda_need_fix_histogram_features[blockIdx_x];
  __shared__ double hist_gradients[FIX_HISTOGRAM_SHARED_MEM_SIZE + 1];
  __shared__ double hist_hessians[FIX_HISTOGRAM_SHARED_MEM_SIZE + 1];
  const uint32_t num_bin_aligned = cuda_need_fix_histogram_features_num_bin_aligned[blockIdx_x];
  const uint32_t feature_hist_offset = cuda_feature_hist_offsets[feature_index];
  const uint32_t most_freq_bin = cuda_feature_most_freq_bins[feature_index];
  const double leaf_sum_gradients = *smaller_leaf_sum_gradients;
  const double leaf_sum_hessians = *smaller_leaf_sum_hessians;
  hist_t* feature_hist = (*cuda_smaller_leaf_hist) + feature_hist_offset * 2;
  const unsigned int threadIdx_x = threadIdx.x;
  const uint32_t num_bin = cuda_feature_num_bins[feature_index];
  const uint32_t hist_pos = threadIdx_x << 1;
  if (threadIdx_x < num_bin) {
    if (threadIdx_x == most_freq_bin) {
      hist_gradients[threadIdx_x] = 0.0f;
      hist_hessians[threadIdx_x] = 0.0f;
    } else {
      hist_gradients[threadIdx_x] = feature_hist[hist_pos];
      hist_hessians[threadIdx_x] = feature_hist[hist_pos + 1];
    }
  } else {
    hist_gradients[threadIdx_x] = 0.0f;
    hist_hessians[threadIdx_x] = 0.0f;
  }
  __syncthreads();
  ReduceSumHistogramConstructor(hist_gradients, num_bin_aligned);
  ReduceSumHistogramConstructor(hist_hessians, num_bin_aligned);
  __syncthreads();
  if (threadIdx_x == most_freq_bin) {
    feature_hist[hist_pos] = leaf_sum_gradients - hist_gradients[0];
    feature_hist[hist_pos + 1] = leaf_sum_hessians - hist_hessians[0];
  }
}

void CUDAHistogramConstructor::LaunchSubtractHistogramKernel(const int* cuda_smaller_leaf_index,
  const int* cuda_larger_leaf_index, const double* smaller_leaf_sum_gradients, const double* smaller_leaf_sum_hessians,
  const double* larger_leaf_sum_gradients, const double* larger_leaf_sum_hessians,
  hist_t** cuda_smaller_leaf_hist, hist_t** cuda_larger_leaf_hist) {
  const int num_subtract_threads = 2 * num_total_bin_;
  const int num_subtract_blocks = (num_subtract_threads + SUBTRACT_BLOCK_SIZE - 1) / SUBTRACT_BLOCK_SIZE;
  global_timer.Start("CUDAHistogramConstructor::FixHistogramKernel");
  FixHistogramKernel<<<need_fix_histogram_features_.size(), FIX_HISTOGRAM_BLOCK_SIZE, 0, cuda_streams_[0]>>>(
    cuda_feature_num_bins_,
    cuda_feature_hist_offsets_,
    cuda_feature_most_freq_bins_, smaller_leaf_sum_gradients, smaller_leaf_sum_hessians,
    cuda_smaller_leaf_hist, cuda_need_fix_histogram_features_,
    cuda_need_fix_histogram_features_num_bin_aligned_);
  //SynchronizeCUDADevice();
  global_timer.Stop("CUDAHistogramConstructor::FixHistogramKernel");
  global_timer.Start("CUDAHistogramConstructor::SubtractHistogramKernel");
  SubtractHistogramKernel<<<num_subtract_blocks, SUBTRACT_BLOCK_SIZE, 0, cuda_streams_[0]>>>(
    cuda_smaller_leaf_index, cuda_larger_leaf_index, cuda_feature_mfb_offsets_,
    cuda_feature_num_bins_, cuda_num_total_bin_, cuda_smaller_leaf_hist, cuda_larger_leaf_hist);
  //SynchronizeCUDADevice();
  global_timer.Stop("CUDAHistogramConstructor::SubtractHistogramKernel");
}

__global__ void GetOrderedGradientsKernel(const data_size_t num_data_in_leaf, const data_size_t** cuda_data_indices_in_leaf,
  const score_t* cuda_gradients, const score_t* cuda_hessians,
  score_t* cuda_ordered_gradients, score_t* cuda_ordered_hessians) {
  const data_size_t* cuda_data_indices_in_leaf_ref = *cuda_data_indices_in_leaf;
  const unsigned int local_data_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (local_data_index < static_cast<unsigned int>(num_data_in_leaf)) {
    const data_size_t global_data_index = cuda_data_indices_in_leaf_ref[local_data_index];
    cuda_ordered_gradients[local_data_index] = cuda_gradients[global_data_index];
    cuda_ordered_hessians[local_data_index] = cuda_hessians[global_data_index];
  }
}

void CUDAHistogramConstructor::LaunchGetOrderedGradientsKernel(
  const data_size_t num_data_in_leaf,
  const data_size_t** cuda_data_indices_in_leaf) {
  if (num_data_in_leaf < num_data_) {
    const int num_data_per_block = 1024;
    const int num_blocks = (num_data_in_leaf + num_data_per_block - 1) / num_data_per_block;
    GetOrderedGradientsKernel<<<num_blocks, num_data_per_block>>>(num_data_in_leaf, cuda_data_indices_in_leaf,
      cuda_gradients_, cuda_hessians_, cuda_ordered_gradients_, cuda_ordered_hessians_);
    SynchronizeCUDADevice();
  }
}

}  // namespace LightGBM

#endif  // USE_CUDA
