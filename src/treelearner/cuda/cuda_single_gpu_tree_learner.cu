
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include <LightGBM/cuda/cuda_algorithms.hpp>

#include "cuda_single_gpu_tree_learner.hpp"

namespace LightGBM {

__global__ void ReduceLeafStatKernel_SharedMemory(
  const score_t* gradients,
  const score_t* hessians,
  const int num_leaves,
  const data_size_t num_data,
  const int* data_index_to_leaf_index,
  double* leaf_grad_stat_buffer,
  double* leaf_hess_stat_buffer) {
  extern __shared__ double shared_mem[];
  double* shared_grad_sum = shared_mem;
  double* shared_hess_sum = shared_mem + num_leaves;
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  for (int leaf_index = static_cast<int>(threadIdx.x); leaf_index < num_leaves; leaf_index += static_cast<int>(blockDim.x)) {
    shared_grad_sum[leaf_index] = 0.0f;
    shared_hess_sum[leaf_index] = 0.0f;
  }
  __syncthreads();
  if (data_index < num_data) {
    const int leaf_index = data_index_to_leaf_index[data_index];
    atomicAdd_block(shared_grad_sum + leaf_index, gradients[data_index]);
    atomicAdd_block(shared_hess_sum + leaf_index, hessians[data_index]);
  }
  __syncthreads();
  for (int leaf_index = static_cast<int>(threadIdx.x); leaf_index < num_leaves; leaf_index += static_cast<int>(blockDim.x)) {
    atomicAdd_system(leaf_grad_stat_buffer + leaf_index, shared_grad_sum[leaf_index]);
    atomicAdd_system(leaf_hess_stat_buffer + leaf_index, shared_hess_sum[leaf_index]);
  }
}

__global__ void ReduceLeafStatKernel_GlobalMemory(
  const score_t* gradients,
  const score_t* hessians,
  const int num_leaves,
  const data_size_t num_data,
  const int* data_index_to_leaf_index,
  double* leaf_grad_stat_buffer,
  double* leaf_hess_stat_buffer) {
  const size_t offset = static_cast<size_t>(num_leaves) * (blockIdx.x + 1);
  double* grad_sum = leaf_grad_stat_buffer + offset;
  double* hess_sum = leaf_hess_stat_buffer + offset;
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  for (int leaf_index = static_cast<int>(threadIdx.x); leaf_index < num_leaves; leaf_index += static_cast<int>(blockDim.x)) {
    grad_sum[leaf_index] = 0.0f;
    hess_sum[leaf_index] = 0.0f;
  }
  __syncthreads();
  if (data_index < num_data) {
    const int leaf_index = data_index_to_leaf_index[data_index];
    atomicAdd_block(grad_sum + leaf_index, gradients[data_index]);
    atomicAdd_block(hess_sum + leaf_index, hessians[data_index]);
  }
  __syncthreads();
  for (int leaf_index = static_cast<int>(threadIdx.x); leaf_index < num_leaves; leaf_index += static_cast<int>(blockDim.x)) {
    atomicAdd_system(leaf_grad_stat_buffer + leaf_index, grad_sum[leaf_index]);
    atomicAdd_system(leaf_hess_stat_buffer + leaf_index, hess_sum[leaf_index]);
  }
}

__global__ void CalcRefitLeafOutputKernel(
  const int num_leaves,
  const double* leaf_grad_stat_buffer,
  const double* leaf_hess_stat_buffer,
  const double lambda_l1,
  const bool use_l1,
  const double lambda_l2,
  const double shrinkage_rate,
  const double refit_decay_rate,
  double* leaf_value) {
  const int leaf_index = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
  if (leaf_index < num_leaves) {
    const double sum_gradients = leaf_grad_stat_buffer[leaf_index];
    const double sum_hessians = leaf_hess_stat_buffer[leaf_index];
    const double old_leaf_value = leaf_value[leaf_index];
    double new_leaf_value = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_gradients, sum_hessians, lambda_l1, use_l1, lambda_l2);
    if (isnan(new_leaf_value)) {
      new_leaf_value = 0.0f;
    } else {
      new_leaf_value *= shrinkage_rate;
    }
    leaf_value[leaf_index] = refit_decay_rate * old_leaf_value + (1.0f - refit_decay_rate) * new_leaf_value;
  }
}

void CUDASingleGPUTreeLearner::LaunchReduceLeafStatKernel(
  const score_t* gradients, const score_t* hessians, const int num_leaves,
  const data_size_t num_data, double* cuda_leaf_value, const double shrinkage_rate) const {
  int num_block = (num_data + CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE - 1) / CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE;
  if (num_leaves <= 2048) {
    ReduceLeafStatKernel_SharedMemory<<<num_block, CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE, 2 * num_leaves * sizeof(double)>>>(
      gradients, hessians, num_leaves, num_data, cuda_data_partition_->cuda_data_index_to_leaf_index(),
      cuda_leaf_gradient_stat_buffer_, cuda_leaf_hessian_stat_buffer_);
  } else {
    ReduceLeafStatKernel_GlobalMemory<<<num_block, CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE>>>(
      gradients, hessians, num_leaves, num_data, cuda_data_partition_->cuda_data_index_to_leaf_index(),
      cuda_leaf_gradient_stat_buffer_, cuda_leaf_hessian_stat_buffer_);
  }
  const bool use_l1 = config_->lambda_l1 > 0.0f;
  num_block = (num_leaves + CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE - 1) / CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE;
  CalcRefitLeafOutputKernel<<<num_block, CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE>>>(
    num_leaves, cuda_leaf_gradient_stat_buffer_, cuda_leaf_hessian_stat_buffer_,
    config_->lambda_l1, use_l1, config_->lambda_l2, shrinkage_rate, config_->refit_decay_rate, cuda_leaf_value);
}

template <typename T, bool IS_INNER>
__global__ void CalcBitsetLenKernel(const CUDASplitInfo* best_split_info, size_t* out_len_buffer) {
  __shared__ size_t shared_mem_buffer[32];
  const T* vals = nullptr;
  if (IS_INNER) {
    vals = reinterpret_cast<const T*>(best_split_info->cat_threshold);
  } else {
    vals = reinterpret_cast<const T*>(best_split_info->cat_threshold_real);
  }
  const int i = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
  size_t len = 0;
  if (i < best_split_info->num_cat_threshold) {
    const T val = vals[i];
    len = (val / 32) + 1;
  }
  const size_t block_max_len = ShuffleReduceMax<size_t>(len, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    out_len_buffer[blockIdx.x] = block_max_len;
  }
}

__global__ void ReduceBlockMaxLen(size_t* out_len_buffer, const int num_blocks) {
  __shared__ size_t shared_mem_buffer[32];
  size_t max_len = 0;
  for (int i = static_cast<int>(threadIdx.x); i < num_blocks; i += static_cast<int>(blockDim.x)) {
    max_len = max(out_len_buffer[i], max_len);
  }
  const size_t all_max_len = ShuffleReduceMax<size_t>(max_len, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    out_len_buffer[0] = max_len;
  }
}

template <typename T, bool IS_INNER>
__global__ void CUDAConstructBitsetKernel(const CUDASplitInfo* best_split_info, uint32_t* out, size_t cuda_bitset_len) {
  const T* vals = nullptr;
  for (size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < cuda_bitset_len; i += blockDim.x) {
    out[i] = 0;
  }
  __syncthreads();
  if (IS_INNER) {
    vals = reinterpret_cast<const T*>(best_split_info->cat_threshold);
  } else {
    vals = reinterpret_cast<const T*>(best_split_info->cat_threshold_real);
  }
  const int i = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
  if (i < best_split_info->num_cat_threshold) {
    const T val = vals[i];
    // can use add instead of or here, because each bit will only be added once
    atomicAdd_system(out + (val / 32), (0x1 << (val % 32)));
  }
}

__global__ void SetRealThresholdKernel(
  const CUDASplitInfo* best_split_info,
  const int* categorical_bin_to_value,
  const int* categorical_bin_offsets) {
  const int num_cat_threshold = best_split_info->num_cat_threshold;
  const int* categorical_bin_to_value_ptr = categorical_bin_to_value + categorical_bin_offsets[best_split_info->inner_feature_index];
  int* cat_threshold_real = best_split_info->cat_threshold_real;
  const uint32_t* cat_threshold = best_split_info->cat_threshold;
  for (int i = 0; i < num_cat_threshold; ++i) {
    cat_threshold_real[i] = categorical_bin_to_value_ptr[cat_threshold[i]];
  }
}

template <typename T, bool IS_INNER>
void CUDAConstructBitset(const CUDASplitInfo* best_split_info, const int num_cat_threshold, uint32_t* out, size_t bitset_len) {
  const int num_blocks = (num_cat_threshold + CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE - 1) / CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE;
  CUDAConstructBitsetKernel<T, IS_INNER><<<num_blocks, CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE>>>(best_split_info, out, bitset_len);
}

template <typename T, bool IS_INNER>
size_t CUDABitsetLen(const CUDASplitInfo* best_split_info, const int num_cat_threshold, size_t* out_len_buffer) {
  const int num_blocks = (num_cat_threshold + CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE - 1) / CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE;
  CalcBitsetLenKernel<T, IS_INNER><<<num_blocks, CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE>>>(best_split_info, out_len_buffer);
  ReduceBlockMaxLen<<<1, CUDA_SINGLE_GPU_TREE_LEARNER_BLOCK_SIZE>>>(out_len_buffer, num_blocks);
  size_t host_max_len = 0;
  CopyFromCUDADeviceToHost<size_t>(&host_max_len, out_len_buffer, 1, __FILE__, __LINE__);
  return host_max_len;
}

void CUDASingleGPUTreeLearner::LaunchConstructBitsetForCategoricalSplitKernel(
  const CUDASplitInfo* best_split_info) {
  SetRealThresholdKernel<<<1, 1>>>(best_split_info, cuda_categorical_bin_to_value_, cuda_categorical_bin_offsets_);
  cuda_bitset_inner_len_ = CUDABitsetLen<uint32_t, true>(best_split_info, num_cat_threshold_, cuda_block_bitset_len_buffer_);
  CUDAConstructBitset<uint32_t, true>(best_split_info, num_cat_threshold_, cuda_bitset_inner_, cuda_bitset_inner_len_);
  cuda_bitset_len_ = CUDABitsetLen<int, false>(best_split_info, num_cat_threshold_, cuda_block_bitset_len_buffer_);
  CUDAConstructBitset<int, false>(best_split_info, num_cat_threshold_, cuda_bitset_, cuda_bitset_len_);
}

}  // namespace LightGBM

#endif  // USE_CUDA
