
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_data_partition.hpp"

#include <LightGBM/cuda/cuda_algorithms.hpp>
#include <LightGBM/tree.h>

#include <algorithm>
#include <vector>

namespace LightGBM {

__global__ void FillDataIndicesBeforeTrainKernel(const data_size_t num_data,
  data_size_t* data_indices, int* cuda_data_index_to_leaf_index) {
  const unsigned int data_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (data_index < num_data) {
    data_indices[data_index] = data_index;
    cuda_data_index_to_leaf_index[data_index] = 0;
  }
}

__global__ void FillDataIndexToLeafIndexKernel(
  const data_size_t num_data,
  const data_size_t* data_indices,
  int* data_index_to_leaf_index) {
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (data_index < num_data) {
    data_index_to_leaf_index[data_indices[data_index]] = 0;
  }
}

void CUDADataPartition::LaunchFillDataIndicesBeforeTrain() {
  const data_size_t num_data_in_root = root_num_data();
  const int num_blocks = (num_data_in_root + FILL_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / FILL_INDICES_BLOCK_SIZE_DATA_PARTITION;
  FillDataIndicesBeforeTrainKernel<<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(num_data_in_root, cuda_data_indices_, cuda_data_index_to_leaf_index_);
}

void CUDADataPartition::LaunchFillDataIndexToLeafIndex() {
  const data_size_t num_data_in_root = root_num_data();
  const int num_blocks = (num_data_in_root + FILL_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / FILL_INDICES_BLOCK_SIZE_DATA_PARTITION;
  FillDataIndexToLeafIndexKernel<<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(num_data_in_root, cuda_data_indices_, cuda_data_index_to_leaf_index_);
}

__device__ __forceinline__ void PrepareOffset(const data_size_t num_data_in_leaf, uint16_t* block_to_left_offset,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const uint16_t thread_to_left_offset_cnt, uint16_t* shared_mem_buffer) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const uint16_t thread_to_left_offset = ShufflePrefixSum<uint16_t>(thread_to_left_offset_cnt, shared_mem_buffer);
  const data_size_t num_data_in_block = (blockIdx.x + 1) * blockDim_x <= num_data_in_leaf ? static_cast<data_size_t>(blockDim_x) :
    num_data_in_leaf - static_cast<data_size_t>(blockIdx.x * blockDim_x);
  if (static_cast<data_size_t>(threadIdx_x) < num_data_in_block) {
    block_to_left_offset[threadIdx_x] = thread_to_left_offset;
  }
  if (threadIdx_x == blockDim_x - 1) {
    if (num_data_in_block > 0) {
      const data_size_t data_to_left = static_cast<data_size_t>(thread_to_left_offset);
      block_to_left_offset_buffer[blockIdx.x + 1] = data_to_left;
      block_to_right_offset_buffer[blockIdx.x + 1] = num_data_in_block - data_to_left;
    } else {
      block_to_left_offset_buffer[blockIdx.x + 1] = 0;
      block_to_right_offset_buffer[blockIdx.x + 1] = 0;
    }
  }
}

template <typename T>
__device__ bool CUDAFindInBitset(const uint32_t* bits, int n, T pos) {
  int i1 = pos / 32;
  if (i1 >= n) {
    return false;
  }
  int i2 = pos % 32;
  return (bits[i1] >> i2) & 1;
}



#define UpdateDataIndexToLeafIndexKernel_PARAMS \
  const BIN_TYPE* column_data, \
  const data_size_t num_data_in_leaf, \
  const data_size_t* data_indices_in_leaf, \
  const uint32_t th, \
  const uint32_t t_zero_bin, \
  const uint32_t max_bin, \
  const uint32_t min_bin, \
  const int left_leaf_index, \
  const int right_leaf_index, \
  const int default_leaf_index, \
  const int missing_default_leaf_index

#define UpdateDataIndexToLeafIndex_ARGS \
  column_data, \
  num_data_in_leaf, \
  data_indices_in_leaf, th, \
  t_zero_bin, \
  max_bin, \
  min_bin, \
  left_leaf_index, \
  right_leaf_index, \
  default_leaf_index, \
  missing_default_leaf_index

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, bool MAX_TO_LEFT, typename BIN_TYPE>
__global__ void UpdateDataIndexToLeafIndexKernel(
  UpdateDataIndexToLeafIndexKernel_PARAMS,
  int* cuda_data_index_to_leaf_index) {
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (!MIN_IS_MAX) {
      if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
        (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin)) {
        cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
      } else if (bin < min_bin || bin > max_bin) {
        if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
          cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
        } else {
          cuda_data_index_to_leaf_index[global_data_index] = default_leaf_index;
        }
      } else if (bin > th) {
        cuda_data_index_to_leaf_index[global_data_index] = right_leaf_index;
      } else {
        cuda_data_index_to_leaf_index[global_data_index] = left_leaf_index;
      }
    } else {
      if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
        cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
      } else if (bin != max_bin) {
        if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
          cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
        } else {
          cuda_data_index_to_leaf_index[global_data_index] = default_leaf_index;
        }
      } else {
        if (MISSING_IS_NA && !MFB_IS_NA) {
          cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
        } else {
          if (!MAX_TO_LEFT) {
            cuda_data_index_to_leaf_index[global_data_index] = right_leaf_index;
          } else {
            cuda_data_index_to_leaf_index[global_data_index] = left_leaf_index;
          }
        }
      }
    }
  }
}

template <typename BIN_TYPE>
void CUDADataPartition::LaunchUpdateDataIndexToLeafIndexKernel(
  UpdateDataIndexToLeafIndexKernel_PARAMS,
  const bool missing_is_zero,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_to_left) {
  if (min_bin < max_bin) {
    if (!missing_is_zero) {
      LaunchUpdateDataIndexToLeafIndexKernel_Inner0<false, false, BIN_TYPE>
        (UpdateDataIndexToLeafIndex_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_to_left);
    } else {
      LaunchUpdateDataIndexToLeafIndexKernel_Inner0<false, true, BIN_TYPE>
        (UpdateDataIndexToLeafIndex_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_to_left);
    }
  } else {
    if (!missing_is_zero) {
      LaunchUpdateDataIndexToLeafIndexKernel_Inner0<true, false, BIN_TYPE>
        (UpdateDataIndexToLeafIndex_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_to_left);
    } else {
      LaunchUpdateDataIndexToLeafIndexKernel_Inner0<true, true, BIN_TYPE>
        (UpdateDataIndexToLeafIndex_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_to_left);
    }
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, typename BIN_TYPE>
void CUDADataPartition::LaunchUpdateDataIndexToLeafIndexKernel_Inner0(
  UpdateDataIndexToLeafIndexKernel_PARAMS,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_to_left) {
  if (!missing_is_na) {
    LaunchUpdateDataIndexToLeafIndexKernel_Inner1<MIN_IS_MAX, MISSING_IS_ZERO, false, BIN_TYPE>
      (UpdateDataIndexToLeafIndex_ARGS, mfb_is_zero, mfb_is_na, max_to_left);
  } else {
    LaunchUpdateDataIndexToLeafIndexKernel_Inner1<MIN_IS_MAX, MISSING_IS_ZERO, true, BIN_TYPE>
      (UpdateDataIndexToLeafIndex_ARGS, mfb_is_zero, mfb_is_na, max_to_left);
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, typename BIN_TYPE>
void CUDADataPartition::LaunchUpdateDataIndexToLeafIndexKernel_Inner1(
  UpdateDataIndexToLeafIndexKernel_PARAMS,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_to_left) {
  if (!mfb_is_zero) {
    LaunchUpdateDataIndexToLeafIndexKernel_Inner2<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, false, BIN_TYPE>
      (UpdateDataIndexToLeafIndex_ARGS, mfb_is_na, max_to_left);
  } else {
    LaunchUpdateDataIndexToLeafIndexKernel_Inner2<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, true, BIN_TYPE>
      (UpdateDataIndexToLeafIndex_ARGS, mfb_is_na, max_to_left);
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, typename BIN_TYPE>
void CUDADataPartition::LaunchUpdateDataIndexToLeafIndexKernel_Inner2(
  UpdateDataIndexToLeafIndexKernel_PARAMS,
  const bool mfb_is_na,
  const bool max_to_left) {
  if (!mfb_is_na) {
    LaunchUpdateDataIndexToLeafIndexKernel_Inner3<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, false, BIN_TYPE>
      (UpdateDataIndexToLeafIndex_ARGS, max_to_left);
  } else {
    LaunchUpdateDataIndexToLeafIndexKernel_Inner3<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, true, BIN_TYPE>
      (UpdateDataIndexToLeafIndex_ARGS, max_to_left);
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, typename BIN_TYPE>
void CUDADataPartition::LaunchUpdateDataIndexToLeafIndexKernel_Inner3(
  UpdateDataIndexToLeafIndexKernel_PARAMS,
  const bool max_to_left) {
  if (!max_to_left) {
    UpdateDataIndexToLeafIndexKernel<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, MFB_IS_NA, false, BIN_TYPE>
      <<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index_);
  } else {
    UpdateDataIndexToLeafIndexKernel<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, MFB_IS_NA, true, BIN_TYPE>
      <<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index_);
  }
}

#define GenDataToLeftBitVectorKernel_PARMS \
  const BIN_TYPE* column_data, \
  const data_size_t num_data_in_leaf, \
  const data_size_t* data_indices_in_leaf, \
  const uint32_t th, \
  const uint32_t t_zero_bin, \
  const uint32_t max_bin, \
  const uint32_t min_bin, \
  const uint8_t split_default_to_left, \
  const uint8_t split_missing_default_to_left

#define GenBitVector_ARGS \
  column_data, \
  num_data_in_leaf, \
  data_indices_in_leaf, \
  th, \
  t_zero_bin, \
  max_bin, \
  min_bin, \
  split_default_to_left,  \
  split_missing_default_to_left

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, bool MAX_TO_LEFT, typename BIN_TYPE>
__global__ void GenDataToLeftBitVectorKernel(
  GenDataToLeftBitVectorKernel_PARMS,
  uint16_t* block_to_left_offset,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer) {
  __shared__ uint16_t shared_mem_buffer[32];
  uint16_t thread_to_left_offset_cnt = 0;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (!MIN_IS_MAX) {
      if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
        (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin)) {
        thread_to_left_offset_cnt = split_missing_default_to_left;
      } else if ((bin < min_bin || bin > max_bin)) {
        if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
          thread_to_left_offset_cnt = split_missing_default_to_left;
        } else {
          thread_to_left_offset_cnt = split_default_to_left;
        }
      } else if (bin <= th) {
        thread_to_left_offset_cnt = 1;
      }
    } else {
      if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
        thread_to_left_offset_cnt = split_missing_default_to_left;
      } else if (bin != max_bin) {
        if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
          thread_to_left_offset_cnt = split_missing_default_to_left;
        } else {
          thread_to_left_offset_cnt = split_default_to_left;
        }
      } else {
        if (MISSING_IS_NA && !MFB_IS_NA) {
          thread_to_left_offset_cnt = split_missing_default_to_left;
        } else if (MAX_TO_LEFT) {
          thread_to_left_offset_cnt = 1;
        }
      }
    }
  }
  __syncthreads();
  PrepareOffset(num_data_in_leaf, block_to_left_offset + blockIdx.x * blockDim.x, block_to_left_offset_buffer, block_to_right_offset_buffer,
    thread_to_left_offset_cnt, shared_mem_buffer);
}

template <typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelInner(
  GenDataToLeftBitVectorKernel_PARMS,
  const bool missing_is_zero,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_bin_to_left) {
  if (min_bin < max_bin) {
    if (!missing_is_zero) {
      LaunchGenDataToLeftBitVectorKernelInner0<false, false, BIN_TYPE>
        (GenBitVector_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_bin_to_left);
    } else {
      LaunchGenDataToLeftBitVectorKernelInner0<false, true, BIN_TYPE>
        (GenBitVector_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_bin_to_left);
    }
  } else {
    if (!missing_is_zero) {
      LaunchGenDataToLeftBitVectorKernelInner0<true, false, BIN_TYPE>
        (GenBitVector_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_bin_to_left);
    } else {
      LaunchGenDataToLeftBitVectorKernelInner0<true, true, BIN_TYPE>
        (GenBitVector_ARGS, missing_is_na, mfb_is_zero, mfb_is_na, max_bin_to_left);
    }
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelInner0(
  GenDataToLeftBitVectorKernel_PARMS,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_bin_to_left) {
  if (!missing_is_na) {
    LaunchGenDataToLeftBitVectorKernelInner1<MIN_IS_MAX, MISSING_IS_ZERO, false, BIN_TYPE>
      (GenBitVector_ARGS, mfb_is_zero, mfb_is_na, max_bin_to_left);
  } else {
    LaunchGenDataToLeftBitVectorKernelInner1<MIN_IS_MAX, MISSING_IS_ZERO, true, BIN_TYPE>
      (GenBitVector_ARGS, mfb_is_zero, mfb_is_na, max_bin_to_left);
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelInner1(
  GenDataToLeftBitVectorKernel_PARMS,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_bin_to_left) {
  if (!mfb_is_zero) {
    LaunchGenDataToLeftBitVectorKernelInner2<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, false, BIN_TYPE>
      (GenBitVector_ARGS, mfb_is_na, max_bin_to_left);
  } else {
    LaunchGenDataToLeftBitVectorKernelInner2<MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, true, BIN_TYPE>
      (GenBitVector_ARGS, mfb_is_na, max_bin_to_left);
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelInner2(
  GenDataToLeftBitVectorKernel_PARMS,
  const bool mfb_is_na,
  const bool max_bin_to_left) {
  if (!mfb_is_na) {
    LaunchGenDataToLeftBitVectorKernelInner3
      <MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, false, BIN_TYPE>
      (GenBitVector_ARGS, max_bin_to_left);
  } else {
    LaunchGenDataToLeftBitVectorKernelInner3
      <MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, true, BIN_TYPE>
      (GenBitVector_ARGS, max_bin_to_left);
  }
}

template <bool MIN_IS_MAX, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelInner3(
  GenDataToLeftBitVectorKernel_PARMS,
  const bool max_bin_to_left) {
  if (!max_bin_to_left) {
    GenDataToLeftBitVectorKernel
      <MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, MFB_IS_NA, false, BIN_TYPE>
      <<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_ARGS,
        cuda_block_to_left_offset_, cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_);
  } else {
    GenDataToLeftBitVectorKernel
      <MIN_IS_MAX, MISSING_IS_ZERO, MISSING_IS_NA, MFB_IS_ZERO, MFB_IS_NA, true, BIN_TYPE>
      <<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_ARGS,
        cuda_block_to_left_offset_, cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_);
  }
}

__device__ void CUDADataPartition::CalcBlockDim(const data_size_t num_data_in_leaf, int* grid_dim, int* block_dim) {
  const int min_num_blocks = num_data_in_leaf <= 100 ? 1 : 80;
  const int num_blocks = max(min_num_blocks, (num_data_in_leaf + SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  int split_indices_block_size_data_partition = (num_data_in_leaf + num_blocks - 1) / num_blocks - 1;
  int split_indices_block_size_data_partition_aligned = 1;
  while (split_indices_block_size_data_partition > 0) {
    split_indices_block_size_data_partition_aligned <<= 1;
    split_indices_block_size_data_partition >>= 1;
  }
  const int num_blocks_final = (num_data_in_leaf + split_indices_block_size_data_partition_aligned - 1) / split_indices_block_size_data_partition_aligned;
  *grid_dim = num_blocks_final;
  *block_dim = split_indices_block_size_data_partition_aligned;
}


__global__ void AggregateBlockOffsetKernel0(
  const int left_leaf_index,
  const int right_leaf_index,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  const data_size_t num_blocks) {
  __shared__ uint32_t shared_mem_buffer[32];
  __shared__ uint32_t to_left_total_count;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[left_leaf_index];
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int threadIdx_x = threadIdx.x;
  const data_size_t num_blocks_plus_1 = num_blocks + 1;
  const uint32_t num_blocks_per_thread = (num_blocks_plus_1 + blockDim_x - 1) / blockDim_x;
  const uint32_t remain = num_blocks_plus_1 - ((num_blocks_per_thread - 1) * blockDim_x);
  const uint32_t remain_offset = remain * num_blocks_per_thread;
  uint32_t thread_start_block_index = 0;
  uint32_t thread_end_block_index = 0;
  if (threadIdx_x < remain) {
    thread_start_block_index = threadIdx_x * num_blocks_per_thread;
    thread_end_block_index = min(thread_start_block_index + num_blocks_per_thread, num_blocks_plus_1);
  } else {
    thread_start_block_index = remain_offset + (num_blocks_per_thread - 1) * (threadIdx_x - remain);
    thread_end_block_index = min(thread_start_block_index + num_blocks_per_thread - 1, num_blocks_plus_1);
  }
  if (threadIdx.x == 0) {
    block_to_right_offset_buffer[0] = 0;
  }
  __syncthreads();
  for (uint32_t block_index = thread_start_block_index + 1; block_index < thread_end_block_index; ++block_index) {
    block_to_left_offset_buffer[block_index] += block_to_left_offset_buffer[block_index - 1];
    block_to_right_offset_buffer[block_index] += block_to_right_offset_buffer[block_index - 1];
  }
  __syncthreads();
  uint32_t block_to_left_offset = 0;
  uint32_t block_to_right_offset = 0;
  if (thread_start_block_index < thread_end_block_index && thread_start_block_index > 1) {
    block_to_left_offset = block_to_left_offset_buffer[thread_start_block_index - 1];
    block_to_right_offset = block_to_right_offset_buffer[thread_start_block_index - 1];
  }
  block_to_left_offset = ShufflePrefixSum<uint32_t>(block_to_left_offset, shared_mem_buffer);
  __syncthreads();
  block_to_right_offset = ShufflePrefixSum<uint32_t>(block_to_right_offset, shared_mem_buffer);
  if (threadIdx_x == blockDim_x - 1) {
    to_left_total_count = block_to_left_offset + block_to_left_offset_buffer[num_blocks];
  }
  __syncthreads();
  const uint32_t to_left_thread_block_offset = block_to_left_offset;
  const uint32_t to_right_thread_block_offset = block_to_right_offset + to_left_total_count;
  for (uint32_t block_index = thread_start_block_index; block_index < thread_end_block_index; ++block_index) {
    block_to_left_offset_buffer[block_index] += to_left_thread_block_offset;
    block_to_right_offset_buffer[block_index] += to_right_thread_block_offset;
  }
  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    const data_size_t old_leaf_data_end = cuda_leaf_data_end[left_leaf_index];
    cuda_leaf_data_end[left_leaf_index] = cuda_leaf_data_start[left_leaf_index] + static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_num_data[left_leaf_index] = static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_data_start[right_leaf_index] = cuda_leaf_data_end[left_leaf_index];
    cuda_leaf_data_end[right_leaf_index] = old_leaf_data_end;
    cuda_leaf_num_data[right_leaf_index] = num_data_in_leaf - static_cast<data_size_t>(to_left_total_count);
  }
}

__global__ void AggregateBlockOffsetKernel1(
  const int left_leaf_index,
  const int right_leaf_index,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  const data_size_t num_blocks) {
  __shared__ uint32_t shared_mem_buffer[32];
  __shared__ uint32_t to_left_total_count;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[left_leaf_index];
  const unsigned int threadIdx_x = threadIdx.x;
  uint32_t block_to_left_offset = 0;
  uint32_t block_to_right_offset = 0;
  if (threadIdx_x < static_cast<unsigned int>(num_blocks)) {
    block_to_left_offset = block_to_left_offset_buffer[threadIdx_x + 1];
    block_to_right_offset = block_to_right_offset_buffer[threadIdx_x + 1];
  }
  block_to_left_offset = ShufflePrefixSum<uint32_t>(block_to_left_offset, shared_mem_buffer);
  __syncthreads();
  block_to_right_offset = ShufflePrefixSum<uint32_t>(block_to_right_offset, shared_mem_buffer);
  if (threadIdx.x == blockDim.x - 1) {
    to_left_total_count = block_to_left_offset;
  }
  __syncthreads();
  if (threadIdx_x < static_cast<unsigned int>(num_blocks)) {
    block_to_left_offset_buffer[threadIdx_x + 1] = block_to_left_offset;
    block_to_right_offset_buffer[threadIdx_x + 1] = block_to_right_offset + to_left_total_count;
  }
  if (threadIdx_x == 0) {
    block_to_right_offset_buffer[0] = to_left_total_count;
  }
  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    const data_size_t old_leaf_data_end = cuda_leaf_data_end[left_leaf_index];
    cuda_leaf_data_end[left_leaf_index] = cuda_leaf_data_start[left_leaf_index] + static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_num_data[left_leaf_index] = static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_data_start[right_leaf_index] = cuda_leaf_data_end[left_leaf_index];
    cuda_leaf_data_end[right_leaf_index] = old_leaf_data_end;
    cuda_leaf_num_data[right_leaf_index] = num_data_in_leaf - static_cast<data_size_t>(to_left_total_count);
  }
}

template <typename BIN_TYPE>
__global__ void GenDataToLeftBitVectorKernelLaunch(
  // data
  const BIN_TYPE* column_data,
  const data_size_t* cuda_data_indices,
  data_size_t* cuda_leaf_data_start,
  const uint8_t* cuda_feature_missing_is_zero,
  const uint8_t* cuda_feature_missing_is_na,
  const uint8_t* cuda_feature_mfb_is_zero,
  const uint8_t* cuda_feature_mfb_is_na,
  const uint32_t* cuda_feature_default_bin,
  const uint32_t* cuda_feature_most_freq_bin,
  const uint32_t* cuda_feature_max_bin,
  const uint32_t* cuda_feature_min_bin,
  // input best split info
  const CUDASplitInfo* best_split_info,
  const int right_leaf_index,
  const uint32_t* categorical_bitset,
  const int categorical_bitset_len,
  // for leaf information update
  CUDALeafSplitsStruct* smaller_leaf_splits,
  CUDALeafSplitsStruct* larger_leaf_splits,
  // members of CUDADataPartition
  uint16_t* block_to_left_offset,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer,
  int* cuda_data_index_to_leaf_index,
  data_size_t* cuda_leaf_data_end,
  data_size_t* cuda_leaf_num_data) {
  const data_size_t num_data_in_leaf = best_split_info->left_count + best_split_info->right_count;
  const int left_leaf_index = best_split_info->leaf_index;
  const data_size_t data_start = cuda_leaf_data_start[left_leaf_index];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + data_start;
  const int inner_feature_index = best_split_info->inner_feature_index;
  const bool missing_is_zero = static_cast<bool>(cuda_feature_missing_is_zero[inner_feature_index]);
  const bool missing_is_na = static_cast<bool>(cuda_feature_missing_is_na[inner_feature_index]);
  const bool mfb_is_zero = static_cast<bool>(cuda_feature_mfb_is_zero[inner_feature_index]);
  const bool mfb_is_na = static_cast<bool>(cuda_feature_mfb_is_na[inner_feature_index]);
  const uint32_t default_bin = cuda_feature_default_bin[inner_feature_index];
  const uint32_t most_freq_bin = cuda_feature_most_freq_bin[inner_feature_index];
  const uint32_t max_bin = cuda_feature_max_bin[inner_feature_index];
  const uint32_t min_bin = cuda_feature_min_bin[inner_feature_index];
  uint32_t th = best_split_info->threshold + min_bin;
  uint32_t t_zero_bin = min_bin + default_bin;
  if (most_freq_bin == 0) {
    --th;
    --t_zero_bin;
  }
  uint8_t split_default_to_left = 0;
  uint8_t split_missing_default_to_left = 0;
  int default_leaf_index = right_leaf_index;
  int missing_default_leaf_index = right_leaf_index;
  if (most_freq_bin <= best_split_info->threshold) {
    split_default_to_left = 1;
    default_leaf_index = left_leaf_index;
  }
  if (missing_is_zero || missing_is_na) {
    if (best_split_info->default_left) {
      split_missing_default_to_left = 1;
      missing_default_leaf_index = left_leaf_index;
    }
  }
  const bool max_bin_to_left = (max_bin <= th);
  const bool min_is_max = (min_bin == max_bin);

  cudaStream_t gen_bit_stream;
  cudaStream_t update_leaf_index_stream;
  cudaStreamCreateWithFlags(&gen_bit_stream, cudaStreamNonBlocking);
  cudaStreamCreateWithFlags(&update_leaf_index_stream, cudaStreamNonBlocking);

  int grid_dim = 0;
  int block_dim = 0;
  CUDADataPartition::CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  if (!min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, false, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, false, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (!min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<false, true, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<false, true, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && !missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, false, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, false, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, false, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, false, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, false, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, false, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, false, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, true, false, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, true, false, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, true, true, false, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  } else if (min_is_max && missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    GenDataToLeftBitVectorKernel<true, true, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, gen_bit_stream>>>(
        GenBitVector_ARGS,
        block_to_left_offset,
        block_to_left_offset_buffer,
        block_to_right_offset_buffer);
    UpdateDataIndexToLeafIndexKernel<true, true, true, true, true, true, BIN_TYPE>
      <<<grid_dim, block_dim, 0, update_leaf_index_stream>>>(
        UpdateDataIndexToLeafIndex_ARGS,
        cuda_data_index_to_leaf_index);
  }

  int num_blocks_final_ref = grid_dim - 1;
  int num_blocks_final_aligned = 1;
  while (num_blocks_final_ref > 0) {
    num_blocks_final_aligned <<= 1;
    num_blocks_final_ref >>= 1;
  }
  if (grid_dim > AGGREGATE_BLOCK_SIZE_DATA_PARTITION) {
    AggregateBlockOffsetKernel0<<<1, AGGREGATE_BLOCK_SIZE_DATA_PARTITION, 0, gen_bit_stream>>>(
      left_leaf_index,
      right_leaf_index,
      block_to_left_offset_buffer,
      block_to_right_offset_buffer, cuda_leaf_data_start, cuda_leaf_data_end,
      cuda_leaf_num_data, cuda_data_indices,
      grid_dim);
  } else {
    AggregateBlockOffsetKernel1<<<1, num_blocks_final_aligned, 0, gen_bit_stream>>>(
      left_leaf_index,
      right_leaf_index,
      block_to_left_offset_buffer,
      block_to_right_offset_buffer, cuda_leaf_data_start, cuda_leaf_data_end,
      cuda_leaf_num_data, cuda_data_indices,
      grid_dim);
  }

  cudaStreamDestroy(gen_bit_stream);
  cudaStreamDestroy(update_leaf_index_stream);
}

__global__ void GenDataToLeftBitVectorKernelLaunchOuter(
  // data
  const int* cuda_feature_to_column,
  const uint8_t* cuda_column_bit_type,
  void* const* cuda_data_by_column,
  const data_size_t* cuda_data_indices,
  data_size_t* cuda_leaf_data_start,
  const uint8_t* cuda_feature_missing_is_zero,
  const uint8_t* cuda_feature_missing_is_na,
  const uint8_t* cuda_feature_mfb_is_zero,
  const uint8_t* cuda_feature_mfb_is_na,
  const uint32_t* cuda_feature_default_bin,
  const uint32_t* cuda_feature_most_freq_bin,
  const uint32_t* cuda_feature_max_bin,
  const uint32_t* cuda_feature_min_bin,
  // input best split info
  CUDASplitInfo* const* best_split_info_pointer,
  const int right_leaf_index,
  const uint32_t* categorical_bitset,
  const int categorical_bitset_len,
  // for leaf information update
  CUDALeafSplitsStruct* smaller_leaf_splits,
  CUDALeafSplitsStruct* larger_leaf_splits,
  // members of CUDADataPartition
  uint16_t* block_to_left_offset,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer,
  int* cuda_data_index_to_leaf_index,
  data_size_t* cuda_leaf_data_end,
  data_size_t* cuda_leaf_num_data) {
  const CUDASplitInfo* best_split_info = *best_split_info_pointer;
  const int inner_feature_index = best_split_info->inner_feature_index;
  const int column_index = cuda_feature_to_column[inner_feature_index];
  const uint8_t bit_type = cuda_column_bit_type[column_index];
  const void* column_data_pointer = cuda_data_by_column[column_index];
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  if (bit_type == 8) {
    const uint8_t* column_data = reinterpret_cast<const uint8_t*>(column_data_pointer);
    GenDataToLeftBitVectorKernelLaunch<uint8_t><<<1, 1, 0, cuda_stream>>>(
      column_data,
      cuda_data_indices,
      cuda_leaf_data_start,
      cuda_feature_missing_is_zero,
      cuda_feature_missing_is_na,
      cuda_feature_mfb_is_zero,
      cuda_feature_mfb_is_na,
      cuda_feature_default_bin,
      cuda_feature_most_freq_bin,
      cuda_feature_max_bin,
      cuda_feature_min_bin,
      best_split_info,
      right_leaf_index,
      categorical_bitset,
      categorical_bitset_len,
      smaller_leaf_splits,
      larger_leaf_splits,
      block_to_left_offset,
      block_to_left_offset_buffer,
      block_to_right_offset_buffer,
      cuda_data_index_to_leaf_index,
      cuda_leaf_data_end,
      cuda_leaf_num_data);
  } else if (bit_type == 16) {
    const uint16_t* column_data = reinterpret_cast<const uint16_t*>(column_data_pointer);
    GenDataToLeftBitVectorKernelLaunch<uint16_t><<<1, 1, 0, cuda_stream>>>(
      column_data,
      cuda_data_indices,
      cuda_leaf_data_start,
      cuda_feature_missing_is_zero,
      cuda_feature_missing_is_na,
      cuda_feature_mfb_is_zero,
      cuda_feature_mfb_is_na,
      cuda_feature_default_bin,
      cuda_feature_most_freq_bin,
      cuda_feature_max_bin,
      cuda_feature_min_bin,
      best_split_info,
      right_leaf_index,
      categorical_bitset,
      categorical_bitset_len,
      smaller_leaf_splits,
      larger_leaf_splits,
      block_to_left_offset,
      block_to_left_offset_buffer,
      block_to_right_offset_buffer,
      cuda_data_index_to_leaf_index,
      cuda_leaf_data_end,
      cuda_leaf_num_data);
  } else if (bit_type == 32) {
    const uint32_t* column_data = reinterpret_cast<const uint32_t*>(column_data_pointer);
    GenDataToLeftBitVectorKernelLaunch<uint32_t><<<1, 1, 0, cuda_stream>>>(
      column_data,
      cuda_data_indices,
      cuda_leaf_data_start,
      cuda_feature_missing_is_zero,
      cuda_feature_missing_is_na,
      cuda_feature_mfb_is_zero,
      cuda_feature_mfb_is_na,
      cuda_feature_default_bin,
      cuda_feature_most_freq_bin,
      cuda_feature_max_bin,
      cuda_feature_min_bin,
      best_split_info,
      right_leaf_index,
      categorical_bitset,
      categorical_bitset_len,
      smaller_leaf_splits,
      larger_leaf_splits,
      block_to_left_offset,
      block_to_left_offset_buffer,
      block_to_right_offset_buffer,
      cuda_data_index_to_leaf_index,
      cuda_leaf_data_end,
      cuda_leaf_num_data);
  }
  cudaStreamDestroy(cuda_stream);
}

void CUDADataPartition::LaunchGenDataToLeftBitVectorKernel(
  // input best split info
  CUDASplitInfo* const* best_split_info,
  const int right_leaf_index,
  const uint32_t* categorical_bitset,
  const int categorical_bitset_len,
  // for leaf information update
  CUDALeafSplitsStruct* smaller_leaf_splits,
  CUDALeafSplitsStruct* larger_leaf_splits) {
  GenDataToLeftBitVectorKernelLaunchOuter<<<1, 1, 0, cuda_streams_[0]>>>(
    cuda_column_data_->cuda_feature_to_column(),
    cuda_column_data_->cuda_column_bit_type(),
    cuda_column_data_->cuda_data_by_column(),
    cuda_data_indices_,
    cuda_leaf_data_start_,
    cuda_column_data_->cuda_feature_missing_is_zero(),
    cuda_column_data_->cuda_feature_missing_is_na(),
    cuda_column_data_->cuda_feature_mfb_is_zero(),
    cuda_column_data_->cuda_feature_mfb_is_na(),
    cuda_column_data_->cuda_feature_default_bin(),
    cuda_column_data_->cuda_feature_most_freq_bin(),
    cuda_column_data_->cuda_feature_max_bin(),
    cuda_column_data_->cuda_feature_min_bin(),
    best_split_info,
    right_leaf_index,
    categorical_bitset,
    categorical_bitset_len,
    smaller_leaf_splits,
    larger_leaf_splits,
    cuda_block_to_left_offset_,
    cuda_block_data_to_left_offset_,
    cuda_block_data_to_right_offset_,
    cuda_data_index_to_leaf_index_,
    cuda_leaf_data_end_,
    cuda_leaf_num_data_);
  SynchronizeCUDADevice(__FILE__, __LINE__);
}

#undef UpdateDataIndexToLeafIndexKernel_PARAMS
#undef UpdateDataIndexToLeafIndex_ARGS
#undef GenDataToLeftBitVectorKernel_PARMS
#undef GenBitVector_ARGS

template <typename BIN_TYPE, bool USE_MIN_BIN>
__global__ void UpdateDataIndexToLeafIndexKernel_Categorical(
  const data_size_t num_data_in_leaf, const data_size_t* data_indices_in_leaf,
  const uint32_t* bitset, const int bitset_len, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t max_bin, const uint32_t min_bin, const int8_t mfb_offset,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index) {
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (USE_MIN_BIN && (bin < min_bin || bin > max_bin)) {
      cuda_data_index_to_leaf_index[global_data_index] = default_leaf_index;
    } else if (!USE_MIN_BIN && bin == 0) {
      cuda_data_index_to_leaf_index[global_data_index] = default_leaf_index;
    } else if (CUDAFindInBitset(bitset, bitset_len, bin - min_bin + mfb_offset)) {
      cuda_data_index_to_leaf_index[global_data_index] = left_leaf_index;
    } else {
      cuda_data_index_to_leaf_index[global_data_index] = right_leaf_index;
    }
  }
}

// for categorical features
template <typename BIN_TYPE, bool USE_MIN_BIN>
__global__ void GenDataToLeftBitVectorKernel_Categorical(
  const data_size_t num_data_in_leaf, const data_size_t* data_indices_in_leaf,
  const uint32_t* bitset, int bitset_len, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t max_bin, const uint32_t min_bin, const int8_t mfb_offset,
  const uint8_t split_default_to_left,
  uint16_t* block_to_left_offset,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer) {
  __shared__ uint16_t shared_mem_buffer[32];
  uint16_t thread_to_left_offset_cnt = 0;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (USE_MIN_BIN && (bin < min_bin || bin > max_bin)) {
      thread_to_left_offset_cnt = split_default_to_left;
    } else if (!USE_MIN_BIN && bin == 0) {
      thread_to_left_offset_cnt = split_default_to_left;
    } else if (CUDAFindInBitset(bitset, bitset_len, bin - min_bin + mfb_offset)) {
      thread_to_left_offset_cnt = 1;
    }
  }
  __syncthreads();
  PrepareOffset(num_data_in_leaf, block_to_left_offset + blockIdx.x * blockDim.x, block_to_left_offset_buffer, block_to_right_offset_buffer,
    thread_to_left_offset_cnt, shared_mem_buffer);
}

#define GenBitVector_Categorical_ARGS \
  num_data_in_leaf, data_indices_in_leaf, \
  bitset, bitset_len, \
  column_data, max_bin, min_bin, mfb_offset, split_default_to_left, \
  cuda_block_to_left_offset_, cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_

#define UpdateDataIndexToLeafIndex_Categorical_ARGS \
  num_data_in_leaf, data_indices_in_leaf, \
  bitset, bitset_len, \
  column_data, max_bin, min_bin, mfb_offset,  \
  cuda_data_index_to_leaf_index_, left_leaf_index, right_leaf_index, default_leaf_index

void CUDADataPartition::LaunchGenDataToLeftBitVectorCategoricalKernel(
  const data_size_t num_data_in_leaf,
  const int split_feature_index,
  const uint32_t* bitset,
  const int bitset_len,
  const uint8_t split_default_left,
  const data_size_t leaf_data_start,
  const int left_leaf_index,
  const int right_leaf_index) {
  const data_size_t* data_indices_in_leaf = cuda_data_indices_ + leaf_data_start;
  const int column_index = cuda_column_data_->feature_to_column(split_feature_index);
  const uint8_t bit_type = cuda_column_data_->column_bit_type(column_index);
  const bool is_single_feature_in_column = is_single_feature_in_column_[split_feature_index];
  const uint32_t min_bin = is_single_feature_in_column ? 1 : cuda_column_data_->feature_min_bin(split_feature_index);
  const uint32_t max_bin = cuda_column_data_->feature_max_bin(split_feature_index);
  const uint32_t most_freq_bin = cuda_column_data_->feature_most_freq_bin(split_feature_index);
  const uint32_t default_bin = cuda_column_data_->feature_default_bin(split_feature_index);
  const void* column_data_pointer = cuda_column_data_->GetColumnData(column_index);
  const int8_t mfb_offset = static_cast<int8_t>(most_freq_bin == 0);
  std::vector<uint32_t> host_bitset(bitset_len, 0);
  CopyFromCUDADeviceToHost<uint32_t>(host_bitset.data(), bitset, bitset_len, __FILE__, __LINE__);
  uint8_t split_default_to_left = 0;
  int default_leaf_index = right_leaf_index;
  if (most_freq_bin > 0 && Common::FindInBitset(host_bitset.data(), bitset_len, most_freq_bin)) {
    split_default_to_left = 1;
    default_leaf_index = left_leaf_index;
  }
  if (bit_type == 8) {
    const uint8_t* column_data = reinterpret_cast<const uint8_t*>(column_data_pointer);
    if (is_single_feature_in_column) {
      GenDataToLeftBitVectorKernel_Categorical<uint8_t, false><<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_Categorical_ARGS);
      UpdateDataIndexToLeafIndexKernel_Categorical<uint8_t, false><<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(UpdateDataIndexToLeafIndex_Categorical_ARGS);
    } else {
      GenDataToLeftBitVectorKernel_Categorical<uint8_t, true><<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_Categorical_ARGS);
      UpdateDataIndexToLeafIndexKernel_Categorical<uint8_t, true><<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(UpdateDataIndexToLeafIndex_Categorical_ARGS);
    }
  } else if (bit_type == 16) {
    const uint16_t* column_data = reinterpret_cast<const uint16_t*>(column_data_pointer);
    if (is_single_feature_in_column) {
      GenDataToLeftBitVectorKernel_Categorical<uint16_t, false><<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_Categorical_ARGS);
      UpdateDataIndexToLeafIndexKernel_Categorical<uint16_t, false><<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(UpdateDataIndexToLeafIndex_Categorical_ARGS);
    } else {
      GenDataToLeftBitVectorKernel_Categorical<uint16_t, true><<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_Categorical_ARGS);
      UpdateDataIndexToLeafIndexKernel_Categorical<uint16_t, true><<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(UpdateDataIndexToLeafIndex_Categorical_ARGS);
    }
  } else if (bit_type == 32) {
    const uint32_t* column_data = reinterpret_cast<const uint32_t*>(column_data_pointer);
    if (is_single_feature_in_column) {
      GenDataToLeftBitVectorKernel_Categorical<uint32_t, false><<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_Categorical_ARGS);
      UpdateDataIndexToLeafIndexKernel_Categorical<uint32_t, false><<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(UpdateDataIndexToLeafIndex_Categorical_ARGS);
    } else {
      GenDataToLeftBitVectorKernel_Categorical<uint32_t, true><<<grid_dim_, block_dim_, 0, cuda_streams_[0]>>>(GenBitVector_Categorical_ARGS);
      UpdateDataIndexToLeafIndexKernel_Categorical<uint32_t, true><<<grid_dim_, block_dim_, 0, cuda_streams_[3]>>>(UpdateDataIndexToLeafIndex_Categorical_ARGS);
    }
  }
}

#undef GenBitVector_Categorical_ARGS
#undef UpdateDataIndexToLeafIndex_Categorical_ARGS


__global__ void SplitTreeStructureKernel(const int left_leaf_index,
  const int right_leaf_index,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  const CUDASplitInfo* best_split_info,
  // for leaf splits information update
  CUDALeafSplitsStruct* smaller_leaf_splits,
  CUDALeafSplitsStruct* larger_leaf_splits,
  const int num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool,
  double* cuda_leaf_output) {
  const unsigned int to_left_total_cnt = cuda_leaf_num_data[left_leaf_index];
  const unsigned int global_thread_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_thread_index == 0) {
    cuda_leaf_output[left_leaf_index] = best_split_info->left_value;
  } else if (global_thread_index == 1) {
    cuda_leaf_output[right_leaf_index] = best_split_info->right_value;
  }

  if (cuda_leaf_num_data[left_leaf_index] < cuda_leaf_num_data[right_leaf_index]) {
    if (global_thread_index == 0) {
      hist_t* parent_hist_ptr = cuda_hist_pool[left_leaf_index];
      cuda_hist_pool[right_leaf_index] = parent_hist_ptr;
      cuda_hist_pool[left_leaf_index] = cuda_hist + 2 * right_leaf_index * num_total_bin;
      smaller_leaf_splits->hist_in_leaf = cuda_hist_pool[left_leaf_index];
      larger_leaf_splits->hist_in_leaf = cuda_hist_pool[right_leaf_index];
    } else if (global_thread_index == 1) {
      smaller_leaf_splits->sum_of_gradients = best_split_info->left_sum_gradients;
    } else if (global_thread_index == 2) {
      smaller_leaf_splits->sum_of_hessians = best_split_info->left_sum_hessians;
    } else if (global_thread_index == 3) {
      smaller_leaf_splits->num_data_in_leaf = to_left_total_cnt;
    } else if (global_thread_index == 4) {
      smaller_leaf_splits->gain = best_split_info->left_gain;
    } else if (global_thread_index == 5) {
      smaller_leaf_splits->leaf_value = best_split_info->left_value;
    } else if (global_thread_index == 6) {
      smaller_leaf_splits->data_indices_in_leaf = cuda_data_indices;
    } else if (global_thread_index == 7) {
      larger_leaf_splits->leaf_index = right_leaf_index;
    } else if (global_thread_index == 8) {
      larger_leaf_splits->sum_of_gradients = best_split_info->right_sum_gradients;
    } else if (global_thread_index == 9) {
      larger_leaf_splits->sum_of_hessians = best_split_info->right_sum_hessians;
    } else if (global_thread_index == 10) {
      larger_leaf_splits->num_data_in_leaf = cuda_leaf_num_data[right_leaf_index];
    } else if (global_thread_index == 11) {
      larger_leaf_splits->gain = best_split_info->right_gain;
    } else if (global_thread_index == 12) {
      larger_leaf_splits->leaf_value = best_split_info->right_value;
    } else if (global_thread_index == 13) {
      larger_leaf_splits->data_indices_in_leaf = cuda_data_indices + cuda_leaf_num_data[left_leaf_index];
    } else if (global_thread_index == 14) {
      smaller_leaf_splits->leaf_index = left_leaf_index;
    }
  } else {
    if (global_thread_index == 0) {
      larger_leaf_splits->leaf_index = left_leaf_index;
    } else if (global_thread_index == 1) {
      larger_leaf_splits->sum_of_gradients = best_split_info->left_sum_gradients;
    } else if (global_thread_index == 2) {
      larger_leaf_splits->sum_of_hessians = best_split_info->left_sum_hessians;
    } else if (global_thread_index == 3) {
      larger_leaf_splits->num_data_in_leaf = to_left_total_cnt;
    } else if (global_thread_index == 4) {
      larger_leaf_splits->gain = best_split_info->left_gain;
    } else if (global_thread_index == 5) {
      larger_leaf_splits->leaf_value = best_split_info->left_value;
    } else if (global_thread_index == 6) {
      larger_leaf_splits->data_indices_in_leaf = cuda_data_indices;
    } else if (global_thread_index == 7) {
      smaller_leaf_splits->leaf_index = right_leaf_index;
    } else if (global_thread_index == 8) {
      smaller_leaf_splits->sum_of_gradients = best_split_info->right_sum_gradients;
    } else if (global_thread_index == 9) {
      smaller_leaf_splits->sum_of_hessians = best_split_info->right_sum_hessians;
    } else if (global_thread_index == 10) {
      smaller_leaf_splits->num_data_in_leaf = cuda_leaf_num_data[right_leaf_index];
    } else if (global_thread_index == 11) {
      smaller_leaf_splits->gain = best_split_info->right_gain;
    } else if (global_thread_index == 12) {
      smaller_leaf_splits->leaf_value = best_split_info->right_value;
    } else if (global_thread_index == 13) {
      smaller_leaf_splits->data_indices_in_leaf = cuda_data_indices + cuda_leaf_num_data[left_leaf_index];
    } else if (global_thread_index == 14) {
      cuda_hist_pool[right_leaf_index] = cuda_hist + 2 * right_leaf_index * num_total_bin;
      smaller_leaf_splits->hist_in_leaf = cuda_hist_pool[right_leaf_index];
    } else if (global_thread_index == 15) {
      larger_leaf_splits->hist_in_leaf = cuda_hist_pool[left_leaf_index];
    }
  }
}

__global__ void SplitTreeStructureKernelLaunch(
  CUDASplitInfo* const* best_split_info_pointer,
  const int right_leaf_index,
  data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  // for leaf splits information update
  CUDALeafSplitsStruct* smaller_leaf_splits,
  CUDALeafSplitsStruct* larger_leaf_splits,
  const int num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool,
  double* cuda_leaf_output) {
  const CUDASplitInfo* best_split_info = *best_split_info_pointer;
  const int left_leaf_index = best_split_info->leaf_index;
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  SplitTreeStructureKernel<<<4, 4, 0, cuda_stream>>>(left_leaf_index, right_leaf_index,
    block_to_left_offset_buffer,
    block_to_right_offset_buffer, cuda_leaf_data_start, cuda_leaf_data_end,
    cuda_leaf_num_data, cuda_data_indices,
    best_split_info,
    smaller_leaf_splits,
    larger_leaf_splits,
    num_total_bin,
    cuda_hist,
    cuda_hist_pool,
    cuda_leaf_output);
  cudaStreamDestroy(cuda_stream);
}

__global__ void SplitInnerKernel(const int left_leaf_index, const int right_leaf_index,
  const data_size_t* cuda_leaf_data_start, const data_size_t* cuda_leaf_num_data,
  const data_size_t* cuda_data_indices,
  const data_size_t* block_to_left_offset_buffer, const data_size_t* block_to_right_offset_buffer,
  const uint16_t* block_to_left_offset, data_size_t* out_data_indices_in_leaf) {
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[left_leaf_index];
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[left_leaf_index] + cuda_leaf_num_data[right_leaf_index];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int global_thread_index = blockIdx.x * blockDim_x + threadIdx_x;
  const data_size_t* cuda_data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  const uint16_t* block_to_left_offset_ptr = block_to_left_offset + blockIdx.x * blockDim_x;
  const uint32_t to_right_block_offset = block_to_right_offset_buffer[blockIdx.x];
  const uint32_t to_left_block_offset = block_to_left_offset_buffer[blockIdx.x];
  data_size_t* left_out_data_indices_in_leaf = out_data_indices_in_leaf + to_left_block_offset;
  data_size_t* right_out_data_indices_in_leaf = out_data_indices_in_leaf + to_right_block_offset;
  if (static_cast<data_size_t>(global_thread_index) < num_data_in_leaf) {
    const uint32_t thread_to_left_offset = (threadIdx_x == 0 ? 0 : block_to_left_offset_ptr[threadIdx_x - 1]);
    const bool to_left = block_to_left_offset_ptr[threadIdx_x] > thread_to_left_offset;
    if (to_left) {
      left_out_data_indices_in_leaf[thread_to_left_offset] = cuda_data_indices_in_leaf[global_thread_index];
    } else {
      const uint32_t thread_to_right_offset = threadIdx.x - thread_to_left_offset;
      right_out_data_indices_in_leaf[thread_to_right_offset] = cuda_data_indices_in_leaf[global_thread_index];
    }
  }
}

__global__ void SplitInnerKernelLaunch(
  CUDASplitInfo* const* best_split_info_pointer,
  const int right_leaf_index,
  const data_size_t* cuda_leaf_data_start, const data_size_t* cuda_leaf_num_data,
  const data_size_t* cuda_data_indices,
  const data_size_t* block_to_left_offset_buffer, const data_size_t* block_to_right_offset_buffer,
  const uint16_t* block_to_left_offset, data_size_t* out_data_indices_in_leaf) {
  const CUDASplitInfo* best_split_info = *best_split_info_pointer;
  const int left_leaf_index = best_split_info->leaf_index;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[left_leaf_index] + cuda_leaf_num_data[right_leaf_index];
  int grid_dim = 0;
  int block_dim = 0;
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  CUDADataPartition::CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  SplitInnerKernel<<<grid_dim, block_dim, 0, cuda_stream>>>(
    left_leaf_index,
    right_leaf_index,
    cuda_leaf_data_start,
    cuda_leaf_num_data,
    cuda_data_indices,
    block_to_left_offset_buffer,
    block_to_right_offset_buffer,
    block_to_left_offset,
    out_data_indices_in_leaf);
  cudaStreamDestroy(cuda_stream);
}

__global__ void CopyDataIndicesKernel(
  const data_size_t num_data_in_leaf,
  const data_size_t* out_data_indices_in_leaf,
  data_size_t* cuda_data_indices) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int global_thread_index = blockIdx.x * blockDim.x + threadIdx_x;
  if (global_thread_index < num_data_in_leaf) {
    cuda_data_indices[global_thread_index] = out_data_indices_in_leaf[global_thread_index];
  }
}

__global__ void CopyDataIndicesKernelLaunch(
  CUDASplitInfo* const* best_split_info_pointer,
  const data_size_t* cuda_leaf_data_start,
  const data_size_t* out_data_indices_in_leaf,
  data_size_t* cuda_data_indices) {
  const CUDASplitInfo* best_split_info = *best_split_info_pointer;
  const int left_leaf_index = best_split_info->leaf_index;
  const data_size_t num_data_in_leaf = best_split_info->left_count + best_split_info->right_count;
  int grid_dim = 0;
  int block_dim = 0;
  cudaStream_t cuda_stream;
  cudaStreamCreateWithFlags(&cuda_stream, cudaStreamNonBlocking);
  CUDADataPartition::CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  CopyDataIndicesKernel<<<grid_dim, block_dim, 0, cuda_stream>>>(
    num_data_in_leaf,
    out_data_indices_in_leaf,
    cuda_data_indices + cuda_leaf_data_start[left_leaf_index]);
  cudaStreamDestroy(cuda_stream);
}

void CUDADataPartition::LaunchSplitInnerKernel(
  // input best split info
  CUDASplitInfo* const* best_split_info,
  const int right_leaf_index,
  const uint32_t* categorical_bitset,
  const int categorical_bitset_len,
  // for leaf information update
  CUDALeafSplitsStruct* smaller_leaf_splits,
  CUDALeafSplitsStruct* larger_leaf_splits) {
  global_timer.Start("CUDADataPartition::SplitInnerKernel");
  SplitInnerKernelLaunch<<<1, 1, 0, cuda_streams_[1]>>>(
    best_split_info, right_leaf_index, cuda_leaf_data_start_, cuda_leaf_num_data_, cuda_data_indices_,
    cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_, cuda_block_to_left_offset_,
    cuda_out_data_indices_in_leaf_);
  global_timer.Stop("CUDADataPartition::SplitInnerKernel");

  // TODO: can remove this synchronization ?
  SynchronizeCUDADevice(__FILE__, __LINE__);

  global_timer.Start("CUDADataPartition::SplitTreeStructureKernel");
  SplitTreeStructureKernelLaunch<<<1, 1, 0, cuda_streams_[0]>>>(best_split_info, right_leaf_index,
    cuda_block_data_to_left_offset_,
    cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
    cuda_leaf_num_data_, cuda_out_data_indices_in_leaf_,
    smaller_leaf_splits,
    larger_leaf_splits,
    num_total_bin_,
    cuda_hist_,
    cuda_hist_pool_,
    cuda_leaf_output_);
  global_timer.Stop("CUDADataPartition::SplitTreeStructureKernel");
  SynchronizeCUDADevice(__FILE__, __LINE__);
  global_timer.Start("CUDADataPartition::CopyDataIndicesKernel");
  CopyDataIndicesKernelLaunch<<<1, 1, 0, cuda_streams_[2]>>>(
    best_split_info,
    cuda_leaf_data_start_,
    cuda_out_data_indices_in_leaf_,
    cuda_data_indices_);
  global_timer.Stop("CUDADataPartition::CopyDataIndicesKernel");
}

template <bool USE_BAGGING>
__global__ void AddPredictionToScoreKernel(
  const data_size_t* used_data_indices,
  const double* leaf_value, double* cuda_scores,
  const int* cuda_data_index_to_leaf_index, const data_size_t num_data) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockIdx_x = blockIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const data_size_t local_data_index = static_cast<data_size_t>(blockIdx_x * blockDim_x + threadIdx_x);
  if (local_data_index < num_data) {
    if (USE_BAGGING) {
      const data_size_t global_data_index = used_data_indices[local_data_index];
      const int leaf_index = cuda_data_index_to_leaf_index[global_data_index];
      const double leaf_prediction_value = leaf_value[leaf_index];
      cuda_scores[global_data_index] += leaf_prediction_value;
    } else {
      const int leaf_index = cuda_data_index_to_leaf_index[local_data_index];
      const double leaf_prediction_value = leaf_value[leaf_index];
      cuda_scores[local_data_index] += leaf_prediction_value;
    }
  }
}

void CUDADataPartition::LaunchAddPredictionToScoreKernel(const double* leaf_value, double* cuda_scores) {
  global_timer.Start("CUDADataPartition::AddPredictionToScoreKernel");
  const data_size_t num_data_in_root = root_num_data();
  const int num_blocks = (num_data_in_root + FILL_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / FILL_INDICES_BLOCK_SIZE_DATA_PARTITION;
  if (use_bagging_) {
    AddPredictionToScoreKernel<true><<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(
      cuda_data_indices_, leaf_value, cuda_scores, cuda_data_index_to_leaf_index_, num_data_in_root);
  } else {
    AddPredictionToScoreKernel<false><<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(
      cuda_data_indices_, leaf_value, cuda_scores, cuda_data_index_to_leaf_index_, num_data_in_root);
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
  global_timer.Stop("CUDADataPartition::AddPredictionToScoreKernel");
}

}  // namespace LightGBM

#endif  // USE_CUDA
