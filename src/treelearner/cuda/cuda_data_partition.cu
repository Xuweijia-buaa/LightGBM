
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_data_partition.hpp"
#include <LightGBM/tree.h>

namespace LightGBM {

#define CONFLICT_FREE_INDEX(n) \
  ((n) + ((n) >> LOG_NUM_BANKS_DATA_PARTITION)) \

__device__ void PrefixSum(uint32_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX(n - 1);
  const uint32_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
      const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
      const uint32_t src_val = elements[conflict_free_src_pos];
      elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
      elements[conflict_free_dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__device__ void PrefixSum_1024(uint32_t* elements, unsigned int n) {
  //unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX(n - 1);
  const uint32_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();

  if (threadIdx_x < 512) {
    const unsigned int src_pos = (2 * threadIdx_x + 1) - 1;
    const unsigned int dst_pos = (2 * threadIdx_x + 2) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();

  if (threadIdx_x < 256) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 1) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 1) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();
  
  if (threadIdx_x < 128) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 2) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 2) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();
  
  if (threadIdx_x < 64) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 3) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 3) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();
  
  if (threadIdx_x < 32) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 4) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 4) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();
  
  if (threadIdx_x < 16) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 5) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 5) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();

  if (threadIdx_x < 8) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 6) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 6) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();

  if (threadIdx_x < 4) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 7) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 7) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();

  if (threadIdx_x < 2) {
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 8) - 1;
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 8) - 1;
    elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
  }
  __syncthreads();

  if (threadIdx_x == 0) {
    //const unsigned int src_pos = 511;
    //const unsigned int dst_pos = 1023;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(1023);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(511);
    //elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
    elements[conflict_free_dst_pos] += elements[conflict_free_src_pos];
  //}
  //__syncthreads();

  /*for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }*/
  //if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  //}
  //__syncthreads();

  //if (threadIdx_x == 0) {
    //const unsigned int dst_pos = 1023;
    //const unsigned int src_pos = 511;
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 2) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 8) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 8) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 4) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 7) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 7) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 8) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 6) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 6) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 16) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 5) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 5) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 32) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 4) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 4) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 64) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 3) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 3) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 128) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 2) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 2) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 256) {
    const unsigned int dst_pos = ((2 * threadIdx_x + 2) << 1) - 1;
    const unsigned int src_pos = ((2 * threadIdx_x + 1) << 1) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  if (threadIdx_x < 512) {
    const unsigned int dst_pos = (2 * threadIdx_x + 2) - 1;
    const unsigned int src_pos = (2 * threadIdx_x + 1) - 1;
    const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
    const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
    const uint32_t src_val = elements[conflict_free_src_pos];
    elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
    elements[conflict_free_dst_pos] += src_val;
  }
  __syncthreads();

  /*for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
      const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
      const uint32_t src_val = elements[conflict_free_src_pos];
      elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
      elements[conflict_free_dst_pos] += src_val;
    }
    __syncthreads();
  }*/
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__device__ void PrefixSum(uint16_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX(n - 1);
  const uint16_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
      const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
      const uint16_t src_val = elements[conflict_free_src_pos];
      elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
      elements[conflict_free_dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__device__ void ReduceSum(uint16_t* array, const size_t size) {
  const unsigned int threadIdx_x = threadIdx.x;
  for (int s = 1; s < size; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < size) {
      array[CONFLICT_FREE_INDEX(threadIdx_x)] += array[CONFLICT_FREE_INDEX(threadIdx_x + s)];
    }
    __syncthreads();
  }
}

__device__ void ReduceSum(double* array, const size_t size) {
  const unsigned int threadIdx_x = threadIdx.x;
  for (int s = 1; s < size; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < size) {
      array[threadIdx_x] += array[threadIdx_x + s];
    }
    __syncthreads();
  }
}

__global__ void FillDataIndicesBeforeTrainKernel(const data_size_t* cuda_num_data,
  data_size_t* data_indices, int* cuda_data_index_to_leaf_index) {
  const data_size_t num_data_ref = *cuda_num_data;
  const unsigned int data_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (data_index < num_data_ref) {
    data_indices[data_index] = data_index;
    cuda_data_index_to_leaf_index[data_index] = 0;
  }
}

void CUDADataPartition::LaunchFillDataIndicesBeforeTrain() {
  const int num_blocks = (num_data_ + FILL_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / FILL_INDICES_BLOCK_SIZE_DATA_PARTITION;
  data_size_t cuda_num_data = 0;
  CopyFromCUDADeviceToHost<data_size_t>(&cuda_num_data, cuda_num_data_, 1);
  Log::Warning("cuda_num_data = %d, num_data_ = %d", cuda_num_data, num_data_);
  FillDataIndicesBeforeTrainKernel<<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(cuda_num_data_, cuda_data_indices_, cuda_data_index_to_leaf_index_);
}

__device__ void PrepareOffset(const data_size_t num_data_in_leaf_ref, const uint8_t* split_to_left_bit_vector,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition,
  uint16_t* thread_to_left_offset_cnt) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  __syncthreads();
  ReduceSum(thread_to_left_offset_cnt, split_indices_block_size_data_partition);
  __syncthreads();
  if (threadIdx_x == 0) {
    const data_size_t num_data_in_block = (blockIdx.x + 1) * blockDim_x <= num_data_in_leaf_ref ? static_cast<data_size_t>(blockDim_x) :
      num_data_in_leaf_ref - static_cast<data_size_t>(blockIdx.x * blockDim_x);
    if (num_data_in_block > 0) {
      const data_size_t data_to_left = static_cast<data_size_t>(thread_to_left_offset_cnt[0]);
      block_to_left_offset_buffer[blockIdx.x + 1] = data_to_left;
      block_to_right_offset_buffer[blockIdx.x + 1] = num_data_in_block - data_to_left;
    } else {
      block_to_left_offset_buffer[blockIdx.x + 1] = 0;
      block_to_right_offset_buffer[blockIdx.x + 1] = 0;
    }
  }
}

template <bool MIN_IS_MAX, bool MAX_TO_LEFT, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, typename BIN_TYPE>
__global__ void UpdateDataIndexToLeafIndexKernel(const data_size_t cuda_leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index) {
  const data_size_t* data_indices_in_leaf = cuda_data_indices + cuda_leaf_data_start;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (!MIN_IS_MAX) {
      if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
        (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
        cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
      } else if (bin < min_bin_ref || bin > max_bin_ref) {
        if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
          cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
        } else {
          cuda_data_index_to_leaf_index[global_data_index] = default_leaf_index;
        }
      } else if (bin > th) {
        cuda_data_index_to_leaf_index[global_data_index] = right_leaf_index;
      }/* else {
        cuda_data_index_to_leaf_index[global_data_index] = left_leaf_index;
      }*/
    } else {
      if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
        cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
      } else if (bin != max_bin_ref) {
        if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
          cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
        } else {
          cuda_data_index_to_leaf_index[global_data_index] = default_leaf_index;
        }
      } else {
        if (MISSING_IS_NA && !MFB_IS_NA) {
          cuda_data_index_to_leaf_index[global_data_index] = missing_default_leaf_index;
        } else {
          if (!MAX_TO_LEFT) {
            /*cuda_data_index_to_leaf_index[global_data_index] = left_leaf_index;
          } else {*/
            cuda_data_index_to_leaf_index[global_data_index] = right_leaf_index;
          }
        }
      }
    }
  }
}

#define UpdateDataIndexToLeafIndex_ARGS leaf_data_start, \
  num_data_in_leaf, cuda_data_indices, th, column_data, \
  t_zero_bin, max_bin_ref, min_bin_ref, cuda_data_index_to_leaf_index, left_leaf_index, right_leaf_index, \
  default_leaf_index, missing_default_leaf_index

template <typename BIN_TYPE>
void CUDADataPartition::LaunchUpdateDataIndexToLeafIndexKernel(const data_size_t leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index,
  const bool missing_is_zero, const bool missing_is_na, const bool mfb_is_zero, const bool mfb_is_na, const bool max_to_left,
  const int num_blocks, const int block_size) {
  if (min_bin_ref < max_bin_ref) {
    if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, false, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, false, true, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, false, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<false, true, true, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    }
  } else {
    if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, false, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, false, true, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, false, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, false, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, false, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, false, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, false, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, true, false, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, true, false, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, true, true, false, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_to_left) {
      UpdateDataIndexToLeafIndexKernel<true, true, true, true, true, true, BIN_TYPE><<<num_blocks, block_size, 0, cuda_streams_[4]>>>(UpdateDataIndexToLeafIndex_ARGS);
    }
  }
}

// min_bin_ref < max_bin_ref
template <typename BIN_TYPE, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA>
__global__ void GenDataToLeftBitVectorKernel0(const int best_split_feature_ref, const data_size_t cuda_leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const int num_features_ref, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t most_freq_bin_ref, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  const uint8_t split_default_to_left, const uint8_t split_missing_default_to_left,
  uint8_t* cuda_data_to_left,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index) {
  __shared__ uint16_t thread_to_left_offset_cnt[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1) / NUM_BANKS_DATA_PARTITION];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + cuda_leaf_data_start;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 1;
    }
  } else {
    thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 0;
  }
  __syncthreads();
  PrepareOffset(num_data_in_leaf, cuda_data_to_left, block_to_left_offset_buffer, block_to_right_offset_buffer,
    split_indices_block_size_data_partition, thread_to_left_offset_cnt);
}

// min_bin_ref < max_bin_ref
template <typename BIN_TYPE, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA>
__global__ void GenDataToLeftBitVectorKernelPacked0(const int best_split_feature_ref, const data_size_t cuda_leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const int num_features_ref, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t most_freq_bin_ref, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  const uint8_t split_default_to_left, const uint8_t split_missing_default_to_left,
  uint8_t* cuda_data_to_left,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index) {
  __shared__ uint16_t thread_to_left_offset_cnt[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION * 4];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + cuda_leaf_data_start;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 1;
    }
  } else {
    thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 0;
  }
  __syncthreads();
  PrepareOffset(num_data_in_leaf, cuda_data_to_left, block_to_left_offset_buffer, block_to_right_offset_buffer,
    split_indices_block_size_data_partition, thread_to_left_offset_cnt);
}

// min_bin_ref == max_bin_ref
template <typename BIN_TYPE, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, bool MAX_TO_LEFT>
__global__ void GenDataToLeftBitVectorKernel16(const int best_split_feature_ref, const data_size_t cuda_leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const int num_features_ref, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t most_freq_bin_ref, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  const uint8_t split_default_to_left, const uint8_t split_missing_default_to_left,
  uint8_t* cuda_data_to_left,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index) {
  __shared__ uint16_t thread_to_left_offset_cnt[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1) / NUM_BANKS_DATA_PARTITION];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + cuda_leaf_data_start;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 0;
        }
      }
    }
  } else {
    thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = 0;
  }
  __syncthreads();
  PrepareOffset(num_data_in_leaf, cuda_data_to_left, block_to_left_offset_buffer, block_to_right_offset_buffer,
    split_indices_block_size_data_partition, thread_to_left_offset_cnt);
}

// min_bin_ref < max_bin_ref
template <typename BIN_TYPE, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA>
__global__ void GenDataToLeftBitVectorKernel0_2(const int best_split_feature_ref, const data_size_t cuda_leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const int num_features_ref, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t most_freq_bin_ref, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  const uint8_t split_default_to_left, const uint8_t split_missing_default_to_left,
  uint8_t* cuda_data_to_left,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index) {
  __shared__ uint16_t thread_to_left_offset_cnt[(SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1) +
    ((SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1) + 1) / NUM_BANKS_DATA_PARTITION];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + cuda_leaf_data_start;
  uint8_t bit0 = 0;
  uint8_t bit1 = 0;
  uint8_t bit2 = 0;
  uint8_t bit3 = 0;
  uint8_t bit4 = 0;
  uint8_t bit5 = 0;
  uint8_t bit6 = 0;
  uint8_t bit7 = 0;
  unsigned int local_data_index = ((blockIdx.x * blockDim.x) << 3) + (threadIdx.x << 2);
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit0 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit0 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit0 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit0 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit0 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit1 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit1 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit1 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit1 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit1 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit2 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit2 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit2 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit2 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit2 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit3 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit3 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit3 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit3 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit3 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  local_data_index = ((blockIdx.x * blockDim.x) << 3) + ((threadIdx.x + blockDim.x) << 2);
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit4 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit4 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit4 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit4 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit4 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit5 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit5 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit5 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit5 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit5 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit6 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit6 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit6 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit6 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit6 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if ((MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) ||
      (MISSING_IS_NA && !MFB_IS_NA && bin == max_bin_ref)) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit7 = split_missing_default_to_left;
    } else if ((bin < min_bin_ref || bin > max_bin_ref)) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO || MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit7 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit7 = split_default_to_left;
      }
    } else if (bin > th) {
      cuda_data_to_left[local_data_index] = 0;
      bit7 = 0;
    } else {
      cuda_data_to_left[local_data_index] = 1;
      bit7 = 1;
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = bit0 + bit1 + bit2 + bit3;
  thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x + blockDim.x)] = bit4 + bit5 + bit6 + bit7;
  __syncthreads();
  ReduceSum(thread_to_left_offset_cnt, (split_indices_block_size_data_partition << 1));
  __syncthreads();
  if (threadIdx.x == 0) {
    const data_size_t num_data_in_block = (((blockIdx.x + 1) * blockDim.x * 8) <= num_data_in_leaf) ?
      static_cast<data_size_t>(blockDim.x * 8) :
      (num_data_in_leaf - static_cast<data_size_t>(blockIdx.x * blockDim.x * 8));
    if (num_data_in_block > 0) {
      const data_size_t data_to_left = static_cast<data_size_t>(thread_to_left_offset_cnt[0]);
      block_to_left_offset_buffer[blockIdx.x + 1] = data_to_left;
      block_to_right_offset_buffer[blockIdx.x + 1] = num_data_in_block - data_to_left;
    } else {
      block_to_left_offset_buffer[blockIdx.x + 1] = 0;
      block_to_right_offset_buffer[blockIdx.x + 1] = 0;
    }
  }
}

// min_bin_ref == max_bin_ref
template <typename BIN_TYPE, bool MISSING_IS_ZERO, bool MISSING_IS_NA, bool MFB_IS_ZERO, bool MFB_IS_NA, bool MAX_TO_LEFT>
__global__ void GenDataToLeftBitVectorKernel16_2(const int best_split_feature_ref, const data_size_t cuda_leaf_data_start,
  const data_size_t num_data_in_leaf, const data_size_t* cuda_data_indices,
  const uint32_t th, const int num_features_ref, const BIN_TYPE* column_data,
  // values from feature
  const uint32_t t_zero_bin, const uint32_t most_freq_bin_ref, const uint32_t max_bin_ref, const uint32_t min_bin_ref,
  const uint8_t split_default_to_left, const uint8_t split_missing_default_to_left,
  uint8_t* cuda_data_to_left,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition,
  int* cuda_data_index_to_leaf_index, const int left_leaf_index, const int right_leaf_index,
  const int default_leaf_index, const int missing_default_leaf_index) {
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    printf("********************************************** calling GenDataToLeftBitVectorKernel16_2 **********************************************\n");
  }
  __shared__ uint16_t thread_to_left_offset_cnt[(SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1) + 1 +
    ((SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1) + 1) / NUM_BANKS_DATA_PARTITION];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + cuda_leaf_data_start;
  uint8_t bit0 = 0;
  uint8_t bit1 = 0;
  uint8_t bit2 = 0;
  uint8_t bit3 = 0;
  uint8_t bit4 = 0;
  uint8_t bit5 = 0;
  uint8_t bit6 = 0;
  uint8_t bit7 = 0;
  unsigned int local_data_index = ((blockIdx.x * blockDim.x) << 3) + (threadIdx.x << 2);
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit0 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit0 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit0 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit0 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit0 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit0 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit1 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit1 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit1 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit1 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit1 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit1 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit2 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit2 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit2 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit2 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit2 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit2 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit3 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit3 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit3 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit3 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit3 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit3 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  local_data_index = ((blockIdx.x * blockDim.x) << 3) + ((threadIdx.x + blockDim.x) << 2);
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit4 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit4 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit4 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit4 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit4 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit4 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit5 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit5 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit5 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit5 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit5 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit5 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit6 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit6 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit6 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit6 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit6 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit6 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  ++local_data_index;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const uint32_t bin = static_cast<uint32_t>(column_data[global_data_index]);
    if (MISSING_IS_ZERO && !MFB_IS_ZERO && bin == t_zero_bin) {
      cuda_data_to_left[local_data_index] = split_missing_default_to_left;
      bit7 = split_missing_default_to_left;
    } else if (bin != max_bin_ref) {
      if ((MISSING_IS_NA && MFB_IS_NA) || (MISSING_IS_ZERO && MFB_IS_ZERO)) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit7 = split_missing_default_to_left;
      } else {
        cuda_data_to_left[local_data_index] = split_default_to_left;
        bit7 = split_default_to_left;
      }
    } else {
      if (MISSING_IS_NA && !MFB_IS_NA) {
        cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        bit7 = split_missing_default_to_left;
      } else {
        if (MAX_TO_LEFT) {
          cuda_data_to_left[local_data_index] = 1;
          bit7 = 1;
        } else {
          cuda_data_to_left[local_data_index] = 0;
          bit7 = 0;
        }
      }
    }
  } else {
    cuda_data_to_left[local_data_index] = 0;
  }
  __syncthreads();
  thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x)] = bit0 + bit1 + bit2 + bit3;
  thread_to_left_offset_cnt[CONFLICT_FREE_INDEX(threadIdx.x + blockDim.x)] = bit4 + bit5 + bit6 + bit7;
  __syncthreads();
  ReduceSum(thread_to_left_offset_cnt, (split_indices_block_size_data_partition << 1));
  __syncthreads();
  if (threadIdx.x == 0) {
    const data_size_t num_data_in_block = (((blockIdx.x + 1) * blockDim.x * 8) <= num_data_in_leaf) ?
      static_cast<data_size_t>(blockDim.x * 8) :
      (num_data_in_leaf - static_cast<data_size_t>(blockIdx.x * blockDim.x * 8));
    if (num_data_in_block > 0) {
      const data_size_t data_to_left = static_cast<data_size_t>(thread_to_left_offset_cnt[0]);
      block_to_left_offset_buffer[blockIdx.x + 1] = data_to_left;
      block_to_right_offset_buffer[blockIdx.x + 1] = num_data_in_block - data_to_left;
    } else {
      block_to_left_offset_buffer[blockIdx.x + 1] = 0;
      block_to_right_offset_buffer[blockIdx.x + 1] = 0;
    }
  }
}

#define GenBitVector_ARGS \
  split_feature_index, leaf_data_start, num_data_in_leaf, cuda_data_indices_, \
  th, num_features_,  \
  column_data, t_zero_bin, most_freq_bin, max_bin, min_bin, split_default_to_left,  \
  split_missing_default_to_left, cuda_data_to_left_, cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_, \
  split_indices_block_size_data_partition_aligned, \
  cuda_data_index_to_leaf_index_, left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index

template <typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelMaxIsMinInner(
  const bool missing_is_zero,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_bin_to_left,
  const int column_index,
  const int num_blocks_final,
  const int split_indices_block_size_data_partition_aligned,
  const int split_feature_index,
  const data_size_t leaf_data_start,
  const data_size_t num_data_in_leaf,
  const uint32_t th,
  const uint32_t t_zero_bin,
  const uint32_t most_freq_bin,
  const uint32_t max_bin,
  const uint32_t min_bin,
  const uint8_t split_default_to_left,
  const uint8_t split_missing_default_to_left,
  const int left_leaf_index,
  const int right_leaf_index,
  const int default_leaf_index,
  const int missing_default_leaf_index) {
  if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, false, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, false, true, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, false, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16<BIN_TYPE, true, true, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  }
}

template <typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelMaxIsMinInner2(
  const bool missing_is_zero,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const bool max_bin_to_left,
  const int column_index,
  const int num_blocks_final,
  const int split_indices_block_size_data_partition_aligned,
  const int split_feature_index,
  const data_size_t leaf_data_start,
  const data_size_t num_data_in_leaf,
  const uint32_t th,
  const uint32_t t_zero_bin,
  const uint32_t most_freq_bin,
  const uint32_t max_bin,
  const uint32_t min_bin,
  const uint8_t split_default_to_left,
  const uint8_t split_missing_default_to_left,
  const int left_leaf_index,
  const int right_leaf_index,
  const int default_leaf_index,
  const int missing_default_leaf_index) {
  int grid_dim = 0;
  int block_dim = 0;
  CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  CHECK_EQ(num_blocks_final, grid_dim);
  CHECK_EQ(split_indices_block_size_data_partition_aligned, block_dim);
  if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, false, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, false, true, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, false, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && !max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na && max_bin_to_left) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel16_2<BIN_TYPE, true, true, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  }
}

template <typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner(
  const bool missing_is_zero,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const int column_index,
  const int num_blocks_final,
  const int split_indices_block_size_data_partition_aligned,
  const int split_feature_index,
  const data_size_t leaf_data_start,
  const data_size_t num_data_in_leaf,
  const uint32_t th,
  const uint32_t t_zero_bin,
  const uint32_t most_freq_bin,
  const uint32_t max_bin,
  const uint32_t min_bin,
  const uint8_t split_default_to_left,
  const uint8_t split_missing_default_to_left,
  const int left_leaf_index,
  const int right_leaf_index,
  const int default_leaf_index,
  const int missing_default_leaf_index) {
  if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, false, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, false, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, false, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, false, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, true, false><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0<BIN_TYPE, true, true, true, true><<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  }
}

template <typename BIN_TYPE>
void CUDADataPartition::LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner2(
  const bool missing_is_zero,
  const bool missing_is_na,
  const bool mfb_is_zero,
  const bool mfb_is_na,
  const int column_index,
  const int num_blocks_final,
  const int split_indices_block_size_data_partition_aligned,
  const int split_feature_index,
  const data_size_t leaf_data_start,
  const data_size_t num_data_in_leaf,
  const uint32_t th,
  const uint32_t t_zero_bin,
  const uint32_t most_freq_bin,
  const uint32_t max_bin,
  const uint32_t min_bin,
  const uint8_t split_default_to_left,
  const uint8_t split_missing_default_to_left,
  const int left_leaf_index,
  const int right_leaf_index,
  const int default_leaf_index,
  const int missing_default_leaf_index) {
  int grid_dim = 0;
  int block_dim = 0;
  CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  CHECK_EQ(num_blocks_final, grid_dim);
  CHECK_EQ(split_indices_block_size_data_partition_aligned, block_dim);
  if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (!missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, false, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && !missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, false, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, false, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && !mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, false, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && !mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, true, false><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  } else if (missing_is_zero && missing_is_na && mfb_is_zero && mfb_is_na) {
    const BIN_TYPE* column_data = reinterpret_cast<const BIN_TYPE*>(cuda_data_by_column_[column_index]);
    GenDataToLeftBitVectorKernel0_2<BIN_TYPE, true, true, true, true><<<grid_dim, block_dim, 0, cuda_streams_[0]>>>(GenBitVector_ARGS);
  }
}

#undef GenBitVector_ARGS

void CUDADataPartition::LaunchGenDataToLeftBitVectorKernel(const data_size_t num_data_in_leaf,
  const int split_feature_index, const uint32_t split_threshold,
  const uint8_t split_default_left, const data_size_t leaf_data_start,
  const int left_leaf_index, const int right_leaf_index) {
  const int min_num_blocks = num_data_in_leaf <= 100 ? 1 : 80;
  const int num_blocks = std::max(min_num_blocks, (num_data_in_leaf + SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  int split_indices_block_size_data_partition = (num_data_in_leaf + num_blocks - 1) / num_blocks - 1;
  int split_indices_block_size_data_partition_aligned = 1;
  while (split_indices_block_size_data_partition > 0) {
    split_indices_block_size_data_partition_aligned <<= 1;
    split_indices_block_size_data_partition >>= 1;
  }
  const int num_blocks_final = (num_data_in_leaf + split_indices_block_size_data_partition_aligned - 1) / split_indices_block_size_data_partition_aligned;
  const uint8_t missing_is_zero = feature_missing_is_zero_[split_feature_index];
  const uint8_t missing_is_na = feature_missing_is_na_[split_feature_index];
  const uint8_t mfb_is_zero = feature_mfb_is_zero_[split_feature_index];
  const uint8_t mfb_is_na = feature_mfb_is_na_[split_feature_index];
  const uint32_t default_bin = feature_default_bins_[split_feature_index];
  const uint32_t most_freq_bin = feature_most_freq_bins_[split_feature_index];
  const uint32_t min_bin = feature_min_bins_[split_feature_index];
  const uint32_t max_bin = feature_max_bins_[split_feature_index];

  uint32_t th = split_threshold + min_bin;
  uint32_t t_zero_bin = min_bin + default_bin;
  if (most_freq_bin == 0) {
    --th;
    --t_zero_bin;  
  }
  uint8_t split_default_to_left = 0;
  uint8_t split_missing_default_to_left = 0;
  int default_leaf_index = right_leaf_index;
  int missing_default_leaf_index = right_leaf_index;
  if (most_freq_bin <= split_threshold) {
    split_default_to_left = 1;
    default_leaf_index = left_leaf_index;
  }
  if (missing_is_zero || missing_is_na) {
    if (split_default_left) {
      split_missing_default_to_left = 1;
      missing_default_leaf_index = left_leaf_index;
    }
  }
  const int column_index = feature_index_to_column_index_[split_feature_index];
  const uint8_t bit_type = column_bit_type_[column_index];

  const bool max_bin_to_left = (max_bin <= th);

  if (min_bin < max_bin) {
    if (bit_type == 8) {
      LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner<uint8_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        column_index,
        num_blocks_final,
        split_indices_block_size_data_partition_aligned,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 16) {
      LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner<uint16_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        column_index,
        num_blocks_final,
        split_indices_block_size_data_partition_aligned,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 32) {
      LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner<uint32_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        column_index,
        num_blocks_final,
        split_indices_block_size_data_partition_aligned,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    }
  } else {
    if (bit_type == 8) {
      LaunchGenDataToLeftBitVectorKernelMaxIsMinInner<uint8_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        max_bin_to_left,
        column_index,
        num_blocks_final,
        split_indices_block_size_data_partition_aligned,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 16) {
      LaunchGenDataToLeftBitVectorKernelMaxIsMinInner<uint16_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        max_bin_to_left,
        column_index,
        num_blocks_final,
        split_indices_block_size_data_partition_aligned,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 32) {
      LaunchGenDataToLeftBitVectorKernelMaxIsMinInner<uint32_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        max_bin_to_left,
        column_index,
        num_blocks_final,
        split_indices_block_size_data_partition_aligned,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    }
  }

  if (bit_type == 8) {
    const uint8_t* column_data = reinterpret_cast<const uint8_t*>(cuda_data_by_column_[column_index]);
    LaunchUpdateDataIndexToLeafIndexKernel<uint8_t>(leaf_data_start, num_data_in_leaf,
      cuda_data_indices_, th, column_data, t_zero_bin, max_bin, min_bin, cuda_data_index_to_leaf_index_,
      left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index,
      static_cast<bool>(missing_is_zero),
      static_cast<bool>(missing_is_na),
      static_cast<bool>(mfb_is_zero),
      static_cast<bool>(mfb_is_na),
      max_bin_to_left,
      num_blocks_final,
      split_indices_block_size_data_partition_aligned);
  } else if (bit_type == 16) {
    const uint16_t* column_data = reinterpret_cast<const uint16_t*>(cuda_data_by_column_[column_index]);
    LaunchUpdateDataIndexToLeafIndexKernel<uint16_t>(leaf_data_start, num_data_in_leaf,
      cuda_data_indices_, th, column_data, t_zero_bin, max_bin, min_bin, cuda_data_index_to_leaf_index_,
      left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index,
      static_cast<bool>(missing_is_zero),
      static_cast<bool>(missing_is_na),
      static_cast<bool>(mfb_is_zero),
      static_cast<bool>(mfb_is_na),
      max_bin_to_left,
      num_blocks_final,
      split_indices_block_size_data_partition_aligned);
  } else if (bit_type == 32) {
    const uint32_t* column_data = reinterpret_cast<const uint32_t*>(cuda_data_by_column_[column_index]);
    LaunchUpdateDataIndexToLeafIndexKernel<uint32_t>(leaf_data_start, num_data_in_leaf,
      cuda_data_indices_, th, column_data, t_zero_bin, max_bin, min_bin, cuda_data_index_to_leaf_index_,
      left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index,
      static_cast<bool>(missing_is_zero),
      static_cast<bool>(missing_is_na),
      static_cast<bool>(mfb_is_zero),
      static_cast<bool>(mfb_is_na),
      max_bin_to_left,
      num_blocks_final,
      split_indices_block_size_data_partition_aligned);
  }
}

void CUDADataPartition::LaunchGenDataToLeftBitVectorKernel2(const data_size_t num_data_in_leaf,
  const int split_feature_index, const uint32_t split_threshold,
  const uint8_t split_default_left, const data_size_t leaf_data_start,
  const int left_leaf_index, const int right_leaf_index) {
  int grid_dim = 0;
  int block_dim = 0;
  CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  const uint8_t missing_is_zero = feature_missing_is_zero_[split_feature_index];
  const uint8_t missing_is_na = feature_missing_is_na_[split_feature_index];
  const uint8_t mfb_is_zero = feature_mfb_is_zero_[split_feature_index];
  const uint8_t mfb_is_na = feature_mfb_is_na_[split_feature_index];
  const uint32_t default_bin = feature_default_bins_[split_feature_index];
  const uint32_t most_freq_bin = feature_most_freq_bins_[split_feature_index];
  const uint32_t min_bin = feature_min_bins_[split_feature_index];
  const uint32_t max_bin = feature_max_bins_[split_feature_index];

  uint32_t th = split_threshold + min_bin;
  uint32_t t_zero_bin = min_bin + default_bin;
  if (most_freq_bin == 0) {
    --th;
    --t_zero_bin;  
  }
  uint8_t split_default_to_left = 0;
  uint8_t split_missing_default_to_left = 0;
  int default_leaf_index = right_leaf_index;
  int missing_default_leaf_index = right_leaf_index;
  if (most_freq_bin <= split_threshold) {
    split_default_to_left = 1;
    default_leaf_index = left_leaf_index;
  }
  if (missing_is_zero || missing_is_na) {
    if (split_default_left) {
      split_missing_default_to_left = 1;
      missing_default_leaf_index = left_leaf_index;
    }
  }
  const int column_index = feature_index_to_column_index_[split_feature_index];
  const uint8_t bit_type = column_bit_type_[column_index];

  const bool max_bin_to_left = (max_bin <= th);

  if (min_bin < max_bin) {
    if (bit_type == 8) {
      LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner2<uint8_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        column_index,
        grid_dim,
        block_dim,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 16) {
      LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner2<uint16_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        column_index,
        grid_dim,
        block_dim,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 32) {
      LaunchGenDataToLeftBitVectorKernelMaxIsNotMinInner2<uint32_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        column_index,
        grid_dim,
        block_dim,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    }
  } else {
    if (bit_type == 8) {
      LaunchGenDataToLeftBitVectorKernelMaxIsMinInner2<uint8_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        max_bin_to_left,
        column_index,
        grid_dim,
        block_dim,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 16) {
      LaunchGenDataToLeftBitVectorKernelMaxIsMinInner2<uint16_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        max_bin_to_left,
        column_index,
        grid_dim,
        block_dim,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    } else if (bit_type == 32) {
      LaunchGenDataToLeftBitVectorKernelMaxIsMinInner2<uint32_t>(
        missing_is_zero,
        missing_is_na,
        mfb_is_zero,
        mfb_is_na,
        max_bin_to_left,
        column_index,
        grid_dim,
        block_dim,
        split_feature_index,
        leaf_data_start,
        num_data_in_leaf,
        th,
        t_zero_bin,
        most_freq_bin,
        max_bin,
        min_bin,
        split_default_to_left,
        split_missing_default_to_left,
        left_leaf_index,
        right_leaf_index,
        default_leaf_index,
        missing_default_leaf_index);
    }
  }

  int grid_dim_copy = 0;
  int block_dim_copy = 0;
  CalcBlockDimInCopy(num_data_in_leaf, &grid_dim_copy, &block_dim_copy);
  if (bit_type == 8) {
    const uint8_t* column_data = reinterpret_cast<const uint8_t*>(cuda_data_by_column_[column_index]);
    LaunchUpdateDataIndexToLeafIndexKernel<uint8_t>(leaf_data_start, num_data_in_leaf,
      cuda_data_indices_, th, column_data, t_zero_bin, max_bin, min_bin, cuda_data_index_to_leaf_index_,
      left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index,
      static_cast<bool>(missing_is_zero),
      static_cast<bool>(missing_is_na),
      static_cast<bool>(mfb_is_zero),
      static_cast<bool>(mfb_is_na),
      max_bin_to_left,
      grid_dim_copy,
      block_dim_copy);
  } else if (bit_type == 16) {
    const uint16_t* column_data = reinterpret_cast<const uint16_t*>(cuda_data_by_column_[column_index]);
    LaunchUpdateDataIndexToLeafIndexKernel<uint16_t>(leaf_data_start, num_data_in_leaf,
      cuda_data_indices_, th, column_data, t_zero_bin, max_bin, min_bin, cuda_data_index_to_leaf_index_,
      left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index,
      static_cast<bool>(missing_is_zero),
      static_cast<bool>(missing_is_na),
      static_cast<bool>(mfb_is_zero),
      static_cast<bool>(mfb_is_na),
      max_bin_to_left,
      grid_dim_copy,
      block_dim_copy);
  } else if (bit_type == 32) {
    const uint32_t* column_data = reinterpret_cast<const uint32_t*>(cuda_data_by_column_[column_index]);
    LaunchUpdateDataIndexToLeafIndexKernel<uint32_t>(leaf_data_start, num_data_in_leaf,
      cuda_data_indices_, th, column_data, t_zero_bin, max_bin, min_bin, cuda_data_index_to_leaf_index_,
      left_leaf_index, right_leaf_index, default_leaf_index, missing_default_leaf_index,
      static_cast<bool>(missing_is_zero),
      static_cast<bool>(missing_is_na),
      static_cast<bool>(mfb_is_zero),
      static_cast<bool>(mfb_is_na),
      max_bin_to_left,
      grid_dim_copy,
      block_dim_copy);
  }
}

__global__ void AggregateBlockOffsetKernel(const int* leaf_index, data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  int* cuda_cur_num_leaves,
  const int* best_split_feature, const uint32_t* best_split_threshold,
  const uint8_t* best_split_default_left, const double* best_split_gain,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  const int* cuda_num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool, const int split_indices_block_size_data_partition) {
  __shared__ uint32_t block_to_left_offset[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint32_t block_to_right_offset[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  const int leaf_index_ref = *leaf_index;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[leaf_index_ref];
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX(threadIdx_x);
  const unsigned int conflict_free_threadIdx_x_plus_blockDim_x = CONFLICT_FREE_INDEX(threadIdx_x + blockDim_x);
  const uint32_t num_blocks = (num_data_in_leaf + split_indices_block_size_data_partition - 1) / split_indices_block_size_data_partition;
  const uint32_t num_aggregate_blocks = (num_blocks + split_indices_block_size_data_partition - 1) / split_indices_block_size_data_partition;
  uint32_t left_prev_sum = 0;
  for (uint32_t block_id = 0; block_id < num_aggregate_blocks; ++block_id) {
    const unsigned int read_index = block_id * blockDim_x * 2 + threadIdx_x;
    if (read_index < num_blocks) {
      block_to_left_offset[conflict_free_threadIdx_x] = block_to_left_offset_buffer[read_index + 1];
    } else {
      block_to_left_offset[conflict_free_threadIdx_x] = 0;
    }
    const unsigned int read_index_plus_blockDim_x = read_index + blockDim_x;
    if (read_index_plus_blockDim_x < num_blocks) {
      block_to_left_offset[conflict_free_threadIdx_x_plus_blockDim_x] = block_to_left_offset_buffer[read_index_plus_blockDim_x + 1];
    } else {
      block_to_left_offset[conflict_free_threadIdx_x_plus_blockDim_x] = 0;
    }
    if (threadIdx_x == 0) {
      block_to_left_offset[0] += left_prev_sum;
    }
    __syncthreads();
    PrefixSum(block_to_left_offset, split_indices_block_size_data_partition);
    __syncthreads();
    if (threadIdx_x == 0) {
      left_prev_sum = block_to_left_offset[CONFLICT_FREE_INDEX(split_indices_block_size_data_partition)];
    }
    if (read_index < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
      block_to_left_offset_buffer[read_index + 1] = block_to_left_offset[conflict_free_threadIdx_x_plus_1];
    }
    if (read_index_plus_blockDim_x < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1_plus_blockDim_x = CONFLICT_FREE_INDEX(threadIdx_x + 1 + blockDim_x);
      block_to_left_offset_buffer[read_index_plus_blockDim_x + 1] = block_to_left_offset[conflict_free_threadIdx_x_plus_1_plus_blockDim_x];
    }
    __syncthreads();
  }
  const unsigned int to_left_total_cnt = block_to_left_offset_buffer[num_blocks];
  uint32_t right_prev_sum = to_left_total_cnt;
  for (uint32_t block_id = 0; block_id < num_aggregate_blocks; ++block_id) {
    const unsigned int read_index = block_id * blockDim_x * 2 + threadIdx_x;
    if (read_index < num_blocks) {
      block_to_right_offset[conflict_free_threadIdx_x] = block_to_right_offset_buffer[read_index + 1];
    } else {
      block_to_right_offset[conflict_free_threadIdx_x] = 0;
    }
    const unsigned int read_index_plus_blockDim_x = read_index + blockDim_x;
    if (read_index_plus_blockDim_x < num_blocks) {
      block_to_right_offset[conflict_free_threadIdx_x_plus_blockDim_x] = block_to_right_offset_buffer[read_index_plus_blockDim_x + 1];
    } else {
      block_to_right_offset[conflict_free_threadIdx_x_plus_blockDim_x] = 0;
    }
    if (threadIdx_x == 0) {
      block_to_right_offset[0] += right_prev_sum;
    }
    __syncthreads();
    PrefixSum(block_to_right_offset, split_indices_block_size_data_partition);
    __syncthreads();
    if (threadIdx_x == 0) {
      right_prev_sum = block_to_right_offset[CONFLICT_FREE_INDEX(split_indices_block_size_data_partition)];
    }
    if (read_index < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
      block_to_right_offset_buffer[read_index + 1] = block_to_right_offset[conflict_free_threadIdx_x_plus_1];
    }
    if (read_index_plus_blockDim_x < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1_plus_blockDim_x = CONFLICT_FREE_INDEX(threadIdx_x + 1 + blockDim_x);
      block_to_right_offset_buffer[read_index_plus_blockDim_x + 1] = block_to_right_offset[conflict_free_threadIdx_x_plus_1_plus_blockDim_x];
    }
    __syncthreads();
  }
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    ++(*cuda_cur_num_leaves);
    const int cur_max_leaf_index = (*cuda_cur_num_leaves) - 1;
    block_to_left_offset_buffer[0] = 0;
    const unsigned int to_left_total_cnt = block_to_left_offset_buffer[num_blocks];
    block_to_right_offset_buffer[0] = to_left_total_cnt;
    const data_size_t old_leaf_data_end = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[leaf_index_ref] = cuda_leaf_data_start[leaf_index_ref] + static_cast<data_size_t>(to_left_total_cnt);
    cuda_leaf_num_data[leaf_index_ref] = static_cast<data_size_t>(to_left_total_cnt);
    cuda_leaf_data_start[cur_max_leaf_index] = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[cur_max_leaf_index] = old_leaf_data_end;
    cuda_leaf_num_data[cur_max_leaf_index] = block_to_right_offset_buffer[num_blocks] - to_left_total_cnt;
  }
}

__global__ void AggregateBlockOffsetKernel2(const int* leaf_index, data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  int* cuda_cur_num_leaves,
  const int* best_split_feature, const uint32_t* best_split_threshold,
  const uint8_t* best_split_default_left, const double* best_split_gain,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  const int* cuda_num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool,
  const data_size_t num_blocks) {
  __shared__ uint32_t block_to_left_offset[AGGREGATE_BLOCK_SIZE + 2 +
    (AGGREGATE_BLOCK_SIZE + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint32_t block_to_right_offset[AGGREGATE_BLOCK_SIZE + 2 +
    (AGGREGATE_BLOCK_SIZE + 2) / NUM_BANKS_DATA_PARTITION];
  const int leaf_index_ref = *leaf_index;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[leaf_index_ref];
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX(threadIdx_x);
  const data_size_t num_blocks_plus_1 = num_blocks + 1;
  const uint32_t num_blocks_per_thread = (num_blocks_plus_1 + blockDim_x - 1) / blockDim_x;
  const uint32_t remain = num_blocks_plus_1 - ((num_blocks_per_thread - 1) * blockDim_x);
  const uint32_t remain_offset = remain * num_blocks_per_thread;
  uint32_t thread_start_block_index = 0;
  uint32_t thread_end_block_index = 0;
  if (threadIdx_x < remain) {
    thread_start_block_index = threadIdx_x * num_blocks_per_thread;
    thread_end_block_index = min(thread_start_block_index + num_blocks_per_thread, num_blocks_plus_1);
  } else {
    thread_start_block_index = remain_offset + (num_blocks_per_thread - 1) * (threadIdx_x - remain);
    thread_end_block_index = min(thread_start_block_index + num_blocks_per_thread - 1, num_blocks_plus_1);
  }
  if (threadIdx.x == 0) {
    block_to_right_offset_buffer[0] = 0;
  }
  __syncthreads();
  for (uint32_t block_index = thread_start_block_index + 1; block_index < thread_end_block_index; ++block_index) {
    block_to_left_offset_buffer[block_index] += block_to_left_offset_buffer[block_index - 1];
    block_to_right_offset_buffer[block_index] += block_to_right_offset_buffer[block_index - 1];
  }
  __syncthreads();
  if (thread_start_block_index < thread_end_block_index) {
    block_to_left_offset[conflict_free_threadIdx_x] = block_to_left_offset_buffer[thread_end_block_index - 1];
    block_to_right_offset[conflict_free_threadIdx_x] = block_to_right_offset_buffer[thread_end_block_index - 1];
  } else {
    block_to_left_offset[conflict_free_threadIdx_x] = 0;
    block_to_right_offset[conflict_free_threadIdx_x] = 0;
  }
  __syncthreads();
  PrefixSum_1024(block_to_left_offset, blockDim_x);
  PrefixSum_1024(block_to_right_offset, blockDim_x);
  __syncthreads();
  const uint32_t to_left_total_count = block_to_left_offset[CONFLICT_FREE_INDEX(blockDim_x)];
  const uint32_t to_left_thread_block_offset = block_to_left_offset[conflict_free_threadIdx_x];
  const uint32_t to_right_thread_block_offset = block_to_right_offset[conflict_free_threadIdx_x] + to_left_total_count;
  for (uint32_t block_index = thread_start_block_index; block_index < thread_end_block_index; ++block_index) {
    block_to_left_offset_buffer[block_index] += to_left_thread_block_offset;
    block_to_right_offset_buffer[block_index] += to_right_thread_block_offset;
  }
  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    ++(*cuda_cur_num_leaves);
    const int cur_max_leaf_index = (*cuda_cur_num_leaves) - 1;
    const data_size_t old_leaf_data_end = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[leaf_index_ref] = cuda_leaf_data_start[leaf_index_ref] + static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_num_data[leaf_index_ref] = static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_data_start[cur_max_leaf_index] = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[cur_max_leaf_index] = old_leaf_data_end;
    cuda_leaf_num_data[cur_max_leaf_index] = num_data_in_leaf - static_cast<data_size_t>(to_left_total_count);
  }
}

__global__ void AggregateBlockOffsetKernel3(const int* leaf_index, data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  int* cuda_cur_num_leaves,
  const int* best_split_feature, const uint32_t* best_split_threshold,
  const uint8_t* best_split_default_left, const double* best_split_gain,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  const int* cuda_num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool,
  const data_size_t num_blocks, const data_size_t num_blocks_aligned) {
  __shared__ uint32_t block_to_left_offset[AGGREGATE_BLOCK_SIZE + 2 +
    (AGGREGATE_BLOCK_SIZE + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint32_t block_to_right_offset[AGGREGATE_BLOCK_SIZE + 2 +
    (AGGREGATE_BLOCK_SIZE + 2) / NUM_BANKS_DATA_PARTITION];
  const int leaf_index_ref = *leaf_index;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[leaf_index_ref];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX(threadIdx_x);
  const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
  if (threadIdx_x < static_cast<unsigned int>(num_blocks)) {
    block_to_left_offset[conflict_free_threadIdx_x] = block_to_left_offset_buffer[threadIdx_x + 1];
    block_to_right_offset[conflict_free_threadIdx_x] = block_to_right_offset_buffer[threadIdx_x + 1];
  } else {
    block_to_left_offset[conflict_free_threadIdx_x] = 0;
    block_to_right_offset[conflict_free_threadIdx_x] = 0;
  }
  __syncthreads();
  PrefixSum(block_to_left_offset, num_blocks_aligned);
  PrefixSum(block_to_right_offset, num_blocks_aligned);
  __syncthreads();
  const uint32_t to_left_total_count = block_to_left_offset[CONFLICT_FREE_INDEX(num_blocks_aligned)];
  if (threadIdx_x < static_cast<unsigned int>(num_blocks)) {
    block_to_left_offset_buffer[threadIdx_x + 1] = block_to_left_offset[conflict_free_threadIdx_x_plus_1];
    block_to_right_offset_buffer[threadIdx_x + 1] = block_to_right_offset[conflict_free_threadIdx_x_plus_1] + to_left_total_count;
  }
  if (threadIdx_x == 0) {
    block_to_right_offset_buffer[0] = to_left_total_count;
  }
  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    ++(*cuda_cur_num_leaves);
    const int cur_max_leaf_index = (*cuda_cur_num_leaves) - 1;
    const data_size_t old_leaf_data_end = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[leaf_index_ref] = cuda_leaf_data_start[leaf_index_ref] + static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_num_data[leaf_index_ref] = static_cast<data_size_t>(to_left_total_count);
    cuda_leaf_data_start[cur_max_leaf_index] = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[cur_max_leaf_index] = old_leaf_data_end;
    cuda_leaf_num_data[cur_max_leaf_index] = num_data_in_leaf - static_cast<data_size_t>(to_left_total_count);
  }
}

__global__ void SplitTreeStructureKernel(const int* leaf_index, data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  int* cuda_cur_num_leaves,
  const int* best_split_feature, const uint32_t* best_split_threshold,
  const uint8_t* best_split_default_left, const double* best_split_gain,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value, uint8_t* best_split_found,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  const int* cuda_num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool, const int split_indices_block_size_data_partition,

  int* tree_split_leaf_index, int* tree_inner_feature_index, uint32_t* tree_threshold,
  double* tree_left_output, double* tree_right_output, data_size_t* tree_left_count, data_size_t* tree_right_count,
  double* tree_left_sum_hessian, double* tree_right_sum_hessian, double* tree_gain, uint8_t* tree_default_left,
  double* data_partition_leaf_output,
  int* cuda_split_info_buffer) {
  const int leaf_index_ref = *leaf_index;
  const int cur_max_leaf_index = (*cuda_cur_num_leaves) - 1;
  const unsigned int to_left_total_cnt = cuda_leaf_num_data[leaf_index_ref];
  const int cuda_num_total_bin_ref = *cuda_num_total_bin;
  double* cuda_split_info_buffer_for_hessians = reinterpret_cast<double*>(cuda_split_info_buffer + 8);
  const unsigned int global_thread_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_thread_index == 0) {
    tree_split_leaf_index[cur_max_leaf_index - 1] = leaf_index_ref;
  } else if (global_thread_index == 1) {
    tree_inner_feature_index[cur_max_leaf_index - 1] = best_split_feature[leaf_index_ref];
  } else if (global_thread_index == 2) {
    tree_threshold[cur_max_leaf_index - 1] = best_split_threshold[leaf_index_ref];
  } else if (global_thread_index == 3) {
    tree_left_output[cur_max_leaf_index - 1] = best_left_leaf_value[leaf_index_ref];
  } else if (global_thread_index == 4) {
    tree_right_output[cur_max_leaf_index - 1] = best_right_leaf_value[leaf_index_ref];
  } else if (global_thread_index == 5) {
    tree_left_count[cur_max_leaf_index - 1] = best_left_count[leaf_index_ref];
  } else if (global_thread_index == 6) {
    tree_right_count[cur_max_leaf_index - 1] = best_right_count[leaf_index_ref];
  } else if (global_thread_index == 7) {
    tree_left_sum_hessian[cur_max_leaf_index - 1] = best_left_sum_hessians[leaf_index_ref];
  } else if (global_thread_index == 8) {
    tree_right_sum_hessian[cur_max_leaf_index - 1] = best_right_sum_hessians[leaf_index_ref];
  } else if (global_thread_index == 9) {
    tree_gain[cur_max_leaf_index - 1] = best_split_gain[leaf_index_ref];
  } else if (global_thread_index == 10) {
    tree_default_left[cur_max_leaf_index - 1] = best_split_default_left[leaf_index_ref];
  } else if (global_thread_index == 11) {
    data_partition_leaf_output[leaf_index_ref] = best_left_leaf_value[leaf_index_ref];
  } else if (global_thread_index == 12) {
    data_partition_leaf_output[cur_max_leaf_index] = best_right_leaf_value[leaf_index_ref];
  } else if (global_thread_index == 13) {
    cuda_split_info_buffer[0] = leaf_index_ref;
  } else if (global_thread_index == 14) {
    cuda_split_info_buffer[1] = cuda_leaf_num_data[leaf_index_ref];
  } else if (global_thread_index == 15) {
    cuda_split_info_buffer[2] = cuda_leaf_data_start[leaf_index_ref];
  } else if (global_thread_index == 16) {
    cuda_split_info_buffer[3] = cur_max_leaf_index;
  } else if (global_thread_index == 17) {
    cuda_split_info_buffer[4] = cuda_leaf_num_data[cur_max_leaf_index];
  } else if (global_thread_index == 18) {
    cuda_split_info_buffer[5] = cuda_leaf_data_start[cur_max_leaf_index];
  } else if (global_thread_index == 19) {
    cuda_split_info_buffer_for_hessians[0] = best_left_sum_hessians[leaf_index_ref];
  } else if (global_thread_index == 20) {
    cuda_split_info_buffer_for_hessians[1] = best_right_sum_hessians[leaf_index_ref];
  } else if (global_thread_index == 21) {
    best_split_found[leaf_index_ref] = 0;
  } else if (global_thread_index == 22) {
    best_split_found[cur_max_leaf_index] = 0;
  }

  if (cuda_leaf_num_data[leaf_index_ref] < cuda_leaf_num_data[cur_max_leaf_index]) {
    if (global_thread_index == 0) {
      hist_t* parent_hist_ptr = cuda_hist_pool[leaf_index_ref];
      cuda_hist_pool[cur_max_leaf_index] = parent_hist_ptr;
      cuda_hist_pool[leaf_index_ref] = cuda_hist + 2 * cur_max_leaf_index * cuda_num_total_bin_ref;
      *smaller_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[leaf_index_ref];
      *larger_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[cur_max_leaf_index];
    } else if (global_thread_index == 1) {
      *smaller_leaf_cuda_sum_of_gradients_pointer = best_left_sum_gradients[leaf_index_ref];
    } else if (global_thread_index == 2) {
      *smaller_leaf_cuda_sum_of_hessians_pointer = best_left_sum_hessians[leaf_index_ref];
    } else if (global_thread_index == 3) {
      *smaller_leaf_cuda_num_data_in_leaf_pointer = to_left_total_cnt;
    } else if (global_thread_index == 4) {
      *smaller_leaf_cuda_gain_pointer = best_left_gain[leaf_index_ref];
    } else if (global_thread_index == 5) {
      *smaller_leaf_cuda_leaf_value_pointer = best_left_leaf_value[leaf_index_ref];
    } else if (global_thread_index == 6) {
      *smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices;
    } else if (global_thread_index == 7) {
      *larger_leaf_cuda_leaf_index_pointer = cur_max_leaf_index;
    } else if (global_thread_index == 8) {
      *larger_leaf_cuda_sum_of_gradients_pointer = best_right_sum_gradients[leaf_index_ref];
    } else if (global_thread_index == 9) {
      *larger_leaf_cuda_sum_of_hessians_pointer = best_right_sum_hessians[leaf_index_ref];
    } else if (global_thread_index == 10) {
      *larger_leaf_cuda_num_data_in_leaf_pointer = cuda_leaf_num_data[cur_max_leaf_index];
    } else if (global_thread_index == 11) {
      *larger_leaf_cuda_gain_pointer = best_right_gain[leaf_index_ref];
    } else if (global_thread_index == 12) {
      *larger_leaf_cuda_leaf_value_pointer = best_right_leaf_value[leaf_index_ref];
    } else if (global_thread_index == 13) {
      *larger_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices + cuda_leaf_num_data[leaf_index_ref];
    } else if (global_thread_index == 14) {
      cuda_split_info_buffer[6] = leaf_index_ref;
    } else if (global_thread_index == 15) {
      cuda_split_info_buffer[7] = cur_max_leaf_index;
    } else if (global_thread_index == 16) {
      *smaller_leaf_cuda_leaf_index_pointer = leaf_index_ref;
    }
  } else {
    if (global_thread_index == 0) {
      *larger_leaf_cuda_leaf_index_pointer = leaf_index_ref;
    } else if (global_thread_index == 1) {
      *larger_leaf_cuda_sum_of_gradients_pointer = best_left_sum_gradients[leaf_index_ref];
    } else if (global_thread_index == 2) {
      *larger_leaf_cuda_sum_of_hessians_pointer = best_left_sum_hessians[leaf_index_ref];
    } else if (global_thread_index == 3) {
      *larger_leaf_cuda_num_data_in_leaf_pointer = to_left_total_cnt;
    } else if (global_thread_index == 4) {
      *larger_leaf_cuda_gain_pointer = best_left_gain[leaf_index_ref];
    } else if (global_thread_index == 5) {
      *larger_leaf_cuda_leaf_value_pointer = best_left_leaf_value[leaf_index_ref];
    } else if (global_thread_index == 6) {
      *larger_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices;
    } else if (global_thread_index == 7) {
      *smaller_leaf_cuda_leaf_index_pointer = cur_max_leaf_index;
    } else if (global_thread_index == 8) {
      *smaller_leaf_cuda_sum_of_gradients_pointer = best_right_sum_gradients[leaf_index_ref];
    } else if (global_thread_index == 9) {
      *smaller_leaf_cuda_sum_of_hessians_pointer = best_right_sum_hessians[leaf_index_ref];
    } else if (global_thread_index == 10) {
      *smaller_leaf_cuda_num_data_in_leaf_pointer = cuda_leaf_num_data[cur_max_leaf_index];
    } else if (global_thread_index == 11) {
      *smaller_leaf_cuda_gain_pointer = best_right_gain[leaf_index_ref];
    } else if (global_thread_index == 12) {
      *smaller_leaf_cuda_leaf_value_pointer = best_right_leaf_value[leaf_index_ref];
    } else if (global_thread_index == 13) {
      *smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices + cuda_leaf_num_data[leaf_index_ref];
    } else if (global_thread_index == 14) {
      cuda_hist_pool[cur_max_leaf_index] = cuda_hist + 2 * cur_max_leaf_index * cuda_num_total_bin_ref;
      *smaller_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[cur_max_leaf_index];
    } else if (global_thread_index == 15) {
      *larger_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[leaf_index_ref];
    } else if (global_thread_index == 16) {
      cuda_split_info_buffer[6] = cur_max_leaf_index;
    } else if (global_thread_index == 17) {
      cuda_split_info_buffer[7] = leaf_index_ref;
    }
  }
}

__global__ void SplitInnerKernel(const int* leaf_index, const int* cuda_cur_num_leaves,
  const data_size_t* cuda_leaf_data_start, const data_size_t* cuda_leaf_num_data,
  const data_size_t* cuda_data_indices, const uint8_t* split_to_left_bit_vector,
  const data_size_t* block_to_left_offset_buffer, const data_size_t* block_to_right_offset_buffer,
  data_size_t* out_data_indices_in_leaf, const int split_indices_block_size_data_partition) {
  //__shared__ uint8_t thread_split_to_left_bit_vector[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION];
  __shared__ uint16_t thread_to_left_pos[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint16_t thread_to_right_pos[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION];
  uint8_t first_to_left = 0;
  uint8_t second_to_left = 0;
  const int leaf_index_ref = *leaf_index;
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[leaf_index_ref];
  const data_size_t num_data_in_leaf_ref = cuda_leaf_num_data[leaf_index_ref] + cuda_leaf_num_data[(*cuda_cur_num_leaves) - 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
  const unsigned int global_thread_index = blockIdx.x * blockDim_x * 2 + threadIdx_x;
  const data_size_t* cuda_data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  if (global_thread_index < num_data_in_leaf_ref) {
    const uint8_t bit = split_to_left_bit_vector[global_thread_index];
    first_to_left = bit;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_1] = bit;
  } else {
    first_to_left = 0;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_1] = 0;
  }
  const unsigned int conflict_free_threadIdx_x_plus_blockDim_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + blockDim_x + 1);
  const unsigned int global_thread_index_plus_blockDim_x = global_thread_index + blockDim_x;
  if (global_thread_index_plus_blockDim_x < num_data_in_leaf_ref) {
    const uint8_t bit = split_to_left_bit_vector[global_thread_index_plus_blockDim_x];
    second_to_left = bit;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = bit;
  } else {
    second_to_left = 0;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = 0;
  }
  __syncthreads();
  const uint32_t to_right_block_offset = block_to_right_offset_buffer[blockIdx.x];
  const uint32_t to_left_block_offset = block_to_left_offset_buffer[blockIdx.x];
  if (threadIdx_x == 0) {
    thread_to_left_pos[0] = 0;
    thread_to_right_pos[0] = 0;
  }
  __syncthreads();
  PrefixSum(thread_to_left_pos, split_indices_block_size_data_partition);
  __syncthreads();
  if (threadIdx_x > 0) {
    thread_to_right_pos[threadIdx_x] = (threadIdx_x - thread_to_left_pos[conflict_free_threadIdx_x_plus_1]);
  }
  thread_to_right_pos[threadIdx_x + blockDim_x] = (threadIdx_x + blockDim_x - thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1]);
  __syncthreads();
  data_size_t* left_out_data_indices_in_leaf = out_data_indices_in_leaf + to_left_block_offset;
  data_size_t* right_out_data_indices_in_leaf = out_data_indices_in_leaf + to_right_block_offset;
  if (global_thread_index < num_data_in_leaf_ref) {
    if (first_to_left == 1) {
      left_out_data_indices_in_leaf[thread_to_left_pos[conflict_free_threadIdx_x_plus_1]] = cuda_data_indices_in_leaf[global_thread_index];
    } else {
      right_out_data_indices_in_leaf[thread_to_right_pos[threadIdx_x]] = cuda_data_indices_in_leaf[global_thread_index];
    }
  }
  if (global_thread_index_plus_blockDim_x < num_data_in_leaf_ref) {
    if (second_to_left == 1) {
      left_out_data_indices_in_leaf[thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1]] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x];
    } else {
      right_out_data_indices_in_leaf[thread_to_right_pos[threadIdx_x + blockDim_x]] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x];
    }
  }
}

__global__ void SplitInnerKernel2(const int* leaf_index, const int* cuda_cur_num_leaves,
  const data_size_t* cuda_leaf_data_start, const data_size_t* cuda_leaf_num_data,
  const data_size_t* cuda_data_indices, const uint8_t* split_to_left_bit_vector,
  const data_size_t* block_to_left_offset_buffer, const data_size_t* block_to_right_offset_buffer,
  data_size_t* out_data_indices_in_leaf, const int split_indices_block_size_data_partition) {
  __shared__ uint16_t thread_to_left_pos[(SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1) + 1 +
    ((SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1) + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint16_t thread_to_right_pos[(SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION << 1)];
  const int leaf_index_ref = *leaf_index;
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[leaf_index_ref];
  const data_size_t num_data_in_leaf_ref = cuda_leaf_num_data[leaf_index_ref] + cuda_leaf_num_data[(*cuda_cur_num_leaves) - 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
  const unsigned int global_thread_index = blockIdx.x * blockDim_x * 2 + threadIdx_x;
  const data_size_t* cuda_data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  const uint32_t* split_to_left_bit_vector_uint32 = reinterpret_cast<const uint32_t*>(split_to_left_bit_vector);
  const uint32_t bit32_0 = split_to_left_bit_vector_uint32[global_thread_index];
  const uint8_t bit_0 = static_cast<uint8_t>(bit32_0 & 0xf);
  uint8_t bit_1 = static_cast<uint8_t>((bit32_0 >> 8) & 0xf);
  uint8_t bit_2 = static_cast<uint8_t>((bit32_0 >> 16) & 0xf);
  uint8_t bit_3 = static_cast<uint8_t>((bit32_0 >> 24) & 0xf);
  const uint8_t bit_1_acc = bit_1 + bit_0;
  const uint8_t bit_2_acc = bit_1_acc + bit_2;
  const uint8_t bit_3_acc = bit_2_acc + bit_3;
  thread_to_left_pos[conflict_free_threadIdx_x_plus_1] = bit_3_acc;
  const unsigned int conflict_free_threadIdx_x_plus_blockDim_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + blockDim_x + 1);
  const unsigned int global_thread_index_plus_blockDim_x = global_thread_index + blockDim_x;
  const uint32_t bit32_1 = split_to_left_bit_vector_uint32[global_thread_index_plus_blockDim_x];
  const uint8_t bit_4 = static_cast<uint8_t>(bit32_1 & 0xf);
  uint8_t bit_5 = static_cast<uint8_t>((bit32_1 >> 8) & 0xf);
  uint8_t bit_6 = static_cast<uint8_t>((bit32_1 >> 16) & 0xf);
  uint8_t bit_7 = static_cast<uint8_t>((bit32_1 >> 24) & 0xf);
  const uint8_t bit_5_acc = bit_4 + bit_5;
  const uint8_t bit_6_acc = bit_5_acc + bit_6;
  const uint8_t bit_7_acc = bit_6_acc + bit_7;
  thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = bit_7_acc;
  __syncthreads();
  const uint32_t to_right_block_offset = block_to_right_offset_buffer[blockIdx.x];
  const uint32_t to_left_block_offset = block_to_left_offset_buffer[blockIdx.x];
  if (threadIdx_x == 0) {
    thread_to_left_pos[0] = 0;
    thread_to_right_pos[0] = 0;
  }
  __syncthreads();
  PrefixSum(thread_to_left_pos, (split_indices_block_size_data_partition << 1));
  __syncthreads();
  if (threadIdx_x > 0) {
    thread_to_right_pos[threadIdx_x] = ((threadIdx_x * 4) - thread_to_left_pos[conflict_free_threadIdx_x_plus_1]);
  }
  thread_to_right_pos[threadIdx_x + blockDim_x] = (((threadIdx_x + blockDim_x) * 4) - thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1]);
  __syncthreads();
  data_size_t* left_out_data_indices_in_leaf = out_data_indices_in_leaf + to_left_block_offset;
  data_size_t* right_out_data_indices_in_leaf = out_data_indices_in_leaf + to_right_block_offset;
  const data_size_t global_thread_index_base = global_thread_index * 4;
  const data_size_t global_thread_index_plus_blockDim_x_base = global_thread_index_plus_blockDim_x * 4;
  const uint16_t to_left_pos_offset_0 = thread_to_left_pos[conflict_free_threadIdx_x_plus_1];
  const uint16_t to_right_pos_offset_0 = thread_to_right_pos[threadIdx_x];
  const uint16_t to_left_pos_offset_1 = thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1];
  const uint16_t to_right_pos_offset_1 = thread_to_right_pos[threadIdx_x + blockDim_x];
  if (global_thread_index_base < num_data_in_leaf_ref) {
    if (bit_0 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_0] = cuda_data_indices_in_leaf[global_thread_index_base];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_0] = cuda_data_indices_in_leaf[global_thread_index_base];
    }
  }
  if (global_thread_index_base + 1 < num_data_in_leaf_ref) {
    if (bit_1 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_0 + bit_0] = cuda_data_indices_in_leaf[global_thread_index_base + 1];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_0 + 1 - bit_0] = cuda_data_indices_in_leaf[global_thread_index_base + 1];
    }
  }
  if (global_thread_index_base + 2 < num_data_in_leaf_ref) {
    if (bit_2 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_0 + bit_1_acc] = cuda_data_indices_in_leaf[global_thread_index_base + 2];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_0 + 2 - bit_1_acc] = cuda_data_indices_in_leaf[global_thread_index_base + 2];
    }
  }
  if (global_thread_index_base + 3 < num_data_in_leaf_ref) {
    if (bit_3 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_0 + bit_2_acc] = cuda_data_indices_in_leaf[global_thread_index_base + 3];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_0 + 3 - bit_2_acc] = cuda_data_indices_in_leaf[global_thread_index_base + 3];
    }
  }
  if (global_thread_index_plus_blockDim_x_base < num_data_in_leaf_ref) {
    if (bit_4 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_1] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_1] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base];
    }
  }
  if (global_thread_index_plus_blockDim_x_base + 1 < num_data_in_leaf_ref) {
    if (bit_5 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_1 + bit_4] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base + 1];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_1 + 1 - bit_4] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base + 1];
    }
  }
  if (global_thread_index_plus_blockDim_x_base + 2 < num_data_in_leaf_ref) {
    if (bit_6 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_1 + bit_5_acc] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base + 2];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_1 + 2 - bit_5_acc] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base + 2];
    }
  }
  if (global_thread_index_plus_blockDim_x_base + 3 < num_data_in_leaf_ref) {
    if (bit_7 == 1) {
      left_out_data_indices_in_leaf[to_left_pos_offset_1 + bit_6_acc] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base + 3];
    } else {
      right_out_data_indices_in_leaf[to_right_pos_offset_1 + 3 - bit_6_acc] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x_base + 3];
    }
  }
}

__global__ void CopyDataIndicesKernel(
  const data_size_t num_data_in_leaf,
  const data_size_t* out_data_indices_in_leaf,
  data_size_t* cuda_data_indices) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int global_thread_index = blockIdx.x * blockDim.x + threadIdx_x;
  if (global_thread_index < num_data_in_leaf) {
    cuda_data_indices[global_thread_index] = out_data_indices_in_leaf[global_thread_index];
  }
}

void CUDADataPartition::LaunchSplitInnerKernel(const int* leaf_index, const data_size_t num_data_in_leaf,
  const int* best_split_feature, const uint32_t* best_split_threshold,
  const uint8_t* best_split_default_left, const double* best_split_gain,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value, uint8_t* best_split_found,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  std::vector<data_size_t>* cpu_leaf_num_data, std::vector<data_size_t>* cpu_leaf_data_start,
  std::vector<double>* cpu_leaf_sum_hessians,
  int* smaller_leaf_index, int* larger_leaf_index, const int cpu_leaf_index) {
  const int min_num_blocks = num_data_in_leaf <= 100 ? 1 : 80;
  const int num_blocks = std::max(min_num_blocks, (num_data_in_leaf + SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  int split_indices_block_size_data_partition = (num_data_in_leaf + num_blocks - 1) / num_blocks - 1;
  int split_indices_block_size_data_partition_aligned = 1;
  while (split_indices_block_size_data_partition > 0) {
    split_indices_block_size_data_partition_aligned <<= 1;
    split_indices_block_size_data_partition >>= 1;
  }
  const int num_blocks_final = (num_data_in_leaf + split_indices_block_size_data_partition_aligned - 1) / split_indices_block_size_data_partition_aligned;
  int num_blocks_final_ref = num_blocks_final - 1;
  int num_blocks_final_aligned = 1;
  while (num_blocks_final_ref > 0) {
    num_blocks_final_aligned <<= 1;
    num_blocks_final_ref >>= 1;
  }
  global_timer.Start("CUDADataPartition::AggregateBlockOffsetKernel");

  if (num_blocks_final > AGGREGATE_BLOCK_SIZE) {
    AggregateBlockOffsetKernel2<<<1, AGGREGATE_BLOCK_SIZE, 0, cuda_streams_[0]>>>(leaf_index, cuda_block_data_to_left_offset_,
      cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
      cuda_leaf_num_data_, cuda_data_indices_,
      cuda_cur_num_leaves_,
      best_split_feature, best_split_threshold, best_split_default_left, best_split_gain,
      best_left_sum_gradients, best_left_sum_hessians, best_left_count,
      best_left_gain, best_left_leaf_value,
      best_right_sum_gradients, best_right_sum_hessians, best_right_count,
      best_right_gain, best_right_leaf_value,

      smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
      smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
      smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
      smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      smaller_leaf_cuda_hist_pointer_pointer,
      larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
      larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
      larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
      larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      larger_leaf_cuda_hist_pointer_pointer,
      cuda_num_total_bin_,
      cuda_hist_,
      cuda_hist_pool_,
      num_blocks_final);
  } else {
    AggregateBlockOffsetKernel3<<<1, num_blocks_final_aligned, 0, cuda_streams_[0]>>>(leaf_index, cuda_block_data_to_left_offset_,
      cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
      cuda_leaf_num_data_, cuda_data_indices_,
      cuda_cur_num_leaves_,
      best_split_feature, best_split_threshold, best_split_default_left, best_split_gain,
      best_left_sum_gradients, best_left_sum_hessians, best_left_count,
      best_left_gain, best_left_leaf_value,
      best_right_sum_gradients, best_right_sum_hessians, best_right_count,
      best_right_gain, best_right_leaf_value,

      smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
      smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
      smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
      smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      smaller_leaf_cuda_hist_pointer_pointer,
      larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
      larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
      larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
      larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      larger_leaf_cuda_hist_pointer_pointer,
      cuda_num_total_bin_,
      cuda_hist_,
      cuda_hist_pool_,
      num_blocks_final, num_blocks_final_aligned);
  }
  SynchronizeCUDADevice();
  global_timer.Stop("CUDADataPartition::AggregateBlockOffsetKernel");
  global_timer.Start("CUDADataPartition::SplitInnerKernel");

  SplitInnerKernel<<<num_blocks_final, split_indices_block_size_data_partition_aligned / 2, 0, cuda_streams_[1]>>>(
    leaf_index, cuda_cur_num_leaves_, cuda_leaf_data_start_, cuda_leaf_num_data_, cuda_data_indices_, cuda_data_to_left_,
    cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_,
    cuda_out_data_indices_in_leaf_, split_indices_block_size_data_partition_aligned);
  //SynchronizeCUDADevice();
  global_timer.Stop("CUDADataPartition::SplitInnerKernel");

  global_timer.Start("CUDADataPartition::SplitTreeStructureKernel");
  SplitTreeStructureKernel<<<4, 6, 0, cuda_streams_[0]>>>(leaf_index, cuda_block_data_to_left_offset_,
    cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
    cuda_leaf_num_data_, cuda_out_data_indices_in_leaf_,
    cuda_cur_num_leaves_,
    best_split_feature, best_split_threshold, best_split_default_left, best_split_gain,
    best_left_sum_gradients, best_left_sum_hessians, best_left_count,
    best_left_gain, best_left_leaf_value,
    best_right_sum_gradients, best_right_sum_hessians, best_right_count,
    best_right_gain, best_right_leaf_value, best_split_found,

    smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
    smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
    smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
    smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
    smaller_leaf_cuda_hist_pointer_pointer,
    larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
    larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
    larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
    larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
    larger_leaf_cuda_hist_pointer_pointer,
    cuda_num_total_bin_,
    cuda_hist_,
    cuda_hist_pool_, split_indices_block_size_data_partition_aligned,

    tree_split_leaf_index_, tree_inner_feature_index_, tree_threshold_,
    tree_left_output_, tree_right_output_, tree_left_count_, tree_right_count_,
    tree_left_sum_hessian_, tree_right_sum_hessian_, tree_gain_, tree_default_left_,
    data_partition_leaf_output_, cuda_split_info_buffer_);
  global_timer.Stop("CUDADataPartition::SplitTreeStructureKernel");
  std::vector<int> cpu_split_info_buffer(12);
  const double* cpu_sum_hessians_info = reinterpret_cast<const double*>(cpu_split_info_buffer.data() + 8);
  global_timer.Start("CUDADataPartition::CopyFromCUDADeviceToHostAsync");
  CopyFromCUDADeviceToHostAsync<int>(cpu_split_info_buffer.data(), cuda_split_info_buffer_, 12, cuda_streams_[0]);
  global_timer.Stop("CUDADataPartition::CopyFromCUDADeviceToHostAsync");
  SynchronizeCUDADevice();
  const data_size_t left_leaf_num_data = cpu_split_info_buffer[1];
  const data_size_t left_leaf_data_start = cpu_split_info_buffer[2];
  const data_size_t right_leaf_num_data = cpu_split_info_buffer[4];
  global_timer.Start("CUDADataPartition::CopyDataIndicesKernel");
  CopyDataIndicesKernel<<<num_blocks_final, split_indices_block_size_data_partition_aligned, 0, cuda_streams_[2]>>>(
    left_leaf_num_data + right_leaf_num_data, cuda_out_data_indices_in_leaf_, cuda_data_indices_ + left_leaf_data_start);
  global_timer.Stop("CUDADataPartition::CopyDataIndicesKernel");
  const int left_leaf_index = cpu_split_info_buffer[0];
  const int right_leaf_index = cpu_split_info_buffer[3];
  const data_size_t right_leaf_data_start = cpu_split_info_buffer[5];
  (*cpu_leaf_num_data)[left_leaf_index] = left_leaf_num_data;
  (*cpu_leaf_data_start)[left_leaf_index] = left_leaf_data_start;
  (*cpu_leaf_num_data)[right_leaf_index] = right_leaf_num_data;
  (*cpu_leaf_data_start)[right_leaf_index] = right_leaf_data_start;
  (*cpu_leaf_sum_hessians)[left_leaf_index] = cpu_sum_hessians_info[0];
  (*cpu_leaf_sum_hessians)[right_leaf_index] = cpu_sum_hessians_info[1];
  *smaller_leaf_index = cpu_split_info_buffer[6];
  *larger_leaf_index = cpu_split_info_buffer[7];
}

void CUDADataPartition::LaunchSplitInnerKernel2(const int* leaf_index, const data_size_t num_data_in_leaf,
  const int* best_split_feature, const uint32_t* best_split_threshold,
  const uint8_t* best_split_default_left, const double* best_split_gain,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value, uint8_t* best_split_found,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  std::vector<data_size_t>* cpu_leaf_num_data, std::vector<data_size_t>* cpu_leaf_data_start,
  std::vector<double>* cpu_leaf_sum_hessians,
  int* smaller_leaf_index, int* larger_leaf_index, const int cpu_leaf_index) {
  int block_dim = 0;
  int grid_dim = 0;
  CalcBlockDim(num_data_in_leaf, &grid_dim, &block_dim);
  int grid_dim_ref = grid_dim - 1;
  int grid_dim_aligned = 1;
  while (grid_dim_ref > 0) {
    grid_dim_aligned <<= 1;
    grid_dim_ref >>= 1;
  }
  global_timer.Start("CUDADataPartition::AggregateBlockOffsetKernel");

  if (grid_dim > AGGREGATE_BLOCK_SIZE) {
    AggregateBlockOffsetKernel2<<<1, AGGREGATE_BLOCK_SIZE, 0, cuda_streams_[0]>>>(leaf_index, cuda_block_data_to_left_offset_,
      cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
      cuda_leaf_num_data_, cuda_data_indices_,
      cuda_cur_num_leaves_,
      best_split_feature, best_split_threshold, best_split_default_left, best_split_gain,
      best_left_sum_gradients, best_left_sum_hessians, best_left_count,
      best_left_gain, best_left_leaf_value,
      best_right_sum_gradients, best_right_sum_hessians, best_right_count,
      best_right_gain, best_right_leaf_value,

      smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
      smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
      smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
      smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      smaller_leaf_cuda_hist_pointer_pointer,
      larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
      larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
      larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
      larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      larger_leaf_cuda_hist_pointer_pointer,
      cuda_num_total_bin_,
      cuda_hist_,
      cuda_hist_pool_,
      grid_dim);
  } else {
    AggregateBlockOffsetKernel3<<<1, grid_dim_aligned, 0, cuda_streams_[0]>>>(leaf_index, cuda_block_data_to_left_offset_,
      cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
      cuda_leaf_num_data_, cuda_data_indices_,
      cuda_cur_num_leaves_,
      best_split_feature, best_split_threshold, best_split_default_left, best_split_gain,
      best_left_sum_gradients, best_left_sum_hessians, best_left_count,
      best_left_gain, best_left_leaf_value,
      best_right_sum_gradients, best_right_sum_hessians, best_right_count,
      best_right_gain, best_right_leaf_value,

      smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
      smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
      smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
      smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      smaller_leaf_cuda_hist_pointer_pointer,
      larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
      larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
      larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
      larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
      larger_leaf_cuda_hist_pointer_pointer,
      cuda_num_total_bin_,
      cuda_hist_,
      cuda_hist_pool_,
      grid_dim, grid_dim_aligned);
  }
  SynchronizeCUDADevice();
  global_timer.Stop("CUDADataPartition::AggregateBlockOffsetKernel");
  global_timer.Start("CUDADataPartition::SplitInnerKernel");

  SplitInnerKernel2<<<grid_dim, block_dim, 0, cuda_streams_[1]>>>(
    leaf_index, cuda_cur_num_leaves_, cuda_leaf_data_start_, cuda_leaf_num_data_, cuda_data_indices_, cuda_data_to_left_,
    cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_,
    cuda_out_data_indices_in_leaf_, block_dim);
  //SynchronizeCUDADevice();
  global_timer.Stop("CUDADataPartition::SplitInnerKernel");

  global_timer.Start("CUDADataPartition::SplitTreeStructureKernel");
  SplitTreeStructureKernel<<<4, 6, 0, cuda_streams_[0]>>>(leaf_index, cuda_block_data_to_left_offset_,
    cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
    cuda_leaf_num_data_, cuda_out_data_indices_in_leaf_,
    cuda_cur_num_leaves_,
    best_split_feature, best_split_threshold, best_split_default_left, best_split_gain,
    best_left_sum_gradients, best_left_sum_hessians, best_left_count,
    best_left_gain, best_left_leaf_value,
    best_right_sum_gradients, best_right_sum_hessians, best_right_count,
    best_right_gain, best_right_leaf_value, best_split_found,

    smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
    smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
    smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
    smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
    smaller_leaf_cuda_hist_pointer_pointer,
    larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
    larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
    larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
    larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
    larger_leaf_cuda_hist_pointer_pointer,
    cuda_num_total_bin_,
    cuda_hist_,
    cuda_hist_pool_, block_dim,

    tree_split_leaf_index_, tree_inner_feature_index_, tree_threshold_,
    tree_left_output_, tree_right_output_, tree_left_count_, tree_right_count_,
    tree_left_sum_hessian_, tree_right_sum_hessian_, tree_gain_, tree_default_left_,
    data_partition_leaf_output_, cuda_split_info_buffer_);
  //SynchronizeCUDADevice();
  global_timer.Stop("CUDADataPartition::SplitTreeStructureKernel");
  std::vector<int> cpu_split_info_buffer(12);
  const double* cpu_sum_hessians_info = reinterpret_cast<const double*>(cpu_split_info_buffer.data() + 8);
  global_timer.Start("CUDADataPartition::CopyFromCUDADeviceToHostAsync");
  CopyFromCUDADeviceToHostAsync<int>(cpu_split_info_buffer.data(), cuda_split_info_buffer_, 12, cuda_streams_[0]);
  global_timer.Stop("CUDADataPartition::CopyFromCUDADeviceToHostAsync");
  SynchronizeCUDADevice();
  const data_size_t left_leaf_num_data = cpu_split_info_buffer[1];
  const data_size_t left_leaf_data_start = cpu_split_info_buffer[2];
  const data_size_t right_leaf_num_data = cpu_split_info_buffer[4];
  global_timer.Start("CUDADataPartition::CopyDataIndicesKernel");
  int grid_dim_copy = 0;
  int block_dim_copy = 0;
  CalcBlockDimInCopy(num_data_in_leaf, &grid_dim_copy, &block_dim_copy);
  CopyDataIndicesKernel<<<grid_dim_copy, block_dim_copy, 0, cuda_streams_[2]>>>(
    left_leaf_num_data + right_leaf_num_data, cuda_out_data_indices_in_leaf_, cuda_data_indices_ + left_leaf_data_start);
  global_timer.Stop("CUDADataPartition::CopyDataIndicesKernel");
  const int left_leaf_index = cpu_split_info_buffer[0];
  const int right_leaf_index = cpu_split_info_buffer[3];
  const data_size_t right_leaf_data_start = cpu_split_info_buffer[5];
  (*cpu_leaf_num_data)[left_leaf_index] = left_leaf_num_data;
  (*cpu_leaf_data_start)[left_leaf_index] = left_leaf_data_start;
  (*cpu_leaf_num_data)[right_leaf_index] = right_leaf_num_data;
  (*cpu_leaf_data_start)[right_leaf_index] = right_leaf_data_start;
  (*cpu_leaf_sum_hessians)[left_leaf_index] = cpu_sum_hessians_info[0];
  (*cpu_leaf_sum_hessians)[right_leaf_index] = cpu_sum_hessians_info[1];
  *smaller_leaf_index = cpu_split_info_buffer[6];
  *larger_leaf_index = cpu_split_info_buffer[7];
}

__global__ void PrefixSumKernel(uint32_t* cuda_elements) {
  __shared__ uint32_t elements[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int global_read_index = blockIdx.x * blockDim.x * 2 + threadIdx_x;
  elements[threadIdx_x] = cuda_elements[global_read_index];
  elements[threadIdx_x + blockDim.x] = cuda_elements[global_read_index + blockDim.x];
  __syncthreads();
  PrefixSum(elements, SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  __syncthreads();
  cuda_elements[global_read_index] = elements[threadIdx_x];
  cuda_elements[global_read_index + blockDim.x] = elements[threadIdx_x + blockDim.x];
}

void CUDADataPartition::LaunchPrefixSumKernel(uint32_t* cuda_elements) {
  PrefixSumKernel<<<1, SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION / 2>>>(cuda_elements);
  SynchronizeCUDADevice();
}

__global__ void AddPredictionToScoreKernel(const double* data_partition_leaf_output,
  const data_size_t* num_data_in_leaf, const data_size_t* data_indices_in_leaf,
  const data_size_t* leaf_data_start, const double learning_rate, double* cuda_scores,
  const int* cuda_data_index_to_leaf_index, const data_size_t num_data) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockIdx_x = blockIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  //const data_size_t num_data = num_data_in_leaf[blockIdx_x];
  //const data_size_t* data_indices = data_indices_in_leaf + leaf_data_start[blockIdx_x];
  const int data_index = static_cast<int>(blockIdx_x * blockDim_x + threadIdx_x);
  //const double leaf_prediction_value = data_partition_leaf_output[blockIdx_x] * learning_rate;
  /*for (unsigned int offset = 0; offset < static_cast<unsigned int>(num_data); offset += blockDim_x) {
    const data_size_t inner_data_index = static_cast<data_size_t>(offset + threadIdx_x);
    if (inner_data_index < num_data) {
      const data_size_t data_index = data_indices[inner_data_index];
      cuda_scores[data_index] += leaf_prediction_value;
    }
  }*/
  if (data_index < num_data) {
    const int leaf_index = cuda_data_index_to_leaf_index[data_index];
    const double leaf_prediction_value = data_partition_leaf_output[leaf_index] * learning_rate;
    cuda_scores[data_index] += leaf_prediction_value;
  }
}

void CUDADataPartition::LaunchAddPredictionToScoreKernel(const double learning_rate, double* cuda_scores) {
  global_timer.Start("CUDADataPartition::AddPredictionToScoreKernel");
  //const int leaf_check_size = 10500000;
  //std::vector<int> cpu_leaf_predict(leaf_check_size);
  //std::vector<double> cpu_predict_value(leaf_check_size);
  //CopyFromCUDADeviceToHost<int>(cpu_leaf_predict.data(), cuda_data_index_to_leaf_index_, leaf_check_size);
  const int num_blocks = (num_data_ + FILL_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / FILL_INDICES_BLOCK_SIZE_DATA_PARTITION;
  AddPredictionToScoreKernel<<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(data_partition_leaf_output_,
    cuda_leaf_num_data_, cuda_data_indices_, cuda_leaf_data_start_, learning_rate, cuda_scores, cuda_data_index_to_leaf_index_, num_data_);
  //SynchronizeCUDADevice();
  //global_timer.Stop("CUDADataPartition::AddPredictionToScoreKernel");
  /*for (int i = 0; i < leaf_check_size; ++i) {
    Log::Warning("cpu_leaf_predict[%d] = %d", i, cpu_leaf_predict[i]);
  }*/
  //static int iter = 0;
  //if (iter == 0) {
  //  OutputToFile("cuda_data_partition.txt", cpu_leaf_predict);
  //}
  //++iter;
}

__global__ void CopyColWiseDataKernel(const uint8_t* row_wise_data,
  const data_size_t num_data, const int num_features,
  uint8_t* col_wise_data) {
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (data_index < num_data) {
    const data_size_t read_offset = data_index * num_features;
    for (int feature_index = 0; feature_index < num_features; ++feature_index) {
      const data_size_t write_pos = feature_index * num_data + data_index;
      col_wise_data[write_pos] = row_wise_data[read_offset + feature_index];
    }
  }
}

__global__ void CUDACheckKernel(const data_size_t** data_indices_in_leaf_ptr,
  const data_size_t num_data_in_leaf,
  const score_t* gradients,
  const score_t* hessians,
  double* gradients_sum_buffer,
  double* hessians_sum_buffer) {
  const data_size_t* data_indices_in_leaf = *data_indices_in_leaf_ptr;
  const data_size_t local_data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  __shared__ double local_gradients[1024];
  __shared__ double local_hessians[1024];
  if (local_data_index < num_data_in_leaf) {
    const data_size_t global_data_index = data_indices_in_leaf[local_data_index];
    local_gradients[threadIdx.x] = gradients[global_data_index];
    local_hessians[threadIdx.x] = hessians[global_data_index];
  } else {
    local_gradients[threadIdx.x] = 0.0f;
    local_hessians[threadIdx.x] = 0.0f;
  }
  __syncthreads();
  ReduceSum(local_gradients, 1024);
  __syncthreads();
  ReduceSum(local_hessians, 1024);
  __syncthreads();
  if (threadIdx.x == 0) {
    gradients_sum_buffer[blockIdx.x] = local_gradients[0];
    hessians_sum_buffer[blockIdx.x] = local_hessians[0];
  }
}

__global__ void CUDACheckKernel2(
  const int leaf_index,
  const data_size_t* num_data_expected,
  const double* sum_gradients_expected,
  const double* sum_hessians_expected,
  const double* gradients_sum_buffer,
  const double* hessians_sum_buffer,
  const int num_blocks) {
  double sum_gradients = 0.0f;
  double sum_hessians = 0.0f;
  for (int i = 0; i < num_blocks; ++i) {
    sum_gradients += gradients_sum_buffer[i];
    sum_hessians += hessians_sum_buffer[i];
  }
  if (fabs(sum_gradients - *sum_gradients_expected) >= 1.0f) {
    printf("error in leaf_index = %d\n", leaf_index);
    printf("num data expected = %d\n", *num_data_expected);
    printf("error sum_gradients: %f vs %f\n", sum_gradients, *sum_gradients_expected);
  }
  if (fabs(sum_hessians - *sum_hessians_expected) >= 1.0f) {
    printf("error in leaf_index = %d\n", leaf_index);
    printf("num data expected = %d\n", *num_data_expected);
    printf("error sum_hessians: %f vs %f\n", sum_hessians, *sum_hessians_expected);
  }
}

void CUDADataPartition::LaunchCUDACheckKernel(
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const std::vector<data_size_t>& num_data_in_leaf,
  const CUDALeafSplits* smaller_leaf_splits,
  const CUDALeafSplits* larger_leaf_splits,
  const score_t* gradients,
  const score_t* hessians) {
  const data_size_t num_data_in_smaller_leaf = num_data_in_leaf[smaller_leaf_index];
  const int block_dim = 1024;
  const int smaller_num_blocks = (num_data_in_smaller_leaf + block_dim - 1) / block_dim;
  CUDACheckKernel<<<smaller_num_blocks, block_dim>>>(smaller_leaf_splits->cuda_data_indices_in_leaf(),
    num_data_in_smaller_leaf,
    gradients,
    hessians,
    cuda_gradients_sum_buffer_,
    cuda_hessians_sum_buffer_);
  CUDACheckKernel2<<<1, 1>>>(
    smaller_leaf_index,
    smaller_leaf_splits->cuda_num_data_in_leaf(),
    smaller_leaf_splits->cuda_sum_of_gradients(),
    smaller_leaf_splits->cuda_sum_of_hessians(),
    cuda_gradients_sum_buffer_,
    cuda_hessians_sum_buffer_,
    smaller_num_blocks);
  if (larger_leaf_index >= 0) {
    const data_size_t num_data_in_larger_leaf = num_data_in_leaf[larger_leaf_index];
    const int larger_num_blocks = (num_data_in_larger_leaf + block_dim - 1) / block_dim;
    CUDACheckKernel<<<larger_num_blocks, block_dim>>>(larger_leaf_splits->cuda_data_indices_in_leaf(),
      num_data_in_larger_leaf,
      gradients,
      hessians,
      cuda_gradients_sum_buffer_,
      cuda_hessians_sum_buffer_);
    CUDACheckKernel2<<<1, 1>>>(
      larger_leaf_index,
      larger_leaf_splits->cuda_num_data_in_leaf(),
      larger_leaf_splits->cuda_sum_of_gradients(),
      larger_leaf_splits->cuda_sum_of_hessians(),
      cuda_gradients_sum_buffer_,
      cuda_hessians_sum_buffer_,
      larger_num_blocks);
  }
}

}  // namespace LightGBM

#endif  // USE_CUDA
