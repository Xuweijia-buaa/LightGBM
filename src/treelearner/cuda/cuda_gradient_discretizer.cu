
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include <LightGBM/cuda/cuda_algorithms.hpp>

#include "cuda_gradient_discretizer.hpp"

namespace LightGBM {

__global__ void ReduceMinMaxKernel(
  const data_size_t num_data,
  const score_t* input_gradients,
  const score_t* input_hessians,
  score_t* grad_min_block_buffer,
  score_t* grad_max_block_buffer,
  score_t* hess_min_block_buffer,
  score_t* hess_max_block_buffer) {
  __shared__ score_t shared_mem_buffer[32];
  const data_size_t index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  score_t grad_max_val = kMinScore;
  score_t grad_min_val = kMaxScore;
  score_t hess_max_val = kMinScore;
  score_t hess_min_val = kMaxScore;
  if (index < num_data) {
    grad_max_val = input_gradients[index];
    grad_min_val = input_gradients[index];
    hess_max_val = input_hessians[index];
    hess_min_val = input_hessians[index];
  }
  grad_min_val = ShuffleReduceMin<score_t>(grad_min_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  grad_max_val = ShuffleReduceMax<score_t>(grad_max_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_min_val = ShuffleReduceMin<score_t>(hess_min_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_max_val = ShuffleReduceMax<score_t>(hess_max_val, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    grad_min_block_buffer[blockIdx.x] = grad_min_val;
    grad_max_block_buffer[blockIdx.x] = grad_max_val;
    hess_min_block_buffer[blockIdx.x] = hess_min_val;
    hess_max_block_buffer[blockIdx.x] = hess_max_val;
  }
}

__global__ void ReduceBlockMinMaxKernel(
  const int num_blocks,
  const int grad_discretize_bins,
  score_t* grad_min_block_buffer,
  score_t* grad_max_block_buffer,
  score_t* hess_min_block_buffer,
  score_t* hess_max_block_buffer) {
  __shared__ score_t shared_mem_buffer[32];
  score_t grad_max_val = kMinScore;
  score_t grad_min_val = kMaxScore;
  score_t hess_max_val = kMinScore;
  score_t hess_min_val = kMaxScore;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks; block_index += static_cast<int>(blockDim.x)) {
    grad_min_val = min(grad_min_val, grad_min_block_buffer[block_index]);
    grad_max_val = max(grad_max_val, grad_max_block_buffer[block_index]);
    hess_min_val = min(hess_min_val, hess_min_block_buffer[block_index]);
    hess_max_val = max(hess_max_val, hess_max_block_buffer[block_index]);
  }
  grad_min_val = ShuffleReduceMin<score_t>(grad_min_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  grad_max_val = ShuffleReduceMax<score_t>(grad_max_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_max_val = ShuffleReduceMax<score_t>(hess_max_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_max_val = ShuffleReduceMax<score_t>(hess_max_val, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    const score_t grad_abs_max = max(fabs(grad_min_val), fabs(grad_max_val));
    const score_t hess_abs_max = max(fabs(hess_min_val), fabs(hess_max_val));
    grad_min_block_buffer[0] = 1.0f / (grad_abs_max / (grad_discretize_bins / 2));
    grad_max_block_buffer[0] = (grad_abs_max / (grad_discretize_bins / 2));
    hess_min_block_buffer[0] = 1.0f / (hess_abs_max / (grad_discretize_bins));
    hess_max_block_buffer[0] = (hess_abs_max / (grad_discretize_bins));
  }
}

__global__ void DiscretizeGradientsKernel(
  const data_size_t num_data,
  const score_t* input_gradients,
  const score_t* input_hessians,
  const score_t* grad_scale_ptr,
  const score_t* hess_scale_ptr,
  const int iter,
  const int* random_values_use_start,
  const score_t* gradient_random_values,
  const score_t* hessian_random_values,
  const int grad_discretize_bins,
  int32_t* output_gradients_and_hessians) {
  const int start = random_values_use_start[iter];
  const data_size_t index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  const score_t grad_scale = *grad_scale_ptr;
  const score_t hess_scale = *hess_scale_ptr;
  int16_t* output_gradients_and_hessians_ptr = reinterpret_cast<int16_t*>(output_gradients_and_hessians);
  if (index < num_data) {
    const data_size_t index_offset = (index + start) % num_data;
    const score_t gradient = input_gradients[index];
    const score_t hessian = input_hessians[index];
    const score_t gradient_random_value = gradient_random_values[index_offset];
    const score_t hessian_random_value = hessian_random_values[index_offset];
    output_gradients_and_hessians_ptr[2 * index + 1] = gradient > 0.0f ?
      static_cast<int16_t>(gradient * grad_scale + gradient_random_value) :
      static_cast<int16_t>(gradient * grad_scale - gradient_random_value);
    output_gradients_and_hessians_ptr[2 * index] = static_cast<int16_t>(hessian * hess_scale + hessian_random_value);
  }
}

void CUDAGradientDiscretizer::DiscretizeGradients(
  const data_size_t num_data,
  const score_t* input_gradients,
  const score_t* input_hessians) {
  ReduceMinMaxKernel<<<num_reduce_blocks_, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(
    num_data, input_gradients, input_hessians,
    grad_min_block_buffer_.RawData(),
    grad_max_block_buffer_.RawData(),
    hess_min_block_buffer_.RawData(),
    hess_max_block_buffer_.RawData());
  ReduceBlockMinMaxKernel<<<1, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(
    num_reduce_blocks_,
    grad_discretize_bins_,
    grad_min_block_buffer_.RawData(),
    grad_max_block_buffer_.RawData(),
    hess_min_block_buffer_.RawData(),
    hess_max_block_buffer_.RawData());
  if (nccl_comm_ != nullptr) {
    SynchronizeCUDADevice(__FILE__, __LINE__);
    cudaStream_t cuda_stream;
    CUDASUCCESS_OR_FATAL(cudaStreamCreate(&cuda_stream));
    NCCLCHECK(ncclGroupStart());
    NCCLCHECK(ncclAllReduce(
      grad_min_block_buffer_.RawData(),
      grad_min_block_buffer_.RawData(), 1, ncclFloat32, ncclMin, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclAllReduce(
      hess_min_block_buffer_.RawData(),
      hess_min_block_buffer_.RawData(), 1, ncclFloat32, ncclMin, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclAllReduce(
      grad_max_block_buffer_.RawData(),
      grad_max_block_buffer_.RawData(), 1, ncclFloat32, ncclMax, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclAllReduce(
      hess_max_block_buffer_.RawData(),
      hess_max_block_buffer_.RawData(), 1, ncclFloat32, ncclMax, *nccl_comm_, cuda_stream));
    NCCLCHECK(ncclGroupEnd());
    CUDASUCCESS_OR_FATAL(cudaStreamSynchronize(cuda_stream));
    CUDASUCCESS_OR_FATAL(cudaStreamDestroy(cuda_stream));
  }
  DiscretizeGradientsKernel<<<num_reduce_blocks_, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(
    num_data,
    input_gradients,
    input_hessians,
    grad_min_block_buffer_.RawData(),
    hess_min_block_buffer_.RawData(),
    iter_,
    random_values_use_start_.RawData(),
    gradient_random_values_.RawData(),
    hessian_random_values_.RawData(),
    grad_discretize_bins_,
    discretized_gradients_and_hessians_.RawData());
  ++iter_;
}

__global__ void ScaleHistogramKernel(
  const int num_total_bin,
  const score_t* grad_scale_ptr,
  const score_t* hess_scale_ptr,
  CUDALeafSplitsStruct* cuda_leaf_splits,
  int32_t* histogram_ptr) {
  const score_t grad_scale = *grad_scale_ptr;
  const score_t hess_scale = *hess_scale_ptr;
  const int32_t* input_histogram = reinterpret_cast<const int32_t*>(cuda_leaf_splits->hist_in_leaf);
  hist_t* histogram = cuda_leaf_splits->hist_in_leaf;
  const int bin = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
  if (bin < num_total_bin) {
    const hist_t grad = input_histogram[((bin << 1) + 1) << 1] * grad_scale;
    const hist_t hess = input_histogram[((bin << 1)) << 1] * hess_scale;
    histogram[(bin << 1)] = grad;
    histogram[(bin << 1) + 1] = hess;
  }
}

void CUDAGradientDiscretizer::ScaleHistogram(
  const int num_total_bin, CUDALeafSplitsStruct* cuda_leaf_splits, cudaStream_t cuda_stream) const {
  const int num_blocks = (num_total_bin + CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE - 1) / CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE;
  int32_t* histogram_ptr = nullptr;
  ScaleHistogramKernel<<<num_blocks, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE, 0, cuda_stream>>>(
    num_total_bin,
    grad_max_block_buffer_.RawData(),
    hess_max_block_buffer_.RawData(),
    cuda_leaf_splits, histogram_ptr);
}

}  // namespace LightGBM

#endif  // USE_CUDA
