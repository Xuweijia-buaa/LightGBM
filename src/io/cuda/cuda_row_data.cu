#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for license information.
 */

#include <LightGBM/cuda/cuda_row_data.hpp>
#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

template <typename BIN_TYPE>
__global__ void CopySubrowDenseKernel(const BIN_TYPE* full_set_bin_data, const int num_column, const data_size_t num_used_indices,
  const data_size_t* used_indices, BIN_TYPE* bin_data) {
  const data_size_t local_data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (local_data_index < num_used_indices) {
    const data_size_t global_data_index = used_indices[local_data_index];
    const BIN_TYPE* src = full_set_bin_data + global_data_index * num_column;
    BIN_TYPE* dst = bin_data + local_data_index * num_column;
    for (int column_index = 0; column_index < num_column; ++column_index) {
      dst[column_index] = src[column_index];
    }
  }
}

void CUDARowData::LaunchCopyDenseSubrowKernel(const CUDARowData* full_set) {
  const int num_column = feature_partition_column_index_offsets_.back();
  const int num_blocks = (num_used_indices_ + COPY_SUBROW_BLOCK_SIZE_ROW_DATA - 1) / COPY_SUBROW_BLOCK_SIZE_ROW_DATA;
  if (bit_type_ == 8) {
    const uint8_t* full_set_bin_data = full_set->cuda_data_uint8_t_;
    CopySubrowDenseKernel<uint8_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      full_set_bin_data, num_column, num_used_indices_, cuda_used_indices_, cuda_data_uint8_t_);
  } else if (bit_type_ == 16) {
    const uint16_t* full_set_bin_data = full_set->cuda_data_uint16_t_;
    CopySubrowDenseKernel<uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      full_set_bin_data, num_column, num_used_indices_, cuda_used_indices_, cuda_data_uint16_t_);
  } else if (bit_type_ == 32) {
    const uint32_t* full_set_bin_data = full_set->cuda_data_uint32_t_;
    CopySubrowDenseKernel<uint32_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      full_set_bin_data, num_column, num_used_indices_, cuda_used_indices_, cuda_data_uint32_t_);
  }
}

template <typename ROW_PTR_TYPE>
__global__ void CalcTotalNumberOfElementsKernel(
  const data_size_t num_used_indices,
  const data_size_t* cuda_used_indices,
  const ROW_PTR_TYPE* cuda_row_ptr,
  const int num_feature_partitions,
  const data_size_t num_data,
  uint64_t* block_sum_buffer) {
  __shared__ uint64_t shared_mem_buffer[32];
  const data_size_t local_data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  const int partition_index = static_cast<int>(blockIdx.y);
  const ROW_PTR_TYPE* partition_row_ptr = cuda_row_ptr + partition_index * (num_data + 1);
  uint64_t num_elements_in_row = 0;
  if (local_data_index < num_used_indices) {
    const data_size_t global_data_index = cuda_used_indices[local_data_index];
    const data_size_t row_start = partition_row_ptr[global_data_index];
    const data_size_t row_end = partition_row_ptr[global_data_index + 1];
    num_elements_in_row += static_cast<uint64_t>(row_end - row_start);
  }
  const uint64_t num_elements_in_block = ShuffleReduceSum<uint64_t>(num_elements_in_row, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    block_sum_buffer[partition_index * blockDim.x + blockIdx.x] = num_elements_in_block;
  }
}

__global__ void ReduceBlockSumKernel(
  const uint64_t* block_sum_buffer,
  const int num_blocks,
  const int num_feature_partitions,
  uint64_t* cuda_partition_ptr_buffer) {
  __shared__ uint64_t shared_mem_buffer[32];
  uint64_t thread_sum = 0;
  const int partition_index = static_cast<int>(blockIdx.y);
  const uint64_t* block_sum_buffer_ptr = block_sum_buffer + partition_index * blockDim.x;
  for (data_size_t block_index = static_cast<data_size_t>(threadIdx.x); block_index < num_blocks; ++block_index) {
    thread_sum += block_sum_buffer_ptr[block_index];
  }
  const uint64_t num_total_elements = ShuffleReduceSum<uint64_t>(thread_sum, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_partition_ptr_buffer[partition_index + 1] = num_total_elements;
    if (blockIdx.x == 0) {
      cuda_partition_ptr_buffer[0] = 0;
    }
  }
}

__global__ void ComputePartitionPtr(
  uint64_t* cuda_partition_ptr_buffer,
  const int num_feature_partitions) {
  __shared__ uint64_t shared_mem_buffer[32];
  const int num_partitions_per_thread = (num_feature_partitions + blockDim.x - 1) / (blockDim.x - 1);
  int start_partition = threadIdx.x == 0 ? 0 : num_partitions_per_thread * static_cast<int>(threadIdx.x - 1);
  int end_partition = threadIdx.x == 0 ? 0 : min(start_partition + num_partitions_per_thread, num_feature_partitions + 1);
  uint64_t thread_sum = 0;
  for (int partition_index = start_partition; partition_index < end_partition; ++partition_index) {
    thread_sum += cuda_partition_ptr_buffer[partition_index];
  }
  const uint64_t thread_base = ShufflePrefixSum<uint64_t>(thread_sum, shared_mem_buffer);
  start_partition = threadIdx.x == blockDim.x - 1 ? 0 : num_partitions_per_thread * static_cast<int>(threadIdx.x);
  end_partition = threadIdx.x == blockDim.x - 1 ? 0 : min(start_partition + num_partitions_per_thread, num_feature_partitions + 1);
  for (int partition_index = start_partition + 1; partition_index < end_partition; ++partition_index) {
    cuda_partition_ptr_buffer[partition_index] += cuda_partition_ptr_buffer[partition_index - 1];
  }
  for (int partition_index = start_partition; partition_index < end_partition; ++partition_index) {
    cuda_partition_ptr_buffer[partition_index] += thread_base;
  }
  if (threadIdx.x == blockDim.x - 1) {
    cuda_partition_ptr_buffer[num_feature_partitions] = thread_sum;
  }
}

uint64_t CUDARowData::LaunchCalcTotalNumberOfElementsKernel(const CUDARowData* full_set) {
  const int num_blocks = (num_data_ + COPY_SUBROW_BLOCK_SIZE_ROW_DATA - 1) / COPY_SUBROW_BLOCK_SIZE_ROW_DATA;
  SetCUDAMemoryOuter<uint64_t>(cuda_block_sum_buffer_, 0, static_cast<size_t>(num_blocks * num_feature_partitions_) + 1, __FILE__, __LINE__);
  if (full_set->row_ptr_bit_type_ == 16) {
    CalcTotalNumberOfElementsKernel<uint16_t><<<dim3(num_blocks, num_feature_partitions_), COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      num_used_indices_,
      cuda_used_indices_,
      full_set->cuda_row_ptr_uint16_t_,
      num_feature_partitions_,
      num_data_,
      cuda_block_sum_buffer_);
  } else if (full_set->row_ptr_bit_type_ == 32) {
    CalcTotalNumberOfElementsKernel<uint32_t><<<dim3(num_blocks, num_feature_partitions_), COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      num_used_indices_,
      cuda_used_indices_,
      full_set->cuda_row_ptr_uint32_t_,
      num_feature_partitions_,
      num_data_,
      cuda_block_sum_buffer_);
  } else if (full_set->row_ptr_bit_type_ == 64) {
    CalcTotalNumberOfElementsKernel<uint64_t><<<dim3(num_blocks, num_feature_partitions_), COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      num_used_indices_,
      cuda_used_indices_,
      full_set->cuda_row_ptr_uint64_t_,
      num_feature_partitions_,
      num_data_,
      cuda_block_sum_buffer_);
  }
  ReduceBlockSumKernel<<<num_feature_partitions_, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
    cuda_block_sum_buffer_, num_blocks, num_feature_partitions_, cuda_partition_ptr_buffer_);
  ComputePartitionPtr<<<1, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(cuda_partition_ptr_buffer_, num_feature_partitions_);
  uint64_t num_total_elements = 0;
  CopyFromCUDADeviceToHostOuter<uint64_t>(&num_total_elements, cuda_partition_ptr_buffer_, num_feature_partitions_, __FILE__, __LINE__);
  return num_total_elements;
}

template <typename ROW_PTR_TYPE>
__global__ void CopyPartitionPtrKernel(
  const uint64_t* cuda_partition_ptr_buffer,
  const int num_feature_partitions,
  ROW_PTR_TYPE* cuda_partition_ptr) {
  for (int partition_index = static_cast<int>(threadIdx.x); partition_index < num_feature_partitions + 1; partition_index += static_cast<int>(blockDim.x)) {
    cuda_partition_ptr[partition_index] = static_cast<ROW_PTR_TYPE>(cuda_partition_ptr_buffer[partition_index]);
  }
}

template <typename IN_ROW_PTR_TYPE, typename OUT_ROW_PTR_TYPE>
__global__ void CopySparseSubrowRowPtrKernel(
  const IN_ROW_PTR_TYPE* cuda_row_ptr,
  const data_size_t num_used_indices,
  const data_size_t* cuda_used_indices,
  OUT_ROW_PTR_TYPE* out_cuda_row_ptr) {
  const data_size_t local_data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (local_data_index > num_used_indices) {
    const data_size_t global_data_index = cuda_used_indices[local_data_index];
    const IN_ROW_PTR_TYPE row_start = cuda_row_ptr[global_data_index];
    const IN_ROW_PTR_TYPE row_end = cuda_row_ptr[global_data_index + 1];
    const OUT_ROW_PTR_TYPE num_elements_in_row = static_cast<OUT_ROW_PTR_TYPE>(row_end - row_start);
    out_cuda_row_ptr[local_data_index + 1] = num_elements_in_row;
  }
}

template <typename BIN_TYPE, typename ROW_PTR_TYPE>
__global__ void CopySparseSubrowDataKernel(
  const BIN_TYPE* in_cuda_data,
  const ROW_PTR_TYPE* cuda_row_ptr,
  const data_size_t num_used_indices,
  const data_size_t* cuda_used_indices,
  BIN_TYPE* out_cuda_data) {
  const data_size_t local_data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (local_data_index < num_used_indices) {
    const data_size_t global_data_index = cuda_used_indices[local_data_index];
    const ROW_PTR_TYPE row_start = cuda_row_ptr[global_data_index];
    const ROW_PTR_TYPE row_end = cuda_row_ptr[global_data_index + 1];
    const ROW_PTR_TYPE num_elements_in_row = row_end - row_start;
    const BIN_TYPE* in_cuda_data_ptr = in_cuda_data + row_start; 
    BIN_TYPE* out_cuda_data_ptr = out_cuda_data + row_start;
    for (ROW_PTR_TYPE element_index = 0; element_index < num_elements_in_row; ++element_index) {
      out_cuda_data_ptr[element_index] = in_cuda_data_ptr[element_index];
    }
  }
}

void CUDARowData::LaunchCopySparseSubrowKernel(const CUDARowData* full_set) {
  if (row_ptr_bit_type_ == 16) {
    CopyPartitionPtrKernel<<<1, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(cuda_partition_ptr_buffer_, num_feature_partitions_, cuda_partition_ptr_uint16_t_);
  } else if (row_ptr_bit_type_ == 32) {
    CopyPartitionPtrKernel<<<1, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(cuda_partition_ptr_buffer_, num_feature_partitions_, cuda_partition_ptr_uint32_t_);
  } else if (row_ptr_bit_type_ == 64) {
    CopyPartitionPtrKernel<<<1, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(cuda_partition_ptr_buffer_, num_feature_partitions_, cuda_partition_ptr_uint64_t_);
  }
  const int num_blocks = (num_used_indices_ + COPY_SUBROW_BLOCK_SIZE_ROW_DATA - 1) / COPY_SUBROW_BLOCK_SIZE_ROW_DATA;
  if (full_set->row_ptr_bit_type_ == 16) {
    CHECK_EQ(row_ptr_bit_type_, 16);
    CopySparseSubrowRowPtrKernel<uint16_t, uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
      full_set->cuda_row_ptr_uint16_t_, num_used_indices_, cuda_used_indices_, cuda_row_ptr_uint16_t_);
  } else if (full_set->row_ptr_bit_type_ == 32) {
    CHECK(row_ptr_bit_type_ == 16 || row_ptr_bit_type_ == 32);
    if (row_ptr_bit_type_ == 16) {
      CopySparseSubrowRowPtrKernel<uint32_t, uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_row_ptr_uint32_t_, num_used_indices_, cuda_used_indices_, cuda_row_ptr_uint16_t_);
    } else if (row_ptr_bit_type_ == 32) {
      CopySparseSubrowRowPtrKernel<uint32_t, uint32_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_row_ptr_uint32_t_, num_used_indices_, cuda_used_indices_, cuda_row_ptr_uint32_t_);
    }
  } else if (full_set->row_ptr_bit_type_ == 64) {
    if (row_ptr_bit_type_ == 16) {
      CopySparseSubrowRowPtrKernel<uint64_t, uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_row_ptr_uint64_t_, num_used_indices_, cuda_used_indices_, cuda_row_ptr_uint16_t_);
    } else if (row_ptr_bit_type_ == 32) {
      CopySparseSubrowRowPtrKernel<uint64_t, uint32_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_row_ptr_uint64_t_, num_used_indices_, cuda_used_indices_, cuda_row_ptr_uint32_t_);
    } else if (row_ptr_bit_type_ == 64) {
      CopySparseSubrowRowPtrKernel<uint64_t, uint64_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_row_ptr_uint64_t_, num_used_indices_, cuda_used_indices_, cuda_row_ptr_uint64_t_);
    }
  }
  if (row_ptr_bit_type_ == 16) {
    ShufflePrefixSumGlobal<uint16_t>(
      cuda_row_ptr_uint16_t_,
      static_cast<size_t>(num_used_indices_) + 1,
      reinterpret_cast<uint16_t*>(cuda_block_sum_buffer_));
    if (bit_type_ == 8) {
      CopySparseSubrowDataKernel<uint8_t, uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint8_t_, cuda_row_ptr_uint16_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint8_t_);
    } else if (bit_type_ == 16) {
      CopySparseSubrowDataKernel<uint16_t, uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint16_t_, cuda_row_ptr_uint16_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint16_t_);
    } else if (bit_type_ == 32) {
      CopySparseSubrowDataKernel<uint32_t, uint16_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint32_t_, cuda_row_ptr_uint16_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint32_t_);
    }
  } else if (row_ptr_bit_type_ == 32) {
    ShufflePrefixSumGlobal<uint32_t>(
      cuda_row_ptr_uint32_t_,
      static_cast<size_t>(num_used_indices_) + 1,
      reinterpret_cast<uint32_t*>(cuda_block_sum_buffer_));
    if (bit_type_ == 8) {
      CopySparseSubrowDataKernel<uint8_t, uint32_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint8_t_, cuda_row_ptr_uint32_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint8_t_);
    } else if (bit_type_ == 16) {
      CopySparseSubrowDataKernel<uint16_t, uint32_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint16_t_, cuda_row_ptr_uint32_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint16_t_);
    } else if (bit_type_ == 32) {
      CopySparseSubrowDataKernel<uint32_t, uint32_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint32_t_, cuda_row_ptr_uint32_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint32_t_);
    }
  } else if (row_ptr_bit_type_ == 64) {
    ShufflePrefixSumGlobal<uint64_t>(
      cuda_row_ptr_uint64_t_,
      static_cast<size_t>(num_used_indices_) + 1,
      reinterpret_cast<uint64_t*>(cuda_block_sum_buffer_));
    if (bit_type_ == 8) {
      CopySparseSubrowDataKernel<uint8_t, uint64_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint8_t_, cuda_row_ptr_uint64_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint8_t_);
    } else if (bit_type_ == 16) {
      CopySparseSubrowDataKernel<uint16_t, uint64_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint16_t_, cuda_row_ptr_uint64_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint16_t_);
    } else if (bit_type_ == 32) {
      CopySparseSubrowDataKernel<uint32_t, uint64_t><<<num_blocks, COPY_SUBROW_BLOCK_SIZE_ROW_DATA>>>(
        full_set->cuda_data_uint32_t_, cuda_row_ptr_uint64_t_, num_used_indices_, cuda_used_indices_, cuda_data_uint32_t_);
    }
  }
}

}  // namespace LightGBM
