#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for license information.
 */

#include <LightGBM/cuda/cuda_tree.hpp>

namespace LightGBM {

__global__ void AddPredictionToScoreKernel(
  // dataset information
  const data_size_t num_data,
  void* const* cuda_data_by_column,
  const int8_t* cuda_column_bit_type,
  const uint32_t* cuda_feature_min_bin,
  const uint32_t* cuda_feature_max_bin,
  const uint32_t* cuda_feature_offset,
  const uint32_t* cuda_feature_default_bin,
  const int* cuda_feature_to_column,
  // tree information
  const uint32_t* cuda_threshold_in_bin,
  const int8_t* cuda_decision_type,
  const int* cuda_split_feature_inner,
  const int* cuda_left_child,
  const int* cuda_right_child,
  const double* cuda_leaf_value,
  // output
  double* score) {
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (data_index < num_data) {
    int node = 0;
    while (node >= 0) {
      const int split_feature_inner = cuda_split_feature_inner[node];
      const int column = cuda_feature_to_column[split_feature_inner];
      const uint32_t default_bin = cuda_feature_default_bin[split_feature_inner];
      const uint32_t max_bin = cuda_feature_max_bin[split_feature_inner];
      const int8_t column_bit_type = cuda_column_bit_type[column];
      uint32_t bin = 0;
      if (column_bit_type == 8) {
        bin = static_cast<uint32_t>((reinterpret_cast<const uint8_t*>(cuda_data_by_column[column]))[data_index]);
      } else if (column_bit_type == 16) {
        bin = static_cast<uint32_t>((reinterpret_cast<const uint16_t*>(cuda_data_by_column[column]))[data_index]);
      } else if (column_bit_type == 32) {
        bin = static_cast<uint32_t>((reinterpret_cast<const uint32_t*>(cuda_data_by_column[column]))[data_index]);
      }
      const int8_t decision_type = cuda_decision_type[node];
      const uint32_t threshold_in_bin = cuda_threshold_in_bin[node];
      const int8_t missing_type = ((decision_type >> 2) & 3);
      const bool default_left = ((decision_type & kDefaultLeftMask) > 0);
      if ((missing_type == 1 && bin == default_bin) || (missing_type == 2 && bin == max_bin)) {
        if (default_left) {
          node = cuda_left_child[node];
        } else {
          node = cuda_right_child[node];
        }
      } else {
        if (bin <= threshold_in_bin) {
          node = cuda_left_child[node];
        } else {
          node = cuda_right_child[node];
        }
      }
      if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("bin = %d, threshold_in_bin = %d\n", bin, threshold_in_bin);
      }
    }
    if (blockIdx.x == 0 && threadIdx.x == 0) {
      printf("node = %d, leaf_value = %f\n", ~node, cuda_leaf_value[~node]);
    }
    score[data_index] += cuda_leaf_value[~node];
  }
}

void CUDATree::LaunchAddPredictionToScoreKernel(
  const Dataset* data,
  data_size_t num_data,
  double* score) const {
  const CUDAColumnData* cuda_column_data = data->cuda_column_data();
  if (cuda_column_data == nullptr) {
    Log::Warning("error cuda_column_data is nullptr");
  }
  const int num_blocks = (num_data + num_threads_per_block_add_prediction_to_score_ - 1) / num_threads_per_block_add_prediction_to_score_;
  // TODO(shiyu1994): consider the offset caused by feature grouping when extracting the bin values from column data
  AddPredictionToScoreKernel<<<num_blocks, num_threads_per_block_add_prediction_to_score_>>>(
    // dataset information
    num_data,
    cuda_column_data->cuda_data_by_column(),
    cuda_column_data->cuda_column_bit_type(),
    cuda_column_data->cuda_feature_min_bin(),
    cuda_column_data->cuda_feature_max_bin(),
    cuda_column_data->cuda_feature_offset(),
    cuda_column_data->cuda_feature_default_bin(),
    cuda_column_data->cuda_feature_to_column(),
    // tree information
    cuda_threshold_in_bin_,
    cuda_decision_type_,
    cuda_split_feature_inner_,
    cuda_left_child_,
    cuda_right_child_,
    cuda_leaf_value_,
    // output
    score);
}

__global__ void ShrinkageKernel(const double rate, double* cuda_leaf_value) {
  const int leaf_index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
  cuda_leaf_value[leaf_index] *= rate;
}

void CUDATree::LaunchShrinkageKernel(const double rate) {
  const int num_threads_per_block = 1024;
  const int num_blocks = (num_leaves_ + num_threads_per_block - 1) / num_threads_per_block;
  ShrinkageKernel<<<num_blocks, num_threads_per_block>>>(rate, cuda_leaf_value_);
}

}  // namespace LightGBM
